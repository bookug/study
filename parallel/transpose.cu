#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "time.h"
#define MAXN 8192
#define MAX_SIZE MAXN*MAXN
#define DELTA 1e-6
			
float ori_mx[MAX_SIZE];
float mx1[MAX_SIZE], mx2[MAX_SIZE], mx3[MAX_SIZE];
 
void simple_cpu_trans(const int n, float* m1, float* m2)
{
     for (int i = 0; i < n; ++i)
	     for (int j = 0; j < n; ++j)
	         m2[i * n + j] = m1[j * n + i];
}
 
__global__ void simple_gpu_trans(const int n, float* m1, float* m2)
{
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     int j = blockIdx.y * blockDim.y + threadIdx.y;
     m2[i * n + j] = m1[j * n + i];
}
 
__global__ void advanced_gpu_trans(const int n, float* m1, float* m2)
{
     int i = threadIdx.x;
     int j = threadIdx.y;
     int i_ = blockIdx.x * blockDim.x + i;
     int j_ = blockIdx.y * blockDim.y + j;
     int i__ = blockIdx.x * blockDim.x + j;
     int j__ = blockIdx.y * blockDim.y + i;
     __shared__ float mems[16 * 16];
	mems[i * 16 + j] = m1[j_ * n + i_];
    __syncthreads();
   m2[i__ * n + j__] = mems[j * 16 + i];
}
	 
bool check(const int n, float* m1, float* m2)
{
	for (int i = 0; i < n * n; ++i)
		if (abs(m1[i]-m2[i]) > DELTA) return false;
		   return true;
}
			  
int main()
{
	srand(time(0));
	for (int i = 0; i < MAX_SIZE; ++i)
		ori_mx[i] = rand();
		 
	int start_cpu_time = clock();
	simple_cpu_trans(MAXN, ori_mx, mx1);
	int end_cpu_time = clock();
	double cpu_cost_time = (double)(end_cpu_time - start_cpu_time) / CLOCKS_PER_SEC * 1000;
	printf("CPU cost %.3lf ms.\n", cpu_cost_time);
				
	float *mxd1, *mxd2;
	if(hipMalloc((void**)&mxd1, MAX_SIZE * sizeof(float)) != hipSuccess)
			printf("cuda malloc failed!\n");
	if (hipMalloc((void**)&mxd2, MAX_SIZE * sizeof(float)) != hipSuccess)
			printf("cuda malloc failed!\n");
	if (hipMemcpy(mxd1, ori_mx, MAX_SIZE * sizeof(float), hipMemcpyHostToDevice))
			printf("cuda memcpy successfully!\n");
					
	hipEvent_t start_gpu, end_gpu;
	hipEventCreate(&start_gpu);
	hipEventCreate(&end_gpu);
	hipEventRecord(start_gpu, 0);

	dim3 dimBlock(16, 16);
	dim3 dimGrid(MAXN / 16, MAXN / 16); //here requires: MAXN % 16 == 0

	simple_gpu_trans << <dimGrid, dimBlock >> > (MAXN, mxd1, mxd2);
    hipEventRecord(end_gpu, 0);
	hipEventSynchronize(end_gpu);
	
	float gpu_cost_time;
	hipEventElapsedTime(&gpu_cost_time, start_gpu, end_gpu);
	printf("GPU cost %.3lf ms.\n", gpu_cost_time);
	hipEventDestroy(start_gpu);
	hipEventDestroy(end_gpu);
	
	if (hipMemcpy(mx2, mxd2, MAX_SIZE * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess)
		printf("cuda memcpy failed!\n");
		
	if (check(MAXN, mx1, mx2))
		printf("check passed.\n");
	else
		printf("check failed.\n");
			   
	hipEventCreate(&start_gpu);
	hipEventCreate(&end_gpu);
	hipEventRecord(start_gpu, 0);
			
	advanced_gpu_trans << <dimGrid, dimBlock >> > (MAXN, mxd1, mxd2);
	hipEventRecord(end_gpu, 0);
	hipEventSynchronize(end_gpu);
			   
	hipEventElapsedTime(&gpu_cost_time, start_gpu, end_gpu);
	printf("GPU cost %.3lf ms.\n", gpu_cost_time);
	hipEventDestroy(start_gpu);
	hipEventDestroy(end_gpu);

    if (hipMemcpy(mx3, mxd2, MAX_SIZE * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess)
	    printf("cuda memcpy failed!\n");
    if (check(MAXN, mx1, mx3))
		printf("check passed.\n");
	else
		printf("check failed.\n");
	hipFree(mxd1);
	hipFree(mxd2);
	return 0;
}

