#include "hip/hip_runtime.h"
//NOTICE: multiply can be used to compute spmm and spmv, with user-defined combine/reduce operators
//http://cusplibrary.github.io/group__matrix__algorithms.html#ga64aac53ca20d88d71aadc7e8b3317788

//NOTICE: array2d is not csr structure! 
//And GPU is not used in this file!

#include <cusp/array1d.h>
#include <cusp/array2d.h>
#include <cusp/detail/functional.h>
#include <cusp/multiply.h>
#include <cusp/print.h>
#include <cusp/csr_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/transpose.h> 

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <iostream> 

using namespace std; 

/*#define SIZE 2*/
#define SIZE 2000

/*__global__ void*/
/*test_kernel(array1d<int,cusp::device_memory> p)*/
/*{*/
    /*p[0] = 1;*/
/*}*/

int main(void)
{
    /*cusp::detail::zero_function<int> initialize;  //C=A*B*/
    /*thrust::plus<int> combine;*/
    /*thrust::plus<int>       reduce;*/
  // Allocate storage for a 5 by 8 sparse matrix in CSR format with 12
  // nonzero entries on the host
  cusp::csr_matrix<int,float,cusp::host_memory> A(4,4,6);
    // initialize matrix entries on host
    A.row_offsets[0] = 0;  // first offset is always zero
      A.row_offsets[1] = 2;
        A.row_offsets[2] = 2;
          A.row_offsets[3] = 3;
            A.row_offsets[4] = 6; // last offset is always num_entries
            A.row_offsets[5] = 6; // last offset is always num_entries
              A.column_indices[0] = 0; A.values[0] = 10;
                A.column_indices[1] = 2; A.values[1] = 20;
                  A.column_indices[2] = 2; A.values[2] = 30;
                    A.column_indices[3] = 0; A.values[3] = 40;
                      A.column_indices[4] = 1; A.values[4] = 50;
                        A.column_indices[5] = 2; A.values[5] = 60;
                          // A now represents the following matrix
                          //    [10  0 20 0]
                          //    [ 0  0  0 0 ]
                          //    [ 0  0 30 0 ]
                          //    [40 50 60 0]
                        cusp::print(A);
                        //The output is COO format.
  cusp::csr_matrix<int,float,cusp::host_memory> At;
                        cusp::transpose(A, At);
                        cusp::print(At);

        // Transfer the matrix to the device
    cusp::csr_matrix<int,float,cusp::device_memory> dA(A);
          // Convert the matrix to HYB format on the device
          /*cusp::hyb_matrix<int,float,cusp::device_memory> csr_device(csr_device);*/
    // initialize matrix
    cusp::csr_matrix<int,float,cusp::device_memory> dB(A);
    cusp::csr_matrix<int,float,cusp::device_memory> dC(A);
    // compute C = A * B
    /*cusp::multiply(A, B, C, initialize, combine, reduce);*/
    cusp::multiply(dA, dB, dC);
    /*getchar();*/
    cusp::print(dC);
    cout<<dC.num_rows<<" "<<dC.num_cols<<" "<<dC.num_entries<<" "<<dC.row_offsets[0]<<endl;
    int* hc = new int[dC.num_rows+1];
    //NOTICE: the row_offsets is array_1d type
    /*hipMemcpy(hc, dC.row_offsets, sizeof(int)*(dC.num_rows+1), hipMemcpyDeviceToHost);*/
    /*test_kernel<<<1,1>>>(dC.row_offsets);*/
    //TODO: no way to transform array1d into cuda pointer, or use it in my own cuda kernels?

    return 0;
}

