#include <thrust/version.h>
#include <cusp/version.h>
#include <iostream>
int main(void)
{
    int cuda_major =  CUDA_VERSION / 1000;
    int cuda_minor = (CUDA_VERSION % 1000) / 10;
    int thrust_major = THRUST_MAJOR_VERSION;
    int thrust_minor = THRUST_MINOR_VERSION;
    int cusp_major = CUSP_MAJOR_VERSION;
    int cusp_minor = CUSP_MINOR_VERSION;
    std::cout << "CUDA   v" << cuda_major   << "." << cuda_minor   << std::endl;
    std::cout << "Thrust v" << thrust_major << "." << thrust_minor << std::endl;
    std::cout << "Cusp   v" << cusp_major   << "." << cusp_minor   << std::endl;
    return 0;
}

