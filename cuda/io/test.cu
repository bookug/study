
#include <hip/hip_runtime.h>
#include <stdio.h>

//WARN: this setting will cause the kernel function unable to work
//As a result, we should set block num < 65536
/*#define NUM_BLOCKS 65536*/
#define NUM_BLOCKS 400
/*#define BLOCK_WIDTH 1*/
#define BLOCK_WIDTH 1024

__global__ void hello()
{
	/*__shared__ int array[128];*/
	printf("Hello, world! I am a thread in block %d\n", blockIdx.x);
	/*__syncthreads();*/
}

int main(int argc, const char* argv[])
{
	//NOTICE: this API is out-ofo-date
	/*cudaPrintfInit(100*NUM_BLOCKS*BLOCK_WIDTH);*/
	size_t io_buffer_size = 0;
	hipDeviceGetLimit(&io_buffer_size, hipLimitPrintfFifoSize);
	printf("io buffer size: %u\n", io_buffer_size);   //1M by default
	//NOTICE: no need to assign space for all threads' output, because there are at most 3840 threads running in parallel really
	//The former analysis is wrong!  IO buffer is flushed at the end of kernel execution, so overwriting will occur!
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 400*1024*200);
	hipDeviceGetLimit(&io_buffer_size, hipLimitPrintfFifoSize);
	printf("io buffer size: %u\n", io_buffer_size);   //1M by default

	hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

	//force the printf()s to flush
	hipDeviceSynchronize();

	printf("That's all!\n");

	return 0;
}

