//In CUDA, the memory is as follows:
//For a thread, local memory
//For a block, shared memory
//For all threads, global memory


#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 1
#define BLOCK_WIDTH 128

//__global__ indicates the kernel function, running on GPu, but launched by cpu
__global__ void hello()
{
	//NOTICE: __syncthreads is used within a block
	__shared__ int array[128];
	int idx = threadIdx.x;
	//sync all writes
	array[idx] = idx;
	__syncthreads();
	if(idx < 127)
	{
		int temp = array[idx+1];
		//sync all reads
		__syncthreads();
		array[idx] = temp;
		//sync all writes
		__syncthreads();
	}
	printf("Hello, world! I am thread %d, the value of array is %d\n", idx, array[idx]);
}

int main(int argc, const char* argv[])
{
	hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
	//NOTICE: different kernels are implicitly synchronized

	//force the printf()s to flush
	hipDeviceSynchronize();

	printf("That's all!\n");

	return 0;
}

