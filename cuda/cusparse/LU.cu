#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "Utilities.cuh"

hipsparseHandle_t    handle; 

hipsparseMatDescr_t  descrA      = 0;
hipsparseMatDescr_t  descr_L     = 0;
hipsparseMatDescr_t  descr_U     = 0;

csrilu02Info_t      info_A      = 0;
csrsv2Info_t        info_L      = 0;
csrsv2Info_t        info_U      = 0;

void                *pBuffer    = 0;

/*****************************/
/* SETUP DESCRIPTOR FUNCTION */
/*****************************/
void setUpDescriptor(hipsparseMatDescr_t &descrA, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase) {
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseSetMatType(descrA, matrixType));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, indexBase));
}

/**************************************************/
/* SETUP DESCRIPTOR FUNCTION FOR LU DECOMPOSITION */
/**************************************************/
void setUpDescriptorLU(hipsparseMatDescr_t &descrLU, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase, hipsparseFillMode_t fillMode, hipsparseDiagType_t diagType) {
    cusparseSafeCall(hipsparseCreateMatDescr(&descrLU));
    cusparseSafeCall(hipsparseSetMatType(descrLU, matrixType));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrLU, indexBase));
    cusparseSafeCall(hipsparseSetMatFillMode(descrLU, fillMode));
    cusparseSafeCall(hipsparseSetMatDiagType(descrLU, diagType));
}

/**********************************************/
/* MEMORY QUERY FUNCTION FOR LU DECOMPOSITION */
/**********************************************/
void memoryQueryLU(csrilu02Info_t &info_A, csrsv2Info_t &info_L, csrsv2Info_t &info_U, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t descr_L,
                   hipsparseMatDescr_t descr_U, double *d_A, int *d_A_RowIndices, int *d_A_ColIndices, hipsparseOperation_t matrixOperation, void **pBuffer) {

    cusparseSafeCall(hipsparseCreateCsrilu02Info(&info_A));
    cusparseSafeCall(hipsparseCreateCsrsv2Info(&info_L));
    cusparseSafeCall(hipsparseCreateCsrsv2Info(&info_U));

    int pBufferSize_M, pBufferSize_L, pBufferSize_U;
    cusparseSafeCall(hipsparseDcsrilu02_bufferSize(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, &pBufferSize_M));
    cusparseSafeCall(hipsparseDcsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, &pBufferSize_L));
    cusparseSafeCall(hipsparseDcsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, &pBufferSize_U));

    int pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));
    gpuErrchk(hipMalloc((void**)pBuffer, pBufferSize));

}

/******************************************/
/* ANALYSIS FUNCTION FOR LU DECOMPOSITION */
/******************************************/
void analysisLUDecomposition(csrilu02Info_t &info_A, csrsv2Info_t &info_L, csrsv2Info_t &info_U, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t descr_L,
    hipsparseMatDescr_t descr_U, double *d_A, int *d_A_RowIndices, int *d_A_ColIndices, hipsparseOperation_t matrixOperation, hipsparseSolvePolicy_t solvePolicy1, hipsparseSolvePolicy_t solvePolicy2, void *pBuffer) {

    int structural_zero;

    cusparseSafeCall(hipsparseDcsrilu02_analysis(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solvePolicy1, pBuffer));
    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){ printf("A(%d,%d) is missing\n", structural_zero, structural_zero); }

    cusparseSafeCall(hipsparseDcsrsv2_analysis(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, solvePolicy1, pBuffer));
    cusparseSafeCall(hipsparseDcsrsv2_analysis(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, solvePolicy2, pBuffer));

}

/************************************************/
/* COMPUTE LU DECOMPOSITION FOR SPARSE MATRICES */
/************************************************/
void computeSparseLU(csrilu02Info_t &info_A, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, double *d_A, int *d_A_RowIndices,
                     int *d_A_ColIndices, hipsparseSolvePolicy_t solutionPolicy ,void *pBuffer) {

    int numerical_zero;

    cusparseSafeCall(hipsparseDcsrilu02(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solutionPolicy, pBuffer));
    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){ printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero); }

}

void solveSparseLinearSystem() {


}

/********/
/* MAIN */
/********/
int main()
{
    // --- Initialize cuSPARSE
    cusparseSafeCall(hipsparseCreate(&handle));

    /**************************/
    /* SETTING UP THE PROBLEM */
    /**************************/
    const int Nrows = 4;                        // --- Number of rows
    const int Ncols = 4;                        // --- Number of columns
    const int N = Nrows;

    // --- Host side dense matrix
    double *h_A_dense = (double*)malloc(Nrows * Ncols * sizeof(*h_A_dense));

    // --- Column-major ordering
    h_A_dense[0] = 0.4612f;  h_A_dense[4] = -0.0006f;   h_A_dense[8] = 0.3566f; h_A_dense[12] = 0.0f;
    h_A_dense[1] = -0.0006f; h_A_dense[5] = 0.4640f;    h_A_dense[9] = 0.0723f; h_A_dense[13] = 0.0f;
    h_A_dense[2] = 0.3566f;  h_A_dense[6] = 0.0723f;    h_A_dense[10] = 0.7543f; h_A_dense[14] = 0.0f;
    h_A_dense[3] = 0.f;      h_A_dense[7] = 0.0f;       h_A_dense[11] = 0.0f;    h_A_dense[15] = 0.1f;

    // --- Create device array and copy host array to it
    double *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, Nrows * Ncols * sizeof(*d_A_dense)));
    gpuErrchk(hipMemcpy(d_A_dense, h_A_dense, Nrows * Ncols * sizeof(*d_A_dense), hipMemcpyHostToDevice));

    // --- Allocating and defining dense host and device data vectors
    double *h_x = (double *)malloc(Nrows * sizeof(double));
    h_x[0] = 100.0;  h_x[1] = 200.0; h_x[2] = 400.0; h_x[3] = 500.0;

    double *d_x;        gpuErrchk(hipMalloc(&d_x, Nrows * sizeof(double)));
    gpuErrchk(hipMemcpy(d_x, h_x, Nrows * sizeof(double), hipMemcpyHostToDevice));

    /*******************************/
    /* FROM DENSE TO SPARSE MATRIX */
    /*******************************/
    // --- Descriptor for sparse matrix A
    setUpDescriptor(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE);

    int nnz = 0;                                // --- Number of nonzero elements in dense matrix
    const int lda = Nrows;                      // --- Leading dimension of dense matrix
    // --- Device side number of nonzero elements per row
    int *d_nnzPerVector;    gpuErrchk(hipMalloc(&d_nnzPerVector, Nrows * sizeof(*d_nnzPerVector)));
    // --- Compute the number of nonzero elements per row and the total number of nonzero elements in the dense d_A_dense
    cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, &nnz));
    // --- Host side number of nonzero elements per row
    int *h_nnzPerVector = (int *)malloc(Nrows * sizeof(*h_nnzPerVector));
    gpuErrchk(hipMemcpy(h_nnzPerVector, d_nnzPerVector, Nrows * sizeof(*h_nnzPerVector), hipMemcpyDeviceToHost));

    printf("Number of nonzero elements in dense matrix = %i\n\n", nnz);
    for (int i = 0; i < Nrows; ++i) printf("Number of nonzero elements in row %i = %i \n", i, h_nnzPerVector[i]);
    printf("\n");

    // --- Device side sparse matrix
    double *d_A;            gpuErrchk(hipMalloc(&d_A, nnz * sizeof(*d_A)));
    int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (Nrows + 1) * sizeof(*d_A_RowIndices)));
    int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnz * sizeof(*d_A_ColIndices)));

    cusparseSafeCall(hipsparseDdense2csr(handle, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, d_A, d_A_RowIndices, d_A_ColIndices));

    // --- Host side sparse matrix
    double *h_A = (double *)malloc(nnz * sizeof(*h_A));
    int *h_A_RowIndices = (int *)malloc((Nrows + 1) * sizeof(*h_A_RowIndices));
    int *h_A_ColIndices = (int *)malloc(nnz * sizeof(*h_A_ColIndices));
    gpuErrchk(hipMemcpy(h_A, d_A, nnz*sizeof(*h_A), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_A_RowIndices, d_A_RowIndices, (Nrows + 1) * sizeof(*h_A_RowIndices), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_A_ColIndices, d_A_ColIndices, nnz * sizeof(*h_A_ColIndices), hipMemcpyDeviceToHost));

    printf("\nOriginal matrix in CSR format\n\n");
    for (int i = 0; i < nnz; ++i) printf("A[%i] = %.0f ", i, h_A[i]); printf("\n");

    printf("\n");
    for (int i = 0; i < (Nrows + 1); ++i) printf("h_A_RowIndices[%i] = %i \n", i, h_A_RowIndices[i]); printf("\n");

    for (int i = 0; i < nnz; ++i) printf("h_A_ColIndices[%i] = %i \n", i, h_A_ColIndices[i]);

    /******************************************/
    /* STEP 1: CREATE DESCRIPTORS FOR L AND U */
    /******************************************/
    setUpDescriptorLU(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE, HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_UNIT);
    setUpDescriptorLU(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    /**************************************************************************************************/
    /* STEP 2: QUERY HOW MUCH MEMORY USED IN LU FACTORIZATION AND THE TWO FOLLOWING SYSTEM INVERSIONS */
    /**************************************************************************************************/
    memoryQueryLU(info_A, info_L, info_U, handle, N, nnz, descrA, descr_L, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_OPERATION_NON_TRANSPOSE, &pBuffer);

    /************************************************************************************************/
    /* STEP 3: ANALYZE THE THREE PROBLEMS: LU FACTORIZATION AND THE TWO FOLLOWING SYSTEM INVERSIONS */
    /************************************************************************************************/
    analysisLUDecomposition(info_A, info_L, info_U, handle, N, nnz, descrA, descr_L, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, 
                            HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);

    /************************************/
    /* STEP 4: FACTORIZATION: A = L * U */
    /************************************/
    computeSparseLU(info_A, handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    /*********************/
    /* STEP 5: L * z = x */
    /*********************/
    // --- Allocating the intermediate result vector
    double *d_z;        gpuErrchk(hipMalloc(&d_z, N * sizeof(double)));

    const double alpha = 1.;
    cusparseSafeCall(hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, d_x, d_z, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer));

    /*********************/
    /* STEP 5: U * y = z */
    /*********************/
    // --- Allocating the result vector
    double *d_y;        gpuErrchk(hipMalloc(&d_y, Ncols * sizeof(double)));

    cusparseSafeCall(hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, d_z, d_y, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer));

    /********************************/
    /* MOVE THE RESULTS TO THE HOST */
    /********************************/
    double *h_y = (double *)malloc(Ncols * sizeof(double));
    gpuErrchk(hipMemcpy(h_x, d_y, N * sizeof(double), hipMemcpyDeviceToHost));
    printf("\n\nFinal result\n");
    for (int k = 0; k<N; k++) printf("x[%i] = %f\n", k, h_x[k]);
}



