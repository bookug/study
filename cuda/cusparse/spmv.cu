//DEBUG: compile error

// *** spmv_example.c ***
// How to compile (assume CUDA is installed at /usr/local/cuda/)
//   nvcc spmv_example.c -o spmv_example -L/usr/local/cuda/lib64 -lcusparse -lcudart
// or, for C compiler
//   cc -I/usr/local/cuda/include -c spmv_example.c -o spmv_example.o -std=c99
//   nvcc -lcusparse -lcudart spmv_example.o -o spmv_example
#include <hip/hip_runtime.h>  // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>      // hipsparseSpMV
#include <stdio.h>         // printf
#include <stdlib.h>        // EXIT_FAILURE

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int main() {
    // Host problem definition
    const int A_num_rows = 4;
    const int A_num_cols = 4;
    const int A_num_nnz  = 9;
    int   hA_csrOffsets[] = { 0, 3, 4, 7, 9 };
    int   hA_columns[]    = { 0, 2, 3, 1, 0, 2, 3, 1, 3 };
    float hA_values[]     = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                              6.0f, 7.0f, 8.0f, 9.0f };
    float hX[]            = { 1.0f, 2.0f, 3.0f, 4.0f };
    const float result[]  = { 19.0f, 8.0f, 51.0f, 52.0f };
    float alpha = 1.0f;
    float beta  = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dX, *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_num_nnz * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_values, A_num_nnz * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dX, A_num_cols * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dY, A_num_rows * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_num_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values,
                           A_num_nnz * sizeof(float), hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, hX, A_num_rows * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = 0;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*  dBuffer    = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_num_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_32F) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_32F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUSPARSE( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    float hY[A_num_rows];
    CHECK_CUDA( hipMemcpy(hY, dY, A_num_rows * sizeof(float),
                           hipMemcpyDeviceToHost) )

    int correct = 1;
    for (int i = 0; i < A_num_rows; i++) {
        if (hY[i] != result[i]) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("spmv_example test PASSED\n");
    else
        printf("spmv_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dX) )
    CHECK_CUDA( hipFree(dY) )
    return EXIT_SUCCESS;
}



