//NOTICE: if only use cublas, it is ok to name this file as *.c or *.cpp
//To compile, add -lcublas


// CUDA runtime 库 + CUBLAS 库 
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <time.h>
#include <iostream>
#include <stdio.h> 

using namespace std;

// 定义测试矩阵的维度
int const M = 5;
int const N = 10;

int main() 
{   
    // 定义状态变量
    hipblasStatus_t status;

    // 在 内存 中为将要计算的矩阵开辟空间
    float *h_A = (float*)malloc (N*M*sizeof(float));
    float *h_B = (float*)malloc (N*M*sizeof(float));
    
    // 在 内存 中为将要存放运算结果的矩阵开辟空间
    float *h_C = (float*)malloc (M*M*sizeof(float));

    // 为待运算矩阵的元素赋予 0-10 范围内的随机数
    for (int i=0; i<N*M; i++) {
        h_A[i] = (float)(rand()%10+1);
        h_B[i] = (float)(rand()%10+1);
    
    }
    
    // 打印待测试的矩阵
    cout << "矩阵 A :" << endl;
    for (int i=0; i<N*M; i++){
        cout << h_A[i] << " ";
        if ((i+1)%N == 0) cout << endl;
    }
    cout << endl;
    cout << "矩阵 B :" << endl;
    for (int i=0; i<N*M; i++){
        cout << h_B[i] << " ";
        if ((i+1)%M == 0) cout << endl;
    }
    cout << endl;
    
    /*
    ** GPU 计算矩阵相乘
    */

    // 创建并初始化 CUBLAS 库对象
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
            cout << "CUBLAS 对象实例化出错" << endl;
        }
        getchar ();
        return EXIT_FAILURE;
    }

    float *d_A, *d_B, *d_C;
    // 在 显存 中为将要计算的矩阵开辟空间
    hipMalloc (
        (void**)&d_A,    // 指向开辟的空间的指针
        N*M * sizeof(float)    //　需要开辟空间的字节数
    );
    hipMalloc (
        (void**)&d_B,    
        N*M * sizeof(float)    
    );

    // 在 显存 中为将要存放运算结果的矩阵开辟空间
    hipMalloc (
        (void**)&d_C,
        M*M * sizeof(float)    
    );

    // 将矩阵数据传递进 显存 中已经开辟好了的空间
    hipblasSetVector (
        N*M,    // 要存入显存的元素个数
        sizeof(float),    // 每个元素大小
        h_A,    // 主机端起始地址
        1,    // 连续元素之间的存储间隔
        d_A,    // GPU 端起始地址
        1    // 连续元素之间的存储间隔
    );
    hipblasSetVector (
        N*M, 
        sizeof(float), 
        h_B, 
        1, 
        d_B, 
        1
    );

    // 同步函数
    hipDeviceSynchronize();

    // 传递进矩阵相乘函数中的参数，具体含义请参考函数手册。
    float a=1; float b=0;
    // 矩阵相乘。该函数必然将数组解析成列优先数组
    hipblasSgemm (
        handle,    // blas 库对象 
        HIPBLAS_OP_T,    // 矩阵 A 属性参数
        HIPBLAS_OP_T,    // 矩阵 B 属性参数
        M,    // A, C 的行数 
        M,    // B, C 的列数
        N,    // A 的列数和 B 的行数
        &a,    // 运算式的 α 值
        d_A,    // A 在显存中的地址
        N,    // lda
        d_B,    // B 在显存中的地址
        M,    // ldb
        &b,    // 运算式的 β 值
        d_C,    // C 在显存中的地址(结果矩阵)
        M    // ldc
    );
    
    // 同步函数
    hipDeviceSynchronize();

	//NOTICE: it is ok to use hipMemcpy
    // 从 显存 中取出运算结果至 内存中去
    hipblasGetVector (
        M*M,    //  要取出元素的个数
        sizeof(float),    // 每个元素大小
        d_C,    // GPU 端起始地址
        1,    // 连续元素之间的存储间隔
        h_C,    // 主机端起始地址
        1    // 连续元素之间的存储间隔
    );
    
    // 打印运算结果
    cout << "计算结果的转置 ( (A*B)的转置 )：" << endl;

    for (int i=0;i<M*M; i++){
            cout << h_C[i] << " ";
            if ((i+1)%M == 0) cout << endl;
    }
    
    // 清理掉使用过的内存
    free (h_A);
    free (h_B);
    free (h_C);
    hipFree (d_A);
    hipFree (d_B);
    hipFree (d_C);

    // 释放 CUBLAS 库对象
    hipblasDestroy (handle);

    getchar();
    
    return 0;
}
