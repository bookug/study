#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_runtime.h"
#include "time.h"

#define MAXN 33554432
#define DELTA 1e-6
#define BN 512

int ori_arr[MAXN], ori_arr2[MAXN * 2]; 
int arr1[MAXN * 2], arr2[MAXN * 2]; 

void simple_cpu_presum(const int n, int* a1, int* a2) 
{
	int tmp = 0;
	for (int i = 0; i < n; ++i)
	{   
		a2[i] = a1[i] + tmp;
		tmp += a1[i];
	}								       
}

__global__ void gpu_presum_up(int* a, int l, int r)
{
	int thid = threadIdx.x;
	int i = l + blockIdx.x * blockDim.x + thid;		
	if (i < r)
		a[i] = a[i * 2] + a[i * 2 + 1]; 
}

__global__ void gpu_presum_down(int* a, int l, int r)
{
	int thid = threadIdx.x;
	int i = l + blockIdx.x * blockDim.x + thid;
	if (i < r)
	{
		a[i * 2 + 1] = a[2 * i] + a[i];
		a[2 * i] = a[i];
	}
}

__global__ void gpu_presum_floor(int* a, int l, int r)
{
	int thid = threadIdx.x;
	int i = l + blockIdx.x * blockDim.x + thid;
	if (i < r)
	{
		a[i * 2 + 1] += a[2 * i] + a[i];
		a[2 * i] += a[i];
	}
}

bool check(const int n, int* a1, int* a2)
{
	for (int i = 0; i < MAXN; ++i)
	{
		if (a1[i] != a2[i + MAXN]) 
			return false;
	}
	return true;
}

int main()
{
	srand(time(0));
	for (int i = 0; i < MAXN; ++i)
	{
		ori_arr[i] = rand();
		ori_arr2[i] = 0;
		ori_arr2[i + MAXN] = ori_arr[i];
	}

	int start_cpu_time = clock();
	simple_cpu_presum(MAXN, ori_arr, arr1);
	int end_cpu_time = clock();
	float cpu_cost_time = (float)(end_cpu_time - start_cpu_time) / CLOCKS_PER_SEC * 1000;
	printf("CPU cost %.3lf ms.\n", cpu_cost_time);							
								
	int *arrd1;
	if (hipMalloc((void**)&arrd1, 2*MAXN * sizeof(int)) != hipSuccess)
		printf("cuda malloc failed!\n");
	if (hipMemcpy(arrd1, ori_arr2, 2*MAXN * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
		printf("cuda memcpy failed!\n");
											
	hipEvent_t start_gpu, end_gpu;
	hipEventCreate(&start_gpu);
	hipEventCreate(&end_gpu);
	hipEventRecord(start_gpu, 0);
															
	int left = MAXN / 2, right = MAXN;
	while (left != 1)
	{
		dim3 dimBlock(BN);
		int block_num = (right - left) / BN;
		if ((right - left) & (BN-1))
			++block_num;
		dim3 dimGrid(block_num);
		gpu_presum_up << <dimGrid, dimBlock >> > (arrd1, left, right);
		left /= 2;
		right /= 2;
	}

    left = 1;
	right = 2;
	while (left != MAXN)
	{
		dim3 dimBlock(BN);
		int block_num = (right - left) / BN;
		if ((right - left) & (BN-1))
			++block_num;
		dim3 dimGrid(block_num);
		if (left*2 != MAXN)
			gpu_presum_down << <dimGrid, dimBlock >> > (arrd1, left, right);
		else
			gpu_presum_floor << <dimGrid, dimBlock >> > (arrd1, left, right);

		left *= 2;
		right *= 2;
	}

	hipEventRecord(end_gpu, 0);
	hipEventSynchronize(end_gpu);

	float gpu_cost_time;
	hipEventElapsedTime(&gpu_cost_time, start_gpu, end_gpu);
	printf("GPU cost %.3lf ms.\n", gpu_cost_time);
	hipEventDestroy(start_gpu);
	hipEventDestroy(end_gpu);
    if (hipMemcpy(arr2, arrd1,2 * MAXN * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
		printf("cuda memcpy failed!\n");
	if (check(MAXN, arr1, arr2))
		printf("check passed.\n");
	else
		printf("check failed.\n");
	hipFree(arrd1);
	return 0;
}

