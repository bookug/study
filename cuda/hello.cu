
#include <hip/hip_runtime.h>
#include <stdio.h>

//WARN: this setting will cause the kernel function unable to work
//As a result, we should set block num < 65536
/*#define NUM_BLOCKS 65536*/
#define NUM_BLOCKS 400
/*#define BLOCK_WIDTH 1*/
#define BLOCK_WIDTH 1024

__global__ void hello()
{
	/*__shared__ int array[128];*/
	printf("Hello, world! I am a thread in block %d\n", blockIdx.x);
	/*__syncthreads();*/
}

int main(int argc, const char* argv[])
{
	/*cudaPrintfInit(100*NUM_BLOCKS*BLOCK_WIDTH);*/

	hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

	//force the printf()s to flush
	hipDeviceSynchronize();

	printf("That's all!\n");

	return 0;
}

