
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello()
{
	/*__shared__ int array[128];*/
	printf("Hello, world! I am a thread in block %d\n", blockIdx.x);
	/*__syncthreads();*/
}

int main(int argc, const char* argv[])
{
	hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

	//force the printf()s to flush
	hipDeviceSynchronize();

	printf("That's all!\n");

	return 0;
}

