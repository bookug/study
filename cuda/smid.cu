//https://gist.github.com/allanmac/4751080
//https://codeday.me/bug/20190709/1417531.html
//compile: nvcc smid.cu -o smid
//QUESTION: a SM may have several blocks at the same moment.
//and, if only 3 block (we ensure the shared memory usage of each block, then 1 SM can only run 1 SM), they may  not be scheduled to run on SM 0-2?
//NOTICE: if we want to perform scalability test wrt the number of SMs, we do not need to care if SM 0-2 are used, we only need to ensure only 3 Sms are used.
//However, each block run a large portion of tasks, this may be severely imbalanced. The original strategy is to issue many blocks and let GPU schedule them freely.
//A better strategy of our case, will be to extract tasks from a queue iteratively. (But the maintainence of queue may bring an observable cost)
//
//STRATEGY: it seems that even without the restriction of shared memory, when the block num <= SM num, they will be scheduled to let only one block run on one SM.
//(It is understandable because the resource of one SM is limited, so one block a SM will deliver the best performance, when SMs are enough.)


#include <hip/hip_runtime.h>
#include <stdio.h>

#define DEVICE_INTRINSIC_QUALIFIERS   __device__ __forceinline__

DEVICE_INTRINSIC_QUALIFIERS
unsigned int
smid()
{
  unsigned int r;
  asm("mov.u32 %0, %%smid;" : "=r"(r));
  return r;
}

DEVICE_INTRINSIC_QUALIFIERS
unsigned int
nsmid()
{
#if (__CUDA_ARCH__ >= 200)
  unsigned int r;
  asm("mov.u32 %0, %%nsmid;" : "=r"(r));
  return r;
#else
  return 30;
#endif
}

__device__ int g_nsmid[1];
__device__ int g_smid[64] = {
  -1, -1, -1, -1, -1, -1, -1, -1,
  -1, -1, -1, -1, -1, -1, -1, -1,
  -1, -1, -1, -1, -1, -1, -1, -1,
  -1, -1, -1, -1, -1, -1, -1, -1,
  -1, -1, -1, -1, -1, -1, -1, -1,
  -1, -1, -1, -1, -1, -1, -1, -1,
  -1, -1, -1, -1, -1, -1, -1, -1,
  -1, -1, -1, -1, -1, -1, -1, -1
};

__global__
void
smidTest()
{
    /*__shared__ int s_pool[1024];*/
    /*__shared__ int s_pool[12*1024];*/
    /*s_pool[0] = 10;*/
    /*s_pool[0] += 2;*/
    //NOTICE: the applied dynamic shared memory is not explicitly used, but they are really allocated.

  const int n = nsmid();
  const int s = smid();

  //NOTICE: if below is commented, no -1 generated
    /*if(blockIdx.x > 20)*/
    /*{*/
        /*printf("found: %d\n", s);*/
    /*}*/

  g_nsmid[0] = n;
  int x = n;
  x *= n;
  if(g_smid[s] == -1)
  g_smid[s]  = s;
  else
      g_smid[s] = -1;

  if(threadIdx.x == 0)
  {
      printf("sm %d\n", s);
  }
}

int main(int argc, char **argv)
{
  hipError_t err;
  int         device = (argc == 1) ? 0 : atoi(argv[1]);

  hipDeviceProp_t props;
  err = hipGetDeviceProperties(&props,device);

  if (err) 
    return -1;

  // if (props.major < 2) {
  //   printf("%s = sm_%d%d\n",props.name,props.major,props.minor);
  //   return -1;
  // }

  hipSetDevice(device);
                            
  //
  // LAUNCH KERNEL
  //
  
  //NOTICE: the setting of shared memory usage is to ensure only one block reside in each SM.
  /*smidTest<<<props.multiProcessorCount,1,props.sharedMemPerBlock-384>>>();*/
  smidTest<<<props.multiProcessorCount,1>>>();
  /*smidTest<<<props.multiProcessorCount,1024>>>();*/
  /*smidTest<<<props.multiProcessorCount+3,1>>>();*/
  /*smidTest<<<props.multiProcessorCount*2,1>>>();*/
  /*smidTest<<<3,1>>>();*/
  hipDeviceSynchronize();

  //
  // LOOK AT RESULTS
  //

  int h_nsmid[1];
  int h_smid[48];

  hipMemcpyFromSymbol(h_nsmid,HIP_SYMBOL(g_nsmid),sizeof(h_nsmid));
  hipMemcpyFromSymbol(h_smid, HIP_SYMBOL(g_smid), sizeof(h_smid));

  printf("%s (%2d) [ %2d",
         props.name,
         (h_nsmid[0] == 30) ? props.multiProcessorCount : h_nsmid[0],
         h_smid[0]);

  int last = 0;
  
  for (int ii=1; ii<h_nsmid[0]; ii++)
    {
      if (h_smid[ii] != -1)
        last = ii;
    }

  for (int ii=1; ii<=last; ii++)
    {
      const int s = h_smid[ii];
      if (s == -1)
        printf(", --");
      else
        printf(", %2d",s);
    }

  printf(" ]\n");

  return 0;
}

