#include "hip/hip_runtime.h"
//HELP: http://blog.csdn.net/augusdi/article/details/12833235
//QUERY: how to assign a specific or several GPUs?
//Shared Memory: http://tech.it168.com/a2011/0708/1215/000001215209_1.shtml
//WARP and BANK: http://blog.163.com/wujiaxing009@126/blog/static/71988399201712735436357/
//hipDeviceSynchronize: http://blog.csdn.net/mathgeophysics/article/details/19905935

//performance optimization
//http://blog.csdn.net/litdaguang/article/details/50520549

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cassert>
#include <cmath>
#include <iostream>

using namespace std;

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void checkWarp_kernel()
{
	__shared__ unsigned s_val;
	if(threadIdx.x > 31)
	{
		return; 
	}
	if(threadIdx.x == 0)
	{
		s_val = 0;
	}
	atomicAdd(&s_val,1);
	/*s_val++;*/
	/*printf("block: %u thread: %u\n", blockIdx.x, threadIdx.x);*/
	if(threadIdx.x == 0)
	{
		printf("check warp: %u\n", s_val);
	}
}

__global__ void
memory_kernel(int** d_m)
{
	/*d_m[0] = new int[2];*/
	d_m[0][0] = 1;
	printf("gpu pointer: %lu\n", d_m[0]);
}

int main()
{
	int **d_m;
	hipMalloc(&d_m, sizeof(int*));
  checkCudaErrors(hipGetLastError());
  int** h_m = new int*[1];
	hipMalloc(&h_m[0], sizeof(int));
  checkCudaErrors(hipGetLastError());
	hipMemcpy(d_m, h_m, sizeof(int*), hipMemcpyHostToDevice);
  checkCudaErrors(hipGetLastError());
	printf("check pointer: %lu\n", h_m[0]);
	memory_kernel<<<1,1>>>(d_m);
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
	hipFree(h_m[0]);
  checkCudaErrors(hipGetLastError());
	hipFree(d_m);
  checkCudaErrors(hipGetLastError());
	hipFree(d_m);
  checkCudaErrors(hipGetLastError());

	//check the output: 32 threads in a warp add 1 to the same variable
	checkWarp_kernel<<<1, 32>>>();
	
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
