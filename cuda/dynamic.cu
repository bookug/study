#include "hip/hip_runtime.h"
//To compile cuda program using dynamic parallelism
//nvcc -arch=sm_35 -lcudadevrt -rdc=true dynamic.cu

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream> 

using namespace std;

__device__ int v = 0;

__global__ void child_k(void)
{
	printf("v = %d\n", v);
}

/*__global__ void parent_k(int* data)*/
__global__ void parent_k()
{
	/*int d_data = 3;*/
	/*hipMemcpy(data, &d_data, sizeof(int), hipMemcpyDeviceToHost);*/
	printf("Hello, World!\n");
	v = 1;
	child_k<<<1,1>>>();
	v = 2; //race condition
	hipDeviceSynchronize();
}

int main(int argc, char **argv)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    hipSetDevice(dev);

	//NOTE: 48KB shared memory per block, 1024 threads per block, 28 SMs
    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz; shared mem: %dB; block threads: %d; SM count: %d\n",
               devProps.name, (int)devProps.totalGlobalMem, 
               (int)devProps.major, (int)devProps.minor, 
               (int)devProps.clockRate,
			   devProps.sharedMemPerBlock, devProps.maxThreadsPerBlock, devProps.multiProcessorCount);
    }

	/*int data;*/
	/*parent_k<<<1,1>>>(&data);*/
	/*cout<<"data: "<<data<<endl;*/
	parent_k<<<1,1>>>();
        
    return 0;
}
