#include "hip/hip_runtime.h"
//To compile cuda program using dynamic parallelism
//nvcc -arch=sm_35 -lcudadevrt -rdc=true dynamic.cu

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ int v = 0;

__global__ void child_k(void)
{
	printf("v = %d\n", v);
}

__global__ void parent_k(void)
{
	printf("Hello, World!\n");
	v = 1;
	child_k<<<1,1>>>();
	v = 2; //race condition
	hipDeviceSynchronize();
}

int main(int argc, char **argv)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    hipSetDevice(dev);

	//NOTE: 48KB shared memory per block, 1024 threads per block, 28 SMs
    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz; shared mem: %dB; block threads: %d; SM count: %d\n",
               devProps.name, (int)devProps.totalGlobalMem, 
               (int)devProps.major, (int)devProps.minor, 
               (int)devProps.clockRate,
			   devProps.sharedMemPerBlock, devProps.maxThreadsPerBlock, devProps.multiProcessorCount);
    }

	parent_k<<<1,1>>>();
        
    return 0;
}
