// or equivalently <cub/device/device_scan.cuh>
#include <hipcub/hipcub.hpp> 
#include <stdio.h>

using namespace std; 

//http://nvlabs.github.io/cub/structcub_1_1_device_scan.html#details
//self-defined operator

int main()
{
    // Declare, allocate, and initialize device-accessible pointers for input and output
    int  num_items = 7;
    unsigned h_in[] = {8,6,7,5,3,0,9};
    unsigned h_out[8];

    unsigned  *d_in = NULL;
    hipMalloc(&d_in, sizeof(unsigned)*7);
    hipMemcpy(d_in, h_in, sizeof(unsigned)*7, hipMemcpyHostToDevice);

    //NOTICE: they can be the same!
    unsigned *d_out = d_in;

    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
    // d_out s<-- [0, 8, 14, 21, 26, 29, 29]

    hipMemcpy(h_out, d_out, sizeof(unsigned)*7, hipMemcpyDeviceToHost);
    for(int i = 0; i < 7; ++i)
    {
        cout<<h_out[i]<<" ";
    }cout<<endl;

    hipFree(d_in);
    //d_out is same as d_in and the same area should not be freed twice
    /*hipFree(d_out);*/
    //NOTICE: this is valid
    hipFree(NULL);
    //this is needed
    hipFree(d_temp_storage);

    return 0;
}

