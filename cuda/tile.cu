#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"
/*#include "utils.h"*/

const int BLOCKSIZE	= 128;
const int NUMBLOCKS = 1000;					// set this to 1 or 2 for debugging
const int N 		= BLOCKSIZE*NUMBLOCKS;

/* 
 * TODO: modify the foo and bar kernels to use tiling: 
 * 		 - copy the input data to shared memory
 *		 - perform the computation there
 *	     - copy the result back to global memory
 *		 - assume thread blocks of 128 threads
 *		 - handle intra-block boundaries correctly
 * You can ignore boundary conditions (we ignore the first 2 and last 2 elements)
 */
__global__ void foo(float out[], float A[], float B[], float C[], float D[], float E[]){
	__shared__ float tmp[BLOCKSIZE];
	int i = threadIdx.x + blockIdx.x*blockDim.x; 
	if(i >= N)
	{
		return; 
	}
	tmp[threadIdx.x] = (A[i] + B[i] + C[i] + D[i] + E[i]) / 5.0f;
	__syncthreads();
	out[i] = tmp[threadIdx.x];
}

__global__ void bar(float out[], float in[]) 
{
	__shared__ float tmp[BLOCKSIZE+4];
	int i = threadIdx.x + blockIdx.x*blockDim.x; 
	tmp[threadIdx.x+2] = in[i];
	if(threadIdx.x == 0 && i >= 2)
	{
		tmp[0] = in[i-2];
		tmp[1] = in[i-1];
	}
	else if(threadIdx.x == BLOCKSIZE - 1 && i <= N-3)
	{
		tmp[BLOCKSIZE+2] = in[i+1];
		tmp[BLOCKSIZE+3] = in[i+2];
	}
	__syncthreads();
	if(i < 2 || i > N-3)
	{
		return; 
	}

	out[i] = (tmp[threadIdx.x] + tmp[threadIdx.x+1] + tmp[threadIdx.x+2] + tmp[threadIdx.x+3] + tmp[threadIdx.x+4]) / 5.0f;
}

void cpuFoo(float out[], float A[], float B[], float C[], float D[], float E[])
{
	for (int i=0; i<N; i++)
	{
		out[i] = (A[i] + B[i] + C[i] + D[i] + E[i]) / 5.0f;
	}
}

void cpuBar(float out[], float in[])
{
	// ignore the boundaries
	for (int i=2; i<N-2; i++)
	{
		out[i] = (in[i-2] + in[i-1] + in[i] + in[i+1] + in[i+2]) / 5.0f;
	}
}

void compareArrays(float* arr1, float* arr2, const int N)
{
	for(int i = 2; i < N-2; ++i)
	{
		if(arr1[i] != arr2[i])
		{
			printf("not matched: %d\n", i);
			return; 
		}
	}
	printf("all matched!\n");
}

int main(int argc, char **argv)
{
	// declare and fill input arrays for foo() and bar()
	float fooA[N], fooB[N], fooC[N], fooD[N], fooE[N], barIn[N];
	for (int i=0; i<N; i++) 
	{
		fooA[i] = i; 
		fooB[i] = i+1;
		fooC[i] = i+2;
		fooD[i] = i+3;
		fooE[i] = i+4;
		barIn[i] = 2*i; 
	}
	// device arrays
	int numBytes = N * sizeof(float);
	float *d_fooA;	 	hipMalloc(&d_fooA, numBytes);
	float *d_fooB; 		hipMalloc(&d_fooB, numBytes);
	float *d_fooC;	 	hipMalloc(&d_fooC, numBytes);
	float *d_fooD; 		hipMalloc(&d_fooD, numBytes);
	float *d_fooE; 		hipMalloc(&d_fooE, numBytes);
	float *d_barIn; 	hipMalloc(&d_barIn, numBytes);
	hipMemcpy(d_fooA, fooA, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_fooB, fooB, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_fooC, fooC, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_fooD, fooD, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_fooE, fooE, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_barIn, barIn, numBytes, hipMemcpyHostToDevice);	

	// output arrays for host and device
	float fooOut[N], barOut[N], *d_fooOut, *d_barOut;
	hipMalloc(&d_fooOut, numBytes);
	hipMalloc(&d_barOut, numBytes);

	// declare and compute reference solutions
	float ref_fooOut[N], ref_barOut[N]; 
	cpuFoo(ref_fooOut, fooA, fooB, fooC, fooD, fooE);
	cpuBar(ref_barOut, barIn);

	// launch and time foo and bar
	GpuTimer fooTimer, barTimer;
	fooTimer.Start();
	foo<<<N/BLOCKSIZE, BLOCKSIZE>>>(d_fooOut, d_fooA, d_fooB, d_fooC, d_fooD, d_fooE);
	fooTimer.Stop();
	
	barTimer.Start();
	bar<<<N/BLOCKSIZE, BLOCKSIZE>>>(d_barOut, d_barIn);
	barTimer.Stop();

	hipMemcpy(fooOut, d_fooOut, numBytes, hipMemcpyDeviceToHost);
	hipMemcpy(barOut, d_barOut, numBytes, hipMemcpyDeviceToHost);
	printf("foo<<<>>>(): %g ms elapsed. Verifying solution...", fooTimer.Elapsed());
	compareArrays(ref_fooOut, fooOut, N);
	printf("bar<<<>>>(): %g ms elapsed. Verifying solution...", barTimer.Elapsed());
	compareArrays(ref_barOut, barOut, N);
}
