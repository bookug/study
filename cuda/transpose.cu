#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"
/*#include "utils.h"*/

void fill_matrix(float* in, int N)
{
	for(int i = 0; i < N; ++i)
	{
		for(int j = 0; j < N; ++j)
		{
			in[i*N+j] = i*N+j;
		}
	}
	return;
}

bool compare_matrices(float* out, float* gold, int N)
{
	bool ret = true;
	for(int i = 0; i < N; ++i)
	{
		for(int j = 0; j < N; ++j)
		{
			if(out[i*N+j] != gold[i*N+j])
			{
				ret = false; 
				break;
			}
		}
	}
	return ret;
}

const int N= 1024;	
const int K= 16;		// TODO, set K to the correct value and tile size will be KxK
//8x8, 16x16, 32x32, 64x64(this is tricker)
//NOTICE: 16x16 is the best
//64x64 will run a very small time, and it is right!(why)

// to be launched with one thread per element, in (tilesize)x(tilesize) threadblocks
// thread blocks read & write tiles, in coalesced fashion
// adjacent threads read adjacent input elements, write adjacent output elmts
__global__ void 
transpose_parallel_per_element_tiled(float in[], float out[])
{
	// (i,j) locations of the tile corners for input & output matrices:
	int in_corner_i  = blockIdx.x * K, in_corner_j  = blockIdx.y * K;
	int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;

	int x = threadIdx.x, y = threadIdx.y;

	__shared__ float tile[K][K];

	// coalesced read from global mem, TRANSPOSED write into shared mem:
	tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y)*N];
	__syncthreads();
	// read from shared mem, coalesced write to global mem:
	out[(out_corner_i + x) + (out_corner_j + y)*N] = tile[x][y];
}

void 
transpose_CPU(float in[], float out[])
{
	for(int j=0; j < N; j++)
    	for(int i=0; i < N; i++)
      		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched on a single thread
__global__ void 
transpose_serial(float in[], float out[])
{
	for(int j=0; j < N; j++)
		for(int i=0; i < N; i++)
			out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per row of output matrix
__global__ void 
transpose_parallel_per_row(float in[], float out[])
{
	int i = threadIdx.x;

	for(int j=0; j < N; j++)
		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per element, in KxK threadblocks
// thread (x,y) in grid writes element (i,j) of output matrix 
__global__ void 
transpose_parallel_per_element(float in[], float out[])
{
	int i = blockIdx.x * K + threadIdx.x;
	int j = blockIdx.y * K + threadIdx.y;

	out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}


int main(int argc, char **argv)
{
	int numbytes = N * N * sizeof(float);

	float *in = (float *) malloc(numbytes);
	float *out = (float *) malloc(numbytes);
	float *gold = (float *) malloc(numbytes);

	fill_matrix(in, N);
	transpose_CPU(in, gold);

	float *d_in, *d_out;

	hipMalloc(&d_in, numbytes);
	hipMalloc(&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

	GpuTimer timer;

/*  
 * Now time each kernel and verify that it produces the correct result.
 *
 * To be really careful about benchmarking purposes, we should run every kernel once
 * to "warm" the system and avoid any compilation or code-caching effects, then run 
 * every kernel 10 or 100 times and average the timings to smooth out any variance. 
 * But this makes for messy code and our goal is teaching, not detailed benchmarking.
 */

	dim3 blocks(N/K,N/K);	//TODO, you need to set the proper blocks per grid
	dim3 threads(K,K);	//TODO, you need to set the proper threads per block

	timer.Start();
	transpose_parallel_per_element_tiled<<<blocks,threads>>>(d_in, d_out);
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled %dx%d: %g ms.\nVerifying ...%s\n", 
		   K, K, timer.Elapsed(), compare_matrices(out, gold, N) ? "Success" : "Failed");

	hipFree(d_in);
	hipFree(d_out);
}
