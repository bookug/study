/*=============================================================================
# Filename: lock.cu
# Author: bookug 
# Mail: bookug@qq.com
# Last Modified: 2019-07-08 22:40
# Description: The implementation of lock mechanism in CUDA.
A straightfoward application is the frontier queue used in BFS/SSSP.
=============================================================================*/

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<math_functions.h>
#include<time.h>
#include<hip/hip_runtime.h>


// number of blocks
#define nob 10

struct Lock
{
    int *mutex;
    Lock(void){
        int state = 0;
        hipMalloc((void**) &mutex, sizeof(int));
        hipMemcpy(mutex, &state, sizeof(int), hipMemcpyHostToDevice);
    }
    ~Lock(void){
        hipFree(mutex);
    }
    __device__ void lock(uint compare)
    {
        while(atomicCAS(mutex, compare, 0xFFFFFFFF) != compare);    //0xFFFFFFFF is just a very large number. The point is no block index can be this big (currently).
    }
    __device__ void unlock(uint val)
    {
        atomicExch(mutex, val+1);
    }
};

__global__ void 
theKernel(Lock myLock)
{
    int index = blockIdx.x; //using only one thread per block
    // execute some parallel code
    // critical section of code (thread with index=0 needs to start, followed by index=1, etc.)
    myLock.lock(index);
    printf("Thread with index=%i inside critical section now...\n", index);
    __threadfence_system();   // For the printf. I'm not sure __threadfence_system() can guarantee the order for calls to printf().
    myLock.unlock(index);
}

int 
main(void)
{
    Lock myLock;
    theKernel<<<nob, 1>>>(myLock);
    return 0;
}

