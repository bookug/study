#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"
#include <stdio.h>

/*#define DEBUG_OPEN 1*/

__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals, int span)
{
  //TODO fill in this kernel to calculate the histogram
  //as quickly as possible

  //Although we provide only one kernel skeleton,
  //feel free to use more if it will help you
  //write faster code

	extern __shared__ unsigned tmph[];
	int tid = threadIdx.x;
	if(tid < span)
	{
		tmph[tid] = 0;
	}
	/*printf("tid: %d\n", tid);*/
	__syncthreads();

	int coarse_key = blockIdx.y;
	/*if(coarse_key > 1)*/
	/*printf("coarse key: %d\n", coarse_key);*/
	int myId = tid + blockDim.x * blockIdx.x;
	/*printf("main key: %d\n", myId);*/
	if(myId >= numVals)
	{
		return;
	}
	/*printf("threadIdx.x: %d  threadIdx.y: %d blockIdx.x: %d blockIdx.y: %d blockDim.x: %d blockDim.y: %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y);*/
	unsigned key = vals[myId];
	unsigned key2 = key / span;
	/*printf("main coarse key: %d\n", key2);*/
	/*printf("key2: %u coarse_key: %u\n", key2, coarse_key);*/

	//NOTICE: if not equal, can not return directly here, otherwise the if(tid==0) below may not work if the 0-th thread already returns
	if(key2 == coarse_key)
	{
		atomicAdd(&tmph[key%span], 1);
		//this block finishes the computation
		__syncthreads();
	}
	/*printf("check: %u %u\n", tmph[0], tmph[1]);*/
	/*if(key == 1 && coarse_key == 0)*/
	/*{*/
		/*printf("check 1 bin: %u\n", tmph[1]);*/
	/*}*/
	
	//write this block's result to global memory
	if(tid == 0)
	{
		/*printf("to write coarse_key %d\n", coarse_key);*/
		/*if(coarse_key == 0)*/
			/*printf("check %u bin: %u\n", coarse_key*span, tmph[0]);*/
		for(int i = 0; i < span; ++i)
		{
/*#ifdef DEBUG_OPEN*/
			/*if(coarse_key==0)*/
			/*{*/
				/*printf("check 0 bin %d : %u\n", i, tmph[i]);*/
			/*}*/
/*#endif*/
			atomicAdd(&histo[coarse_key*span+i], tmph[i]);
		}
		/*if(coarse_key == 0)*/
			/*printf("check 0 bin: %u\n", histo[0]);*/
	}
}

void myHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
	//the num of eles is 10240000, the num of bins is 1024, the range of value is 0~999, normal distribution
	/*printf("numBins: %u numElems: %u\n", numBins, numElems);*/
	/*unsigned* h_vals = (unsigned*)malloc(sizeof(unsigned) * numElems);*/
	/*checkCudaErrors(hipMemcpy(h_vals, d_vals, sizeof(unsigned) * numElems, hipMemcpyDeviceToHost));*/
	/*unsigned* h_histo = (unsigned*)malloc(sizeof(unsigned) * numBins);*/
	/*memset(h_histo, 0, sizeof(unsigned) * numBins);*/
	/*FILE* fp = fopen("data.txt", "w+");*/
	/*if(fp == NULL)*/
	/*{*/
		/*printf("error to open file!\n");*/
	/*}*/
	/*for(unsigned i = 0; i < numElems; ++i)*/
	/*{*/
		/*[>printf("%u ", h_vals[i]);<]*/
		/*[>fprintf(fp, "%u\n", h_vals[i]);<]*/
		/*h_histo[h_vals[i]]++;*/
	/*}*/
	/*for(unsigned i = 0; i < numBins; ++i)*/
	/*{*/
		/*fprintf(fp, "%u\n", h_histo[i]);*/
	/*}*/
	/*fclose(fp);*/
	/*free(h_vals);*/
	
	//TODO: use coarse bins for optimization
	//NOTICE: the key point is to keep all SMs busy
	//BETTER: consider not divide evenly, utilize the normally distribution
	//TODO+DEBUG: when numCoarse is 1 or 2, the program is right, but not more efficient
	//However, when numCoarse>=4, the answer is not right!

	//the num of coarse bins
	int numCoarse = 32;
	/*int numCoarse = 16;*/
	//TODO; adjust the num of coarse bins, if 1, then store all bins in each block
	int span = numBins / numCoarse;
/*#ifdef DEBUG_OPEN*/
	/*printf("span: %u\n", span);*/
/*#endif*/
	int size = 1024;
	const dim3 threads(size, 1, 1);
	const dim3 blocks((numElems+size-1)/size, numCoarse, 1);
	checkCudaErrors(hipMemset(d_histo, 0, sizeof(unsigned) * numBins));
	//the size in <<<>>> shoukd be number of bytes
	yourHisto<<<blocks, threads, sizeof(unsigned) * span>>>(d_vals, d_histo, numElems, span);

	/*checkCudaErrors(hipMemcpy(h_histo, d_histo, sizeof(unsigned) * numBins, hipMemcpyDeviceToHost));*/
	/*for(int i = 0; i < numBins; ++i)*/
	/*{*/
		/*printf("%u\n", h_histo[i]);*/
	/*}*/
	/*free(h_histo);*/
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  //TODO Launch the yourHisto kernel

  //if you want to use/launch more than one kernel,
  //feel free

	//NOTICE: thsi si a simple check program
/*#ifdef DEBUG_OPEN*/
	/*unsigned numBins2 = 4, numElems2 = 8;*/
	/*unsigned h_vals2[] = {3, 1, 0, 2, 0, 1, 2, 3};*/
	/*unsigned *d_vals2, *d_histo2;*/
	/*checkCudaErrors(hipMalloc(&d_vals2, sizeof(unsigned) * numElems2));*/
	/*checkCudaErrors(hipMalloc(&d_histo2, sizeof(unsigned) * numBins2));*/
	/*checkCudaErrors(hipMemcpy(d_vals2, h_vals2, sizeof(unsigned) * numElems2, hipMemcpyHostToDevice));*/
	/*myHistogram(d_vals2, d_histo2, numBins2, numElems2);*/
	/*unsigned h_histo2[4];*/
	/*checkCudaErrors(hipMemcpy(h_histo2, d_histo2, sizeof(unsigned) * numBins2, hipMemcpyDeviceToHost));*/
	/*for(int i = 0; i < 4; ++i)*/
	/*{*/
		/*printf("%u\n", h_histo2[i]);*/
	/*}*/
	/*checkCudaErrors(hipFree(d_vals2));*/
	/*checkCudaErrors(hipFree(d_histo2));*/
/*#endif*/

	myHistogram(d_vals, d_histo, numBins, numElems);

	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());
}
