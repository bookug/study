#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"
#include <stdio.h>

__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals, int span)
{
  //TODO fill in this kernel to calculate the histogram
  //as quickly as possible

  //Although we provide only one kernel skeleton,
  //feel free to use more if it will help you
  //write faster code

	extern __shared__ unsigned tmph[];
	int tid = threadIdx.x;
	if(tid < span)
	{
		tmph[tid] = 0;
	}
	/*printf("tid: %d\n", tid);*/
	__syncthreads();

	int coarse_key = blockIdx.y;
	/*printf("coarse key: %d\n", coarse_key);*/
	int myId = tid + blockDim.x * blockIdx.x;
	/*printf("main key: %d\n", myId);*/
	if(myId >= numVals)
	{
		return;
	}
	unsigned key = vals[myId];
	unsigned key2 = key / span;
	/*printf("main coarse key: %d\n", key2);*/
	if(key2 != coarse_key)
	{
		return;
	}
	atomicAdd(&tmph[key%span], 1);
	//this block finishes the computation
	__syncthreads();
	//write this block's result to global memory
	if(tid == 0)
	{
		for(int i = 0; i < span; ++i)
		{
			atomicAdd(&histo[coarse_key*span+i], tmph[i]);
		}
	}
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  //TODO Launch the yourHisto kernel

  //if you want to use/launch more than one kernel,
  //feel free

	//the num of eles is 10240000, the num of bins is 1024, the range of value is 0~999, normal distribution
	/*printf("numBins: %u numElems: %u\n", numBins, numElems);*/
	/*unsigned* h_vals = (unsigned*)malloc(sizeof(unsigned) * numElems);*/
	/*checkCudaErrors(hipMemcpy(h_vals, d_vals, sizeof(unsigned) * numElems, hipMemcpyDeviceToHost));*/
	unsigned* h_histo = (unsigned*)malloc(sizeof(unsigned) * numBins);
	memset(h_histo, 0, sizeof(unsigned) * numBins);
	/*FILE* fp = fopen("data.txt", "w+");*/
	/*if(fp == NULL)*/
	/*{*/
		/*printf("error to open file!\n");*/
	/*}*/
	/*for(unsigned i = 0; i < numElems; ++i)*/
	/*{*/
		/*[>printf("%u ", h_vals[i]);<]*/
		/*[>fprintf(fp, "%u\n", h_vals[i]);<]*/
		/*h_histo[h_vals[i]]++;*/
	/*}*/
	/*for(unsigned i = 0; i < numBins; ++i)*/
	/*{*/
		/*fprintf(fp, "%u\n", h_histo[i]);*/
	/*}*/
	/*fclose(fp);*/
	/*free(h_vals);*/
	/*free(h_histo);*/
	
	//the num of coarse bins
	int numCoarse = 16;
	//TODO; adjust the num of coarse bins, if 1, then store all bins in each block
	int span = numBins / numCoarse;
	/*printf("span: %u\n", span);*/
	int size = 1024;
	const dim3 threads(size, 1, 1);
	const dim3 blocks((numElems+size-1)/size, numCoarse, 1);
	checkCudaErrors(hipMemset(d_histo, 0, sizeof(unsigned) * numBins));
	yourHisto<<<blocks, threads, span>>>(d_vals, d_histo, numElems, span);
	//TODO: use coarse bins for optimization
	//NOTICE: the key point is to keep all SMs busy

	checkCudaErrors(hipMemcpy(h_histo, d_histo, sizeof(unsigned) * numBins, hipMemcpyDeviceToHost));
	for(int i = 0; i < numBins; ++i)
	{
		printf("%u\n", h_histo[i]);
	}

	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());
}
