#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

__global__ void
histogram_kernel(const unsigned int* const d_inputVals, const size_t numElems, unsigned bitBase, unsigned* const d_count, unsigned int* const d_outputVals, unsigned int* const d_outputPos)
{
	__shared__ unsigned s_cnt;
	s_cnt = 0;
	__syncthreads();

	int tid = threadIdx.x;
	int myId = tid + blockDim.x * blockIdx.x;
	if(myId >= numElems)
	{
		return;
	}
	//NOTICE: the result of & should not be compared with 1!
	/*unsigned ret = d_inputVals[myId] & bitBase;*/
	/*if(ret != 0)*/
	//NOTICE: we must use () for bitwise operations, otherwise error will come(priority of != is higher than &)
	/*if(d_inputVals[myId] & bitBase != 0)*/
	if((d_inputVals[myId] & bitBase) != 0)
	{
		d_outputPos[myId] = 1;
		d_outputVals[myId] = 0;
	}
	else
	{
		d_outputPos[myId] = 0;
		d_outputVals[myId] = 1;
		atomicAdd(&s_cnt, 1);
	}
	__syncthreads();

	if(tid == 0)  //the end
	{
		atomicAdd(&d_count[0], s_cnt);
	}
}

void histogram(const unsigned int* const d_inputVals, const size_t numElems, const unsigned bitBase, unsigned* const d_count, unsigned int* const d_outputVals, unsigned int* const d_outputPos)
{
	/*printf("here is histogram!\n");*/
	int size = 1024;
	const dim3 threads(size, 1, 1);
	const dim3 blocks((numElems+size-1)/size, 1, 1);
	//Only use the 0-th position
	checkCudaErrors(hipMemset(d_count, 0, sizeof(unsigned)));
	histogram_kernel<<<blocks, threads>>>(d_inputVals, numElems, bitBase, d_count, d_outputVals, d_outputPos);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	/*unsigned ret;*/
	/*checkCudaErrors(hipMemcpy(&ret, &d_count[1], sizeof(unsigned), hipMemcpyDeviceToHost));*/
	/*return ret;*/
}

//NOTICE: the minium unit of executing threads is WARP(a group of 16 or 32 threads)
//If the if-else or loops in a kernel --> too many threads do nop operations in a warp! then the performance will be bad
//http://www.myexception.cn/cuda/1931284.html

//METHOD: to sync between Blocks
//1. construct a lock variable in global memory, and use barrier for each thread
//http://bbs.csdn.net/topics/330255319
//2. divide into different kernels, and do each kernel a time
//3. http://bbs.csdn.net/topics/390828048  http://blog.csdn.net/groundhappy/article/details/54173387

__global__ void
presum_kernel(unsigned* const d_in, unsigned* const d_out, const size_t numElems, const unsigned span)
{
	int tid = threadIdx.x;
	int myId = tid + blockDim.x * blockIdx.x;
	if(myId >= numElems)
	{
		return;
	}

	/*for(unsigned span = 1; span < numElems; span <<= 1)*/
	if(myId < span)
	{
		d_out[myId] = d_in[myId];
	}
	else
	{
		d_out[myId] = d_in[myId] + d_in[myId-span];
	}
}

__global__ void
combine_kernel(const unsigned int* d_inputVals, unsigned int* const d_outputVals, unsigned int* const d_outputPos, unsigned int* const d_buffer, const size_t numElems, const unsigned bitBase, const unsigned* const d_count)
{
	__shared__ unsigned d_base;
	d_base = d_count[0];
	__syncthreads();

	int tid = threadIdx.x;
	int myId = tid + blockDim.x * blockIdx.x;
	if(myId >= numElems)
	{
		return;
	}
	//set d_out by exclusive presum
	unsigned ret = d_inputVals[myId] & bitBase;
	if(myId == 0)
	{
		if(ret != 0)
		{
			d_buffer[0] = d_base;
		}
		else
		{
			d_buffer[0] = 0;
		}
		return;
	}
	if(ret != 0)
	{
		d_buffer[myId] = d_outputPos[myId-1] + d_base;
	}
	else
	{
		d_buffer[myId] = d_outputVals[myId-1];
	}
}

void presum(const unsigned int* d_inputVals, unsigned int* const d_outputVals, unsigned int* const d_outputPos, const size_t numElems, unsigned* const d_buffer, const unsigned bitBase, const unsigned* const d_count)
{
	/*printf("here is histogram!\n");*/
	int size = 1024;
	const dim3 threads(size, 1, 1);
	const dim3 blocks((numElems+size-1)/size, 1, 1);
	//NOTICE: the threads can not be played in a single block, so the __syncthreads not works for all threads
	//As a result, we can not do this prefix-sum in-place, which means we need another array for help
	unsigned* d_ptr[2]; d_ptr[0] = d_outputVals, d_ptr[1] = d_outputPos;

	for(unsigned idx = 0; idx < 2; ++idx)
	{
		unsigned *d_out = d_ptr[idx], *d_in = d_buffer, *d_tmp;
		//BETTER: when there are only a few additions for a large span, no need for so many threads
		for(unsigned span = 1; span < numElems; span <<= 1)
		{
			d_tmp = d_in; d_in = d_out; d_out = d_tmp;
			presum_kernel<<<blocks, threads>>>(d_in, d_out, numElems, span);
			hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
		}
		if(d_out != d_ptr[idx])
		{
			checkCudaErrors(hipMemcpy(d_ptr[idx], d_out, sizeof(unsigned) * numElems, hipMemcpyDeviceToDevice));
		}
	}

	/*unsigned xxx[7];*/
		/*checkCudaErrors(hipMemcpy(xxx, d_outputVals, sizeof(unsigned) * 7, hipMemcpyDeviceToHost));*/
		/*printf("check index before combine: %u %u %u %u %u %u %u\n", xxx[0], xxx[1], xxx[2], xxx[3], xxx[4], xxx[5], xxx[6]);*/
		/*checkCudaErrors(hipMemcpy(xxx, d_outputPos, sizeof(unsigned) * 7, hipMemcpyDeviceToHost));*/
		/*printf("check index before combine: %u %u %u %u %u %u %u\n", xxx[0], xxx[1], xxx[2], xxx[3], xxx[4], xxx[5], xxx[6]);*/

	//combine two position arrays, and place in d_buffer
	combine_kernel<<<blocks, threads>>>(d_inputVals, d_outputVals, d_outputPos, d_buffer, numElems, bitBase, d_count);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

__global__ void
scatter_kernel(const unsigned int* const d_inputVals, const unsigned int* d_inputPos, unsigned int* const d_outputVals, unsigned int* const d_outputPos, const size_t numElems, unsigned* const d_buffer)
{
	int tid = threadIdx.x;
	int myId = tid + blockDim.x * blockIdx.x;
	if(myId >= numElems)
	{
		return;
	}
	int key = d_buffer[myId];
	d_outputVals[key] = d_inputVals[myId];
	d_outputPos[key] = d_inputPos[myId];
}

void scatter(unsigned int* const d_inputVals, unsigned int* const d_inputPos, unsigned int* const d_outputVals, unsigned int* const d_outputPos, const size_t numElems, unsigned* const d_buffer)
{
	/*printf("here is histogram!\n");*/
	int size = 1024;
	const dim3 threads(size, 1, 1);
	const dim3 blocks((numElems+size-1)/size, 1, 1);
	scatter_kernel<<<blocks, threads>>>(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems, d_buffer);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

__global__ void
reduce_kernel(unsigned* const d_out, unsigned* const d_in, const size_t curlen)
{
	int tid = threadIdx.x;
	int myId = tid + blockDim.x * blockIdx.x;
	if(myId >= curlen)
	{
		return;
	}
	//NOTICE: it is ok to be negative
	//PERORMANCE: use IO in GPU is very slow, need to transfer to cpu memory and output
	/*if(d_in[myId] < 0)*/
	/*{*/
		/*printf("error: %f!\n", d_in[myId]);*/
	/*}*/

	for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if(tid < s && myId + s < curlen)
		{
			unsigned tmp1 = d_in[myId], tmp2 = d_in[myId+s];
			d_in[myId] = max(tmp1, tmp2);
		}
		__syncthreads();
	}

	if(tid == 0)
	{
		d_out[blockIdx.x] = d_in[myId];
	}
	/*if(myId == 0)*/
	/*{*/
		/*float mine = d_in[0], maxe = d_in[0];*/
		/*for(unsigned i = 1; i < curlen; ++i)*/
		/*{*/
			/*mine = min(d_in[i], mine);*/
			/*maxe = max(d_in[i], maxe);*/
		/*}*/
		/*printf("check min %f max %f\n", mine, maxe);*/
	/*}*/
}

//find the maxium number
unsigned reduce(const unsigned* const d_inputVals, const size_t numlen)
{
	/*printf("here is reduce!\n");*/
	int limit = log(numlen)/log(2) + 1;
	int size = 1024;
	const dim3 threads(size, 1, 1);
	int curlen = numlen;
	unsigned*  d_in;
	unsigned* d_mid;
	checkCudaErrors(hipMalloc((void**)&d_in, sizeof(unsigned) * numlen));
	checkCudaErrors(hipMemcpy(d_in, d_inputVals, sizeof(unsigned) * numlen, hipMemcpyDeviceToDevice));

	/*printf("begin while loop!\n");*/
	while(limit--)
	{
		/*printf("loop loop\n");*/
		int num_blocks = curlen / size;
		if(curlen % size != 0)
		{
			num_blocks ++;
		}
		checkCudaErrors(hipMalloc((void**)&d_mid, sizeof(unsigned) * num_blocks));

		const dim3 blocks(num_blocks, 1, 1);
		reduce_kernel<<<blocks, threads>>>(d_mid, d_in, curlen);
		hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

		/*printf("before cuda free!\n");*/
		/*printf("curlen: %d num_blocks: %d\n", curlen, num_blocks);*/
		checkCudaErrors(hipFree(d_in));
		/*printf("after cuda free!\n");*/
		curlen = num_blocks;
		if(curlen == 1)
		{
			break;
		}
		d_in = d_mid;
	}

	unsigned ret;
	//the result is in d_mid
	checkCudaErrors(hipMemcpy(&ret, &d_mid[0], sizeof(unsigned), hipMemcpyDeviceToHost));
	//NOTICE: below is error, device memory can not be copied directly in the host code
	//while in kernel code, device memory can be used directly
	/**d_border = d_mid[0];*/
	checkCudaErrors(hipFree(d_mid));
	return ret;
}

void radix_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
	//NOTICE: there can be duplicates in an array to be sorted
	/*printf("numElems: %lu\n", numElems);  //220480*/
	//inputVals is the key and inputPos is the value
	int size = 1024;
	const dim3 threads(size, 1, 1);
	const dim3 blocks((numElems+size-1)/size, 1, 1);
	//NOTICE: Radix Sort can use many kinds of Hash Function, we use 2 as the base here
	unsigned hashBase = 2;
	unsigned bitBase = 1;  //use << bitwise operation to change in each loop
	unsigned *d_count;
	checkCudaErrors(hipMalloc((void**)&d_count, sizeof(unsigned) * hashBase));
	unsigned *d_buffer;
	checkCudaErrors(hipMalloc((void**)&d_buffer, sizeof(unsigned) * numElems));
	unsigned addBase = 0;
	unsigned *d_in1 = d_inputVals, *d_in2 = d_inputPos, *d_out1 = d_outputVals, *d_out2 = d_outputPos, *d_tmp;
	unsigned step = 0;
	unsigned limit = 32;
	/*unsigned limit = reduce(d_inputVals, numElems);*/
	/*limit = log(limit)/log(2) + 1;*/
	/*if(limit > 32) limit = 32;*/
	/*printf("limit: %u\n", limit);*/
	
	hipEvent_t start ,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	while(step < limit)  //32 is the limit of bits of unsigned type
	{
		hipEventRecord(start, 0);
		/*printf("this is the %d-th step\n", step);*/
		/*unsigned xxx[7]; checkCudaErrors(hipMemcpy(xxx, d_in1, sizeof(unsigned) * 7, hipMemcpyDeviceToHost));*/
		/*printf("check: %u %u %u %u %u %u %u\n", xxx[0], xxx[1], xxx[2], xxx[3], xxx[4], xxx[5], xxx[6]);*/
		/*printf("check: %u %u %u\n", xxx[0] & bitBase, xxx[1] & bitBase, xxx[2] & bitBase);*/

		//base1 is for digit 0, while base2 is for digit 1
		//NOTICE: we needn't count the num of 1, just count the num of 0 is ok
		histogram(d_in1, numElems, bitBase, d_count, d_out1, d_out2);
		/*printf("histogram finished\n");*/
		step++;

		/*checkCudaErrors(hipMemcpy(&addBase, &d_count[0], sizeof(unsigned), hipMemcpyDeviceToHost));*/
		/*printf("addBase: %u bitBase: %u\n", addBase, bitBase);*/
		/*checkCudaErrors(hipMemcpy(xxx, d_out1, sizeof(unsigned) * 7, hipMemcpyDeviceToHost));*/
		/*printf("check out1: %u %u %u %u %u %u %u\n", xxx[0], xxx[1], xxx[2], xxx[3], xxx[4], xxx[5], xxx[6]);*/
		/*checkCudaErrors(hipMemcpy(xxx, d_out2, sizeof(unsigned) * 7, hipMemcpyDeviceToHost));*/
		/*printf("check out2: %u %u %u %u %u %u %u\n", xxx[0], xxx[1], xxx[2], xxx[3], xxx[4], xxx[5], xxx[6]);*/
		/*getchar();*/

		if(addBase == numElems)
		{
			/*printf("needless loop\n");*/
			bitBase <<= 1;
			//NOTICE: we can not break directly here, because it is ok for all numbers to be 0 at the i-th digit
			continue;
		}

		//use d_count as addBase in shared memory
		presum(d_in1, d_out1, d_out2, numElems, d_buffer, bitBase, d_count);
		/*printf("presum finished\n");*/

		/*checkCudaErrors(hipMemcpy(xxx, d_buffer, sizeof(unsigned) * 3, hipMemcpyDeviceToHost));*/
		/*printf("check index: %u %u %u\n", xxx[0], xxx[1], xxx[2]);*/

		//scatter: both key and value to output according to mapping in d_buffer
		scatter(d_in1, d_in2, d_out1, d_out2, numElems, d_buffer);
		/*printf("scatter finished\n");*/
		bitBase <<= 1;
		d_tmp = d_in1; d_in1 = d_out1; d_out1 = d_tmp;
		d_tmp = d_in2; d_in2 = d_out2; d_out2 = d_tmp;

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float elapsedTime;
		hipEventElapsedTime(&elapsedTime, start, stop);
		/*elapsedTime /= 100.0f;*/
		printf("average time elapsed: %fms\n", elapsedTime);
	}

	if(d_in1 != d_outputVals)
	{
		//NOTICE: we believe device-to-device copy is very fast, so do not use multithreading to copy by myself
		checkCudaErrors(hipMemcpy(d_outputVals, d_in1, sizeof(unsigned) * numElems, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(d_outputPos, d_in2, sizeof(unsigned) * numElems, hipMemcpyDeviceToDevice));
	}

	checkCudaErrors(hipFree(d_count));
	checkCudaErrors(hipFree(d_buffer));
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
  //TODO
  //PUT YOUR SORT HERE

	/*unsigned arr[] = {6, 0, 3, 1, 4, 2, 5};*/
	/*unsigned *d_in1, *d_in2, *d_out1, *d_out2, num = 7;*/
	/*checkCudaErrors(hipMalloc((void**)&d_in1, sizeof(unsigned) * num));*/
	/*checkCudaErrors(hipMalloc((void**)&d_in2, sizeof(unsigned) * num));*/
	/*checkCudaErrors(hipMalloc((void**)&d_out1, sizeof(unsigned) * num));*/
	/*checkCudaErrors(hipMalloc((void**)&d_out2, sizeof(unsigned) * num));*/
	/*checkCudaErrors(hipMemcpy(d_in1, arr, sizeof(unsigned) * num, hipMemcpyHostToDevice));*/
	/*checkCudaErrors(hipMemcpy(d_in2, arr, sizeof(unsigned) * num, hipMemcpyHostToDevice));*/
	/*radix_sort(d_in1, d_in2, d_out1, d_out2, num);*/
	/*checkCudaErrors(hipFree(d_in1));*/
	/*checkCudaErrors(hipFree(d_in2));*/
	/*checkCudaErrors(hipFree(d_out1));*/
	/*checkCudaErrors(hipFree(d_out2));*/

	hipSetDevice(0);
	radix_sort(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems);
}

