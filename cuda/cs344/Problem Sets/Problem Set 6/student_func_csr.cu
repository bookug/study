#include "hip/hip_runtime.h"
//Udacity HW 6
//Poisson Blending

/* Background
   ==========

   The goal for this assignment is to take one image (the source) and
   paste it into another image (the destination) attempting to match the
   two images so that the pasting is non-obvious. This is
   known as a "seamless clone".

   The basic ideas are as follows:

   1) Figure out the interior and border of the source image
   2) Use the values of the border pixels in the destination image 
      as boundary conditions for solving a Poisson equation that tells
      us how to blend the images.
   
      No pixels from the destination except pixels on the border
      are used to compute the match.

   Solving the Poisson Equation
   ============================

   There are multiple ways to solve this equation - we choose an iterative
   method - specifically the Jacobi method. Iterative methods start with
   a guess of the solution and then iterate to try and improve the guess
   until it stops changing.  If the problem was well-suited for the method
   then it will stop and where it stops will be the solution.

   The Jacobi method is the simplest iterative method and converges slowly - 
   that is we need a lot of iterations to get to the answer, but it is the
   easiest method to write.

   Jacobi Iterations
   =================

   Our initial guess is going to be the source image itself.  This is a pretty
   good guess for what the blended image will look like and it means that
   we won't have to do as many iterations compared to if we had started far
   from the final solution.

   ImageGuess_prev (Floating point)
   ImageGuess_next (Floating point)

   DestinationImg
   SourceImg

   Follow these steps to implement one iteration:

   1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
      Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

      Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

   2) Calculate the new pixel value:
      float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
      ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]


    In this assignment we will do 800 iterations.
   */



#include "utils.h"
#include <thrust/host_vector.h>

//ANALYSIS: the source image is really sparse, so we must compress it to be placed in a block's shared memory(<= 48KB)
//However, this sparse graph is very special, and CSR is not fit here due to its complexity
//(and maybe not so efficient due to its separated visits to memory)
//In the source graph, only the small part of center is useful, so we can use skew and a small matrix to represent
//(which is more simple, and can help save more memory)
//In fact, CSR is just like place an adjacent list in an array
typedef struct ImageMask
{
	//BETTER: use bitwise
	//false: interior    true: border
	bool* value;
	//BETTER: use unsigned short if not exceeding 65534
	unsigned* index;
	//-1 if all zeros in this row
	int* rowptr;
	unsigned num;  //edges' number
}Mask;
unsigned countMask(Mask* mask, const size_t numRows, const size_t numCols)
{
	return sizeof(bool) * mask->num + sizeof(unsigned) * mask->num + sizeof(int) * numRows;
}
//we assume that x and y is valid
int checkMask(Mask* mask, unsigned x, unsigned y, const size_t numRows, const size_t numCols)
{
	//0:white  1:interior  2:border
	int row = mask->rowptr[y];
	if(row < 0)
	{
		return 0;
	}
	int begin = row, end = y+1;
	while(end < numRows && mask->rowptr[end] < 0)
	{
		end++;
	}
	if(end == numRows)
	{
		end = mask->num;
	}
	else
	{
		end = mask->rowptr[end];
	}
	//search in [begin, end) for x, if not found then return 0
	//BETTER: binary search
	for(int i = begin; i < end; ++i)
	{
		if(mask->index[i] == x)
		{
			if(mask->value[i])
			{
				return 2;
			}
			else
			{
				return 1;
			}
		}
	}
	return 0;
}
void freeMask(Mask* d_mask)
{
	checkCudaErrors(hipFree(d_mask->value));
	checkCudaErrors(hipFree(d_mask->index));
	checkCudaErrors(hipFree(d_mask->rowptr));
	checkCudaErrors(hipFree(d_mask));
	d_mask = NULL;
}

void computeMask(const uchar4* const d_sourceImg, Mask* const d_mask, const size_t numRows, const size_t numCols)
{
	//TODO: alloc and build the mask
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	//TODO: compute the border(value is true) and interior(value is false)
}

void Jacobi(float* d_color, const size_t numRows, const size_t numCols)
{
	//TODO
	float *d_bufin, *d_bufout;
	const size_t numPixels = numRows * numCols;
	checkCudaErrors(hipMalloc(&d_bufin, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_bufout, sizeof(float) * numPixels));
	checkCudaErrors(hipMemcpy(d_bufin, d_color, sizeof(float) * numPixels, hipMemcpyDeviceToDevice));

	//Jacobi  800 iterations
	//TODO: free buffers and store result in d_color
}

void splitChannel(const uchar4* const d_sourceImg, float* d_red, float* d_green, float* d_blue, const size_t numRows, const size_t numCols)
{
	const size_t numPixels = numRows * numCols;
	checkCudaErrors(hipMalloc(&d_red, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_green, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_blue, sizeof(float) * numPixels));
	//TODO: split kernel, uchar to float (not double!)

	Jacobi(d_red, numRows, numCols);
	Jacobi(d_green, numRows, numCols);
	Jacobi(d_blue, numRows, numCols);
}

void getResult(const uchar4* const d_destImg, float* d_red, float* d_green, float* d_blue, uchar4* d_blendedImg, const size_t numRows, const size_t numCols)
{
	//TODO
}

void your_blend(const uchar4* const h_sourceImg,  //IN
                const size_t numRowsSource, const size_t numColsSource,
                const uchar4* const h_destImg, //IN
                uchar4* const h_blendedImg) //OUT
{

  /* To Recap here are the steps you need to implement
  
     1) Compute a mask of the pixels from the source image to be copied
        The pixels that shouldn't be copied are completely white, they
        have R=255, G=255, B=255.  Any other pixels SHOULD be copied.

     2) Compute the interior and border regions of the mask.  An interior
        pixel has all 4 neighbors also inside the mask.  A border pixel is
        in the mask itself, but has at least one neighbor that isn't.

     3) Separate out the incoming image into three separate channels

     4) Create two float(!) buffers for each color channel that will
        act as our guesses.  Initialize them to the respective color
        channel of the source image since that will act as our intial guess.

     5) For each color channel perform the Jacobi iteration described 
        above 800 times.

     6) Create the output image by replacing all the interior pixels
        in the destination image with the result of the Jacobi iterations.
        Just cast the floating point values to unsigned chars since we have
        already made sure to clamp them to the correct range.

      Since this is final assignment we provide little boilerplate code to
      help you.  Notice that all the input/output pointers are HOST pointers.

      You will have to allocate all of your own GPU memory and perform your own
      memcopies to get data in and out of the GPU memory.

      Remember to wrap all of your calls with checkCudaErrors() to catch any
      thing that might go wrong.  After each kernel call do:

      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

      to catch any errors that happened while executing the kernel.
  */

	//NOTICE: the source image's size is the same as the target image
	printf("numRowsSource: %lu   numColsSource: %lu\n", numRowsSource, numColsSource); //500x333
	uchar4 *d_sourceImg, *d_destImg, *d_blendedImg;
	size_t numPixels = numRowsSource * numColsSource;
	checkCudaErrors(hipMalloc(&d_sourceImg, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMalloc(&d_destImg, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMalloc(&d_blendedImg, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMemcpy(d_sourceImg, h_sourceImg, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_destImg, h_destImg, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));

	//NOTICE: we believe that the useful part in source image is very small, and 
	//the border line is smaller than the interior
	Mask* d_mask;
	checkCudaErrors(hipMalloc(&d_mask, sizeof(Mask)));
	//find the border of the source image, compressed in CSR and can be placed in a block's shared memory
	computeMask(d_sourceImg, d_mask, numRowsSource, numColsSource);

	//separate into 3 channels, and two buffers
	float *d_red = NULL, *d_green = NULL, *d_blue = NULL;
	splitChannel(d_sourceImg, d_red, d_green, d_blue, numRowsSource, numColsSource);

    //create the output image by replacing all the interior pixels
	getResult(d_destImg, d_red, d_green, d_blue, d_blendedImg, numRowsSource, numColsSource);

	freeMask(d_mask);
	checkCudaErrors(hipMemcpy(h_blendedImg, d_blendedImg, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_red));
	checkCudaErrors(hipFree(d_green));
	checkCudaErrors(hipFree(d_blue));
	checkCudaErrors(hipFree(d_sourceImg));
	checkCudaErrors(hipFree(d_destImg));
	checkCudaErrors(hipFree(d_blendedImg));
}
