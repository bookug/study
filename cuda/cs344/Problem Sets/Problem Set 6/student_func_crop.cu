#include "hip/hip_runtime.h"
//Udacity HW 6
//Poisson Blending

/* Background
   ==========

   The goal for this assignment is to take one image (the source) and
   paste it into another image (the destination) attempting to match the
   two images so that the pasting is non-obvious. This is
   known as a "seamless clone".

   The basic ideas are as follows:

   1) Figure out the interior and border of the source image
   2) Use the values of the border pixels in the destination image 
      as boundary conditions for solving a Poisson equation that tells
      us how to blend the images.
   
      No pixels from the destination except pixels on the border
      are used to compute the match.

   Solving the Poisson Equation
   ============================

   There are multiple ways to solve this equation - we choose an iterative
   method - specifically the Jacobi method. Iterative methods start with
   a guess of the solution and then iterate to try and improve the guess
   until it stops changing.  If the problem was well-suited for the method
   then it will stop and where it stops will be the solution.

   The Jacobi method is the simplest iterative method and converges slowly - 
   that is we need a lot of iterations to get to the answer, but it is the
   easiest method to write.

   Jacobi Iterations
   =================

   Our initial guess is going to be the source image itself.  This is a pretty
   good guess for what the blended image will look like and it means that
   we won't have to do as many iterations compared to if we had started far
   from the final solution.

   ImageGuess_prev (Floating point)
   ImageGuess_next (Floating point)

   DestinationImg
   SourceImg

   Follow these steps to implement one iteration:

   1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
      Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

      Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

   2) Calculate the new pixel value:
      float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
      ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]


    In this assignment we will do 800 iterations.
   */



#include "utils.h"
#include <thrust/host_vector.h>

//NOTICE: there are two ways to end the iterations:
//1. set a tiny threshold, and stop when difference between the previous and the current is smaller than it
//2. set a limit of iteration num
#define LIMIT 800

//ANALYSIS: the source image is really sparse, so we must compress it to be placed in a block's shared memory(<= 48KB)
//However, this sparse graph is very special, and CSR is not fit here due to its complexity
//(and maybe not so efficient due to its separated visits to memory)
//In the source graph, only the small part of center is useful, so we can use offset and a small matrix to represent
//(which is more simple, and can help save more memory)
//In fact, CSR is just like place an adjacent list in an array
typedef unsigned char uchar;
typedef struct ImageMask
{
	//BETTER: use bitwise
	//0:white 1:interior 2:border
	uchar* value;
	unsigned start_x, start_y;
	unsigned size_x, size_y;
}Mask;
unsigned countMask(Mask* mask)
{
	return sizeof(bool) * mask->size_x * mask->size_y;
}
//we assume that x and y is valid
int checkMask(Mask* mask, unsigned x, unsigned y)
{
	//0:white  1:interior  2:border
	unsigned end_x = mask->start_x + mask->size_x - 1;
	unsigned end_y = mask->start_y + mask->size_y - 1;
	if(x < mask->start_x || y < mask->start_y || x > end_x || y > end_y)
	{
		return 0;
	}
	unsigned pos = y * mask->size_x + x;
	return mask->value[pos];
}
void freeMask(Mask* d_mask)
{
	checkCudaErrors(hipFree(d_mask->value));
	checkCudaErrors(hipFree(d_mask));
	d_mask = NULL;
}

void computeMask(const uchar4* const d_sourceImg, Mask* const d_mask, const size_t numRows, const size_t numCols)
{
	//TODO: alloc and build the mask
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	//TODO: compute the border(value is true) and interior(value is false)
}

void Jacobi(float* d_color_blended1, float* d_color_blended2, const float* const d_color_pre, const uchar* const d_color_dst, const Mask* const d_mask)
{
	//TODO
	/*float *d_bufin, *d_bufout;*/
	/*const size_t numPixels = numRows * numCols;*/
	/*checkCudaErrors(hipMalloc(&d_bufin, sizeof(float) * numPixels));*/
	/*checkCudaErrors(hipMalloc(&d_bufout, sizeof(float) * numPixels));*/
	/*checkCudaErrors(hipMemcpy(d_bufin, d_color, sizeof(float) * numPixels, hipMemcpyDeviceToDevice));*/

	//Jacobi  800 iterations
	//TODO: final result in d_color_blended1
}

//crop the source graph and only use the core part
void splitChannel(const uchar4* const d_img, uchar*& d_red, uchar*& d_blue, uchar*& d_green, const size_t numRows, const size_t numCols, float* d_red_blended1 = NULL, float* d_blue_blended1 = NULL, float* d_green_blended1 = NULL)
{
	const size_t numPixels = numRows * numCols;
	checkCudaErrors(hipMalloc(&d_red, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_green, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_blue, sizeof(float) * numPixels));
	//TODO: split kernel, uchar to float (not double!)
	//TODO: for source img, copy result to blended1(uchar to float)

  /*for (int i = 0; i < srcSize; ++i) {*/
    /*red_src[i]   = h_sourceImg[i].x;*/
    /*blue_src[i]  = h_sourceImg[i].y;*/
    /*green_src[i] = h_sourceImg[i].z;*/
  /*}*/
}

__global__
void map_kernel(float* d_red, float* d_blue, float* d_green, uchar4* d_blendedImg, const Mask* d_mask)
{
	extern __shared__ bool mask[];
	unsigned start_x = d_mask->start_x, start_y = d_mask->start_y, size_x = d_mask->size_x, size_y = d_mask->size_y;
	unsigned end_x = start_x + size_x - 1, end_y = start_y + size_y - 1;
	int xpos = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int ypos = threadIdx.y + blockIdx.y * blockDim.y + 1;
	if(xpos > size_x || ypos > size_y)
	{
		return; 
	}
	mask[ypos][xpos] = d_mask[ypos*size_x + xpos];
	__syncthreads();
}

void getResult(float* d_red, float* d_blue, float* d_green, uchar4* d_blendedImg, const Mask* const d_mask, const size_t numRows_mask, const size_t numCols_mask)
{
	//if true and interior in mask, then copy from buffer
	//NOTICE: assume the mask is 0~N, so the interior is just possible in 1~N-1
	unsigned shared_size = sizeof(bool) * numRows_mask * numCols_mask;
	int xsize = 64, ysize = 16;
	const dim3 threads(xsize, ysize, 1);
	const dim3 blocks((numCols_mask-2+xsize-1)/xsize, (numRows_mask-2+ysize-1)/ysize, 1);
	//cache a part of the mask in shared memory(including the border), how about the final blocks?
	map_kernel<<<blocks, threads, shared_size>>>(d_red, d_blue, d_green, d_blendedImg, d_mask);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

void your_blend(const uchar4* const h_sourceImg,  //IN
                const size_t numRowsSource, const size_t numColsSource,
                const uchar4* const h_destImg, //IN
                uchar4* const h_blendedImg) //OUT
{

  /* To Recap here are the steps you need to implement
  
     1) Compute a mask of the pixels from the source image to be copied
        The pixels that shouldn't be copied are completely white, they
        have R=255, G=255, B=255.  Any other pixels SHOULD be copied.

     2) Compute the interior and border regions of the mask.  An interior
        pixel has all 4 neighbors also inside the mask.  A border pixel is
        in the mask itself, but has at least one neighbor that isn't.

     3) Separate out the incoming image into three separate channels

     4) Create two float(!) buffers for each color channel that will
        act as our guesses.  Initialize them to the respective color
        channel of the source image since that will act as our intial guess.

     5) For each color channel perform the Jacobi iteration described 
        above 800 times.

     6) Create the output image by replacing all the interior pixels
        in the destination image with the result of the Jacobi iterations.
        Just cast the floating point values to unsigned chars since we have
        already made sure to clamp them to the correct range.

      Since this is final assignment we provide little boilerplate code to
      help you.  Notice that all the input/output pointers are HOST pointers.

      You will have to allocate all of your own GPU memory and perform your own
      memcopies to get data in and out of the GPU memory.

      Remember to wrap all of your calls with checkCudaErrors() to catch any
      thing that might go wrong.  After each kernel call do:

      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

      to catch any errors that happened while executing the kernel.
  */

	//NOTICE: the source image's size is the same as the target image
	printf("numRowsSource: %lu   numColsSource: %lu\n", numRowsSource, numColsSource); //500x333
	uchar4 *d_sourceImg, *d_destImg, *d_blendedImg;
	size_t numPixels = numRowsSource * numColsSource;
	checkCudaErrors(hipMalloc(&d_sourceImg, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMalloc(&d_destImg, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMalloc(&d_blendedImg, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMemcpy(d_sourceImg, h_sourceImg, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_destImg, h_destImg, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));

	//NOTICE: we believe that the useful part in source image is very small, and 
	//the border line is smaller than the interior
	Mask* d_mask;
	checkCudaErrors(hipMalloc(&d_mask, sizeof(Mask)));
	//find the border of the source image, compressed in CSR and can be placed in a block's shared memory
	computeMask(d_sourceImg, d_mask, numRowsSource, numColsSource);

	//TODO: crop the source image by mask, in splitChannel
	size_t numRows_mask = 0, numCols_mask = 0;
	checkCudaErrors(hipMemcpy(&numRows_mask, &(d_mask->size_y), sizeof(unsigned), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&numCols_mask, &(d_mask->size_x), sizeof(unsigned), hipMemcpyDeviceToHost));
	size_t numPixels_mask = numRows_mask * numCols_mask;

	float *d_red_pre, *d_blue_pre, *d_green_pre;
	checkCudaErrors(hipMalloc(&d_red_pre, sizeof(float) * numPixels_mask));
	checkCudaErrors(hipMalloc(&d_blue_pre, sizeof(float) * numPixels_mask));
	checkCudaErrors(hipMalloc(&d_green_pre, sizeof(float) * numPixels_mask));
	checkCudaErrors(hipMemset(d_red_pre, 0, sizeof(float) * numPixels_mask));
	checkCudaErrors(hipMemset(d_blue_pre, 0, sizeof(float) * numPixels_mask));
	checkCudaErrors(hipMemset(d_green_pre, 0, sizeof(float) * numPixels_mask));
	//TODO: pre-compute the sum of source image and neighbors

	float *d_red_blended1, *d_red_blended2, *d_blue_blended1, *d_blue_blended2, *d_green_blended1, *d_green_blended2;
	checkCudaErrors(hipMalloc(&d_red_blended1, sizeof(float) * numPixels_mask));
	checkCudaErrors(hipMalloc(&d_red_blended2, sizeof(float) * numPixels_mask));
	checkCudaErrors(hipMalloc(&d_blue_blended1, sizeof(float) * numPixels_mask));
	checkCudaErrors(hipMalloc(&d_blue_blended2, sizeof(float) * numPixels_mask));
	checkCudaErrors(hipMalloc(&d_green_blended1, sizeof(float) * numPixels_mask));
	checkCudaErrors(hipMalloc(&d_green_blended2, sizeof(float) * numPixels_mask));
	//separate into 3 channels, and two buffers
	uchar *d_red_src = NULL, *d_green_src = NULL, *d_blue_src = NULL;
	uchar *d_red_dst = NULL, *d_green_dst = NULL, *d_blue_dst = NULL;
	splitChannel(d_sourceImg, d_red_src, d_blue_src, d_green_src, numRows_mask, numCols_mask, d_red_blended1, d_blue_blended1, d_green_blended1);
	splitChannel(d_destImg, d_red_dst, d_blue_dst, d_green_dst, numRows_mask, numCols_mask);

	//NOTICE: iterations are very costly, so the main focus is to reduce the work in iterations
	Jacobi(d_red_blended1, d_red_blended2, d_red_pre, d_red_dst, d_mask);
	Jacobi(d_blue_blended1, d_blue_blended2, d_blue_pre, d_blue_dst, d_mask);
	Jacobi(d_green_blended1, d_green_blended2, d_green_pre, d_green_dst, d_mask);

	checkCudaErrors(hipMemcpy(d_blendedImg, d_destImg, sizeof(uchar4) * numPixels, hipMemcpyDeviceToDevice));
    //create the output image by replacing all the interior pixels
	getResult(d_red_blended1, d_blue_blended1, d_green_blended1, d_blendedImg, d_mask, numRows_mask, numCols_mask);

	freeMask(d_mask);
	checkCudaErrors(hipMemcpy(h_blendedImg, d_blendedImg, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_red_src));
	checkCudaErrors(hipFree(d_blue_src));
	checkCudaErrors(hipFree(d_green_src));
	checkCudaErrors(hipFree(d_red_dst));
	checkCudaErrors(hipFree(d_blue_dst));
	checkCudaErrors(hipFree(d_green_dst));
	checkCudaErrors(hipFree(d_red_pre));
	checkCudaErrors(hipFree(d_blue_pre));
	checkCudaErrors(hipFree(d_green_pre));
	checkCudaErrors(hipFree(d_red_blended1));
	checkCudaErrors(hipFree(d_blue_blended1));
	checkCudaErrors(hipFree(d_green_blended1));
	checkCudaErrors(hipFree(d_red_blended2));
	checkCudaErrors(hipFree(d_blue_blended2));
	checkCudaErrors(hipFree(d_green_blended2));
	checkCudaErrors(hipFree(d_sourceImg));
	checkCudaErrors(hipFree(d_destImg));
	checkCudaErrors(hipFree(d_blendedImg));
}
