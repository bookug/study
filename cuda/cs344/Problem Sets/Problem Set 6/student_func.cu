#include "hip/hip_runtime.h"
//Udacity HW 6
//Poisson Blending

/* Background
   ==========

   The goal for this assignment is to take one image (the source) and
   paste it into another image (the destination) attempting to match the
   two images so that the pasting is non-obvious. This is
   known as a "seamless clone".

   The basic ideas are as follows:

   1) Figure out the interior and border of the source image
   2) Use the values of the border pixels in the destination image 
      as boundary conditions for solving a Poisson equation that tells
      us how to blend the images.
   
      No pixels from the destination except pixels on the border
      are used to compute the match.

   Solving the Poisson Equation
   ============================

   There are multiple ways to solve this equation - we choose an iterative
   method - specifically the Jacobi method. Iterative methods start with
   a guess of the solution and then iterate to try and improve the guess
   until it stops changing.  If the problem was well-suited for the method
   then it will stop and where it stops will be the solution.

   The Jacobi method is the simplest iterative method and converges slowly - 
   that is we need a lot of iterations to get to the answer, but it is the
   easiest method to write.

   Jacobi Iterations
   =================

   Our initial guess is going to be the source image itself.  This is a pretty
   good guess for what the blended image will look like and it means that
   we won't have to do as many iterations compared to if we had started far
   from the final solution.

   ImageGuess_prev (Floating point)
   ImageGuess_next (Floating point)

   DestinationImg
   SourceImg

   Follow these steps to implement one iteration:

   1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
      Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

      Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

   2) Calculate the new pixel value:
      float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
      ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]


    In this assignment we will do 800 iterations.
   */



#include "utils.h"
#include <thrust/host_vector.h>

//NOTICE: there are two ways to end the iterations:
//1. set a tiny threshold, and stop when difference between the previous and the current is smaller than it
//2. set a limit of iteration num
#define LIMIT 800

//ANALYSIS: the source image is really sparse, so we must compress it to be placed in a block's shared memory(<= 48KB)
//However, this sparse graph is very special, and CSR is not fit here due to its complexity
//(and maybe not so efficient due to its separated visits to memory)
//In the source graph, only the small part of center is useful, so we can use offset and a small matrix to represent
//(which is more simple, and can help save more memory)
//In fact, CSR is just like place an adjacent list in an array
typedef unsigned char uchar;

__global__
void mask_kernel(const uchar4* const d_sourceImg, uchar* d_mask, size_t numRows, size_t numCols)
{
	int xpos = threadIdx.x + blockIdx.x * blockDim.x;
	int ypos = threadIdx.y + blockIdx.y * blockDim.y;
	if(xpos >= numCols || ypos >= numRows)
	{
		return; 
	}
	int coord = ypos * numCols + xpos;
	uchar4 pixel = d_sourceImg[coord];
	if(pixel.x != 255 || pixel.y != 255 || pixel.z != 255)
	{
		d_mask[coord] = 1;
	}
	/*else*/
	/*{*/
		/*return;*/
	/*}*/
	/*__syncthreads();*/
	/*//WARN: maybe not all blocks synced!!!  even they are the neighbors(may not in a block)*/
	/*bool up = false, down = false, left = false, right = false;*/
	/*if(ypos > 0 && d_mask[coord-numCols] == 1)*/
	/*{*/
		/*up = true;*/
	/*}*/
	/*if(ypos < numRows-1 && d_mask[coord+numCols] == 1)*/
	/*{*/
		/*down = true;*/
	/*}*/
	/*if(xpos > 0 && d_mask[coord-1] == 1)*/
	/*{*/
		/*left = true;*/
	/*}*/
	/*if(xpos < numCols-1 && d_mask[coord+1] == 1)*/
	/*{*/
		/*right = true;*/
	/*}*/
	/*__syncthreads();*/
	/*if(!up || !down || !left || !right)*/
	/*{*/
		/*d_mask[coord] = 2;*/
	/*}*/
}
	
__global__
void border_kernel(const uchar* const d_mask, uchar* const d_mask_tmp, const size_t numRows, const size_t numCols)
{
	int xpos = threadIdx.x + blockIdx.x * blockDim.x;
	int ypos = threadIdx.y + blockIdx.y * blockDim.y;
	if(xpos >= numCols || ypos >= numRows)
	{
		return; 
	}
	int coord = ypos * numCols + xpos;
	if(d_mask[coord] == 0)
	{
		d_mask_tmp[coord] = 0;
		return; 
	}

	bool up = false, down = false, left = false, right = false;
	if(ypos > 0 && d_mask[coord-numCols] == 1)
	{
		up = true;
	}
	if(ypos < numRows-1 && d_mask[coord+numCols] == 1)
	{
		down = true;
	}
	if(xpos > 0 && d_mask[coord-1] == 1)
	{
		left = true;
	}
	if(xpos < numCols-1 && d_mask[coord+1] == 1)
	{
		right = true;
	}
	if(!up || !down || !left || !right)
	{
		d_mask_tmp[coord] = 2;
	}
	else
	{
		d_mask_tmp[coord] = 1;
	}
}

void computeMask(const uchar4* const d_sourceImg, uchar*& d_mask, const size_t numRows, const size_t numCols)
{
	size_t numPixels = numRows * numCols;
	checkCudaErrors(hipMemset(d_mask, 0, sizeof(uchar) * numPixels));
	int xsize = 32, ysize = 32;
	const dim3 threads(xsize, ysize, 1);
	const dim3 blocks((numCols+xsize-1)/xsize, (numRows+ysize-1)/ysize, 1);
	mask_kernel<<<blocks, threads>>>(d_sourceImg, d_mask, numRows, numCols);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
//BETTER: how about judge from d_sourceImg directly, 1+4

	//NOTICE: we can not finish this process in a round because there are many blocks and we can not synchronize them
	uchar* d_mask_tmp = NULL;
	checkCudaErrors(hipMalloc(&d_mask_tmp, sizeof(uchar) * numRows * numCols));

	border_kernel<<<blocks, threads>>>(d_mask, d_mask_tmp, numRows, numCols);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipFree(d_mask));
	d_mask = d_mask_tmp;
}

__global__
void jacobi_kernel(float* d_in, float* d_out, float* d_color_pre, uchar* d_color_dst, uchar* d_mask, const size_t numRows, const size_t numCols)
{
	int xpos = threadIdx.x + blockIdx.x * blockDim.x;
	int ypos = threadIdx.y + blockIdx.y * blockDim.y;
	if(xpos >= numCols || ypos >= numRows)
	{
		return; 
	}
	int coord = ypos * numCols + xpos;
	if(d_mask[coord] != 1)
	{
		return; 
	}
    float blendedSum = 0.f;
    float borderSum  = 0.f;

	//NOTICE: interior pixel can not be in the border, so we do not need to check
    if (d_mask[coord - 1] == 1) 
	{
      blendedSum += d_in[coord - 1];
    }
    else 
	{
      borderSum += d_color_dst[coord - 1];
    }

    if (d_mask[coord + 1] == 1) 
	{
      blendedSum += d_in[coord + 1];
    }
    else 
	{
      borderSum += d_color_dst[coord + 1];
    }

    if (d_mask[coord - numCols] == 1) 
	{
      blendedSum += d_in[coord - numCols];
    }
    else 
	{
      borderSum += d_color_dst[coord - numCols];
    }

    if (d_mask[coord + numCols] == 1) 
	{
      blendedSum += d_in[coord + numCols];
    }
    else 
	{
      borderSum += d_color_dst[coord + numCols];
    }

    float f_next_val = (blendedSum + borderSum + d_color_pre[coord]) / 4.f;
    d_out[coord] = min(255.f, max(0.f, f_next_val)); //clip to [0, 255]
}

void Jacobi(float* d_color_blended1, float* d_color_blended2, float* d_color_pre, uchar* d_color_dst, uchar* d_mask, const size_t numRows, const size_t numCols)
{
	int xsize = 32, ysize = 32;
	const dim3 threads(xsize, ysize, 1);
	const dim3 blocks((numCols+xsize-1)/xsize, (numRows+ysize-1)/ysize, 1);
	float *d_in = d_color_blended1, *d_out = d_color_blended2, *d_tmp = NULL;
	for(int step = 0; step < LIMIT; ++step)
	{
		jacobi_kernel<<<blocks, threads>>>(d_in, d_out, d_color_pre, d_color_dst, d_mask, numRows, numCols);
		hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
		d_tmp = d_in; d_in = d_out; d_out = d_tmp;
	}
	//final result in d_in, just d_color_blended1 here
}

__global__
void split_kernel(const uchar4* const d_img, uchar* d_red, uchar* d_blue, uchar* d_green, const size_t numRows, const size_t numCols, float* d_red_blended = NULL, float* d_blue_blended = NULL, float* d_green_blended = NULL)
{
	int xpos = threadIdx.x + blockIdx.x * blockDim.x;
	int ypos = threadIdx.y + blockIdx.y * blockDim.y;
	if(xpos >= numCols || ypos >= numRows)
	{
		return; 
	}
	int coord = ypos * numCols + xpos;
	d_red[coord] = d_img[coord].x;
	d_blue[coord] = d_img[coord].y;
	d_green[coord] = d_img[coord].z;
	if(d_red_blended != NULL)
	{
		d_red_blended[coord] = static_cast<float>(d_red[coord]);
		d_blue_blended[coord] = static_cast<float>(d_blue[coord]);
		d_green_blended[coord] = static_cast<float>(d_green[coord]);
	}
}

void splitChannel(const uchar4* const d_img, uchar*& d_red, uchar*& d_blue, uchar*& d_green, const size_t numRows, const size_t numCols, float* d_red_blended = NULL, float* d_blue_blended = NULL, float* d_green_blended = NULL)
{
	const size_t numPixels = numRows * numCols;
	checkCudaErrors(hipMalloc(&d_red, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_green, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_blue, sizeof(float) * numPixels));
	int xsize = 32, ysize = 32;
	const dim3 threads(xsize, ysize, 1);
	const dim3 blocks((numCols+xsize-1)/xsize, (numRows+ysize-1)/ysize, 1);
	split_kernel<<<blocks, threads>>>(d_img, d_red, d_blue, d_green, numRows, numCols, d_red_blended, d_blue_blended, d_green_blended);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

__global__
void pre_kernel(uchar* d_color_src, float* d_color_pre, uchar* d_mask, size_t numRows, size_t numCols)
{
	int xpos = threadIdx.x + blockIdx.x * blockDim.x;
	int ypos = threadIdx.y + blockIdx.y * blockDim.y;
	if(xpos >= numCols || ypos >= numRows)
	{
		return; 
	}
	int coord = ypos * numCols + xpos;
	if(d_mask[coord] != 1)
	{
		return; 
	}
    float sum = 4.f * (float)d_color_src[coord];
    sum = sum - (float)d_color_src[coord - 1] - (float)d_color_src[coord + 1];
    sum = sum - (float)d_color_src[coord + numCols] - (float)d_color_src[coord - numCols];
    d_color_pre[coord] = sum;
}

void precompute(uchar* d_color_src, float* d_color_pre, uchar* d_mask, const size_t numRows, const size_t numCols)
{
	int xsize = 32, ysize = 32;
	const dim3 threads(xsize, ysize, 1);
	const dim3 blocks((numCols+xsize-1)/xsize, (numRows+ysize-1)/ysize, 1);
	pre_kernel<<<blocks, threads>>>(d_color_src, d_color_pre, d_mask, numRows, numCols);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

__global__
void map_kernel(float* d_red, float* d_blue, float* d_green, uchar4* d_blendedImg, uchar* d_mask, size_t numRows, size_t numCols)
{
	int xpos = threadIdx.x + blockIdx.x * blockDim.x;
	int ypos = threadIdx.y + blockIdx.y * blockDim.y;
	if(xpos >= numCols || ypos >= numRows)
	{
		return; 
	}
	int coord = ypos * numCols + xpos;
	if(d_mask[coord] == 1)
	{
		d_blendedImg[coord].x = static_cast<uchar>(d_red[coord]);
		d_blendedImg[coord].y = static_cast<uchar>(d_blue[coord]);
		d_blendedImg[coord].z = static_cast<uchar>(d_green[coord]);
	}
}

void mapResult(float* d_red, float* d_blue, float* d_green, uchar4* d_blendedImg, uchar* d_mask, const size_t numRows, const size_t numCols)
{
	//if true and interior in mask, then copy from buffer
	//NOTICE: assume the mask is 0~N, so the interior is just possible in 1~N-1
	int xsize = 32, ysize = 32;
	const dim3 threads(xsize, ysize, 1);
	const dim3 blocks((numCols+xsize-1)/xsize, (numRows+ysize-1)/ysize, 1);
	map_kernel<<<blocks, threads>>>(d_red, d_blue, d_green, d_blendedImg, d_mask, numRows, numCols);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

void your_blend(const uchar4* const h_sourceImg,  //IN
                const size_t numRowsSource, const size_t numColsSource,
                const uchar4* const h_destImg, //IN
                uchar4* const h_blendedImg) //OUT
{

  /* To Recap here are the steps you need to implement
  
     1) Compute a mask of the pixels from the source image to be copied
        The pixels that shouldn't be copied are completely white, they
        have R=255, G=255, B=255.  Any other pixels SHOULD be copied.

     2) Compute the interior and border regions of the mask.  An interior
        pixel has all 4 neighbors also inside the mask.  A border pixel is
        in the mask itself, but has at least one neighbor that isn't.

     3) Separate out the incoming image into three separate channels

     4) Create two float(!) buffers for each color channel that will
        act as our guesses.  Initialize them to the respective color
        channel of the source image since that will act as our intial guess.

     5) For each color channel perform the Jacobi iteration described 
        above 800 times.

     6) Create the output image by replacing all the interior pixels
        in the destination image with the result of the Jacobi iterations.
        Just cast the floating point values to unsigned chars since we have
        already made sure to clamp them to the correct range.

      Since this is final assignment we provide little boilerplate code to
      help you.  Notice that all the input/output pointers are HOST pointers.

      You will have to allocate all of your own GPU memory and perform your own
      memcopies to get data in and out of the GPU memory.

      Remember to wrap all of your calls with checkCudaErrors() to catch any
      thing that might go wrong.  After each kernel call do:

      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

      to catch any errors that happened while executing the kernel.
  */

	//NOTICE: the source image's size is the same as the target image
	printf("numRowsSource: %lu   numColsSource: %lu\n", numRowsSource, numColsSource); //500x333
	uchar4 *d_sourceImg, *d_destImg, *d_blendedImg;
	size_t numPixels = numRowsSource * numColsSource;
	checkCudaErrors(hipMalloc(&d_sourceImg, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMalloc(&d_destImg, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMalloc(&d_blendedImg, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMemcpy(d_sourceImg, h_sourceImg, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_destImg, h_destImg, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));

	//BETTER: we may release unnecessary memory as early as possible
	//Besides, shared memory should be utilized! 
	//And we should improve the occupancy, which means we may crop the core part of the images

	//NOTICE: we believe that the useful part in source image is very small, and 
	//the border line is smaller than the interior
	uchar* d_mask;
	checkCudaErrors(hipMalloc(&d_mask, sizeof(uchar) * numPixels));
	//NOTICE: it is not so easy to build an interior list
	/*uint2* d_interior;*/
	/*checkCudaErrors(hipMalloc(&d_interior, sizeof(uint2) * numPixels));*/
	//find the border of the source image
	computeMask(d_sourceImg, d_mask, numRowsSource, numColsSource);

	//check the mask
	/*uchar* h_mask = (uchar*)malloc(sizeof(uchar)*numPixels);*/
	/*checkCudaErrors(hipMemcpy(h_mask, d_mask, sizeof(uchar)*numPixels, hipMemcpyDeviceToHost));*/
	/*uchar* h_mask_cmp = (uchar*)malloc(sizeof(uchar)*numPixels);*/
	/*for (int i = 0; i < numPixels; ++i) */
	/*{*/
		/*h_mask_cmp[i] = (h_sourceImg[i].x + h_sourceImg[i].y + h_sourceImg[i].z < 3 * 255) ? 1 : 0;*/
	/*}*/
	/*for(int r = 1; r < numRowsSource-1; ++r)*/
	/*{*/
		/*for(int c = 1; c < numColsSource-1; ++c)*/
		/*{*/
			/*int pos = r * numColsSource + c;*/
			/*if(h_mask_cmp[pos] == 0)*/
			/*{*/
				/*continue;*/
			/*}*/
			/*if(h_mask_cmp[pos-1] == 0 || h_mask_cmp[pos+1] == 0 || h_mask_cmp[pos-numColsSource] == 0 || h_mask_cmp[pos+numColsSource] == 0)*/
			/*{*/
				/*h_mask_cmp[pos] = 2;*/
			/*}*/
		/*}*/
	/*}*/
	/*for(int i = 0; i < numPixels; ++i)*/
	/*{*/
		/*if(h_mask[i] != h_mask_cmp[i])*/
		/*{*/
			/*printf("Not matched for mask!\n");*/
			/*break;*/
		/*}*/
	/*}*/
	/*printf("check mask!\n");*/
	/*free(h_mask); free(h_mask_cmp);*/

	//TODO+DEBUG: the bug is in the border, and the mask is good

	float *d_red_blended1, *d_red_blended2, *d_blue_blended1, *d_blue_blended2, *d_green_blended1, *d_green_blended2;
	checkCudaErrors(hipMalloc(&d_red_blended1, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_red_blended2, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_blue_blended1, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_blue_blended2, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_green_blended1, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_green_blended2, sizeof(float) * numPixels));
	//separate into 3 channels, and two buffers
	uchar *d_red_src = NULL, *d_green_src = NULL, *d_blue_src = NULL;
	uchar *d_red_dst = NULL, *d_green_dst = NULL, *d_blue_dst = NULL;
	splitChannel(d_sourceImg, d_red_src, d_blue_src, d_green_src, numRowsSource, numColsSource, d_red_blended1, d_blue_blended1, d_green_blended1);
	splitChannel(d_destImg, d_red_dst, d_blue_dst, d_green_dst, numRowsSource, numColsSource);

	float *d_red_pre, *d_blue_pre, *d_green_pre;
	checkCudaErrors(hipMalloc(&d_red_pre, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_blue_pre, sizeof(float) * numPixels));
	checkCudaErrors(hipMalloc(&d_green_pre, sizeof(float) * numPixels));
	checkCudaErrors(hipMemset(d_red_pre, 0, sizeof(float) * numPixels));
	checkCudaErrors(hipMemset(d_blue_pre, 0, sizeof(float) * numPixels));
	checkCudaErrors(hipMemset(d_green_pre, 0, sizeof(float) * numPixels));
	//pre-compute the sum of source image and neighbors
	precompute(d_red_src, d_red_pre, d_mask, numRowsSource, numColsSource);
	precompute(d_blue_src, d_blue_pre, d_mask, numRowsSource, numColsSource);
	precompute(d_green_src, d_green_pre, d_mask, numRowsSource, numColsSource);

	//NOTICE: iterations are very costly, so the main focus is to reduce the work in iterations
	Jacobi(d_red_blended1, d_red_blended2, d_red_pre, d_red_dst, d_mask, numRowsSource, numColsSource);
	Jacobi(d_blue_blended1, d_blue_blended2, d_blue_pre, d_blue_dst, d_mask, numRowsSource, numColsSource);
	Jacobi(d_green_blended1, d_green_blended2, d_green_pre, d_green_dst, d_mask, numRowsSource, numColsSource);
	//BETTER: consider using cuda stream here, or async memcpy, or hipHostRegister

	checkCudaErrors(hipMemcpy(d_blendedImg, d_destImg, sizeof(uchar4) * numPixels, hipMemcpyDeviceToDevice));
    //create the output image by replacing all the interior pixels
	mapResult(d_red_blended1, d_blue_blended1, d_green_blended1, d_blendedImg, d_mask, numRowsSource, numColsSource);

	checkCudaErrors(hipMemcpy(h_blendedImg, d_blendedImg, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_mask));
	checkCudaErrors(hipFree(d_red_src));
	checkCudaErrors(hipFree(d_blue_src));
	checkCudaErrors(hipFree(d_green_src));
	checkCudaErrors(hipFree(d_red_dst));
	checkCudaErrors(hipFree(d_blue_dst));
	checkCudaErrors(hipFree(d_green_dst));
	checkCudaErrors(hipFree(d_red_pre));
	checkCudaErrors(hipFree(d_blue_pre));
	checkCudaErrors(hipFree(d_green_pre));
	checkCudaErrors(hipFree(d_red_blended1));
	checkCudaErrors(hipFree(d_blue_blended1));
	checkCudaErrors(hipFree(d_green_blended1));
	checkCudaErrors(hipFree(d_red_blended2));
	checkCudaErrors(hipFree(d_blue_blended2));
	checkCudaErrors(hipFree(d_green_blended2));
	checkCudaErrors(hipFree(d_sourceImg));
	checkCudaErrors(hipFree(d_destImg));
	checkCudaErrors(hipFree(d_blendedImg));
}
