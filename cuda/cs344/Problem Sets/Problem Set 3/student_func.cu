#include "hip/hip_runtime.h"
//HELP: http://blog.csdn.net/suda072605/article/details/19965857

/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"
#include <stdio.h>
/*#include <iostream>*/

__global__ void
reduce_kernel(float* const d_out, float* const d_in, const size_t curlen, const bool less)
{
	int tid = threadIdx.x;
	int myId = tid + blockDim.x * blockIdx.x;
	if(myId >= curlen)
	{
		return;
	}
	//NOTICE: it is ok to be negative
	//PERORMANCE: use IO in GPU is very slow, need to transfer to cpu memory and output
	/*if(d_in[myId] < 0)*/
	/*{*/
		/*printf("error: %f!\n", d_in[myId]);*/
	/*}*/

	for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if(tid < s && myId + s < curlen)
		{
			float tmp1 = d_in[myId], tmp2 = d_in[myId+s];
			if(less)
			{
				d_in[myId] = min(tmp1, tmp2);
			}
			else
			{
				d_in[myId] = max(tmp1, tmp2);
			}
		}
		__syncthreads();
	}

	if(tid == 0)
	{
		d_out[blockIdx.x] = d_in[myId];
	}
	/*if(myId == 0)*/
	/*{*/
		/*float mine = d_in[0], maxe = d_in[0];*/
		/*for(unsigned i = 1; i < curlen; ++i)*/
		/*{*/
			/*mine = min(d_in[i], mine);*/
			/*maxe = max(d_in[i], maxe);*/
		/*}*/
		/*printf("check min %f max %f\n", mine, maxe);*/
	/*}*/
}

void reduce(const float* const d_logluminance, const size_t numlen, float* const d_border, const bool less)
{
	//BETTER: return min/max dierctly
	/*printf("here is reduce!\n");*/
	int limit = log(numlen)/log(2) + 1;
	int size = 1024;
	const dim3 threads(size, 1, 1);
	int curlen = numlen;
	float*  d_in;
	float *d_mid;
	checkCudaErrors(hipMalloc((void**)&d_in, sizeof(float) * numlen));
	checkCudaErrors(hipMemcpy(d_in, d_logluminance, sizeof(float) * numlen, hipMemcpyDeviceToDevice));

	/*printf("begin while loop!\n");*/
	while(limit--)
	{
		/*printf("loop loop\n");*/
		int num_blocks = curlen / size;
		if(curlen % size != 0)
		{
			num_blocks ++;
		}
		checkCudaErrors(hipMalloc((void**)&d_mid, sizeof(float) * num_blocks));

		const dim3 blocks(num_blocks, 1, 1);
		reduce_kernel<<<blocks, threads>>>(d_mid, d_in, curlen, less);
		hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

		/*printf("before cuda free!\n");*/
		/*printf("curlen: %d num_blocks: %d\n", curlen, num_blocks);*/
		checkCudaErrors(hipFree(d_in));
		/*printf("after cuda free!\n");*/
		curlen = num_blocks;
		if(curlen == 1)
		{
			break;
		}
		d_in = d_mid;
	}

	//the result is in d_mid
	checkCudaErrors(hipMemcpy(d_border, &d_mid[0], sizeof(float), hipMemcpyDeviceToDevice));
	//NOTICE: below is error, device memory can not be copied directly in the host code
	//while in kernel code, device memory can be used directly
	/**d_border = d_mid[0];*/
	checkCudaErrors(hipFree(d_mid));
}

__global__ void
histogram_kernel(const float* const d_in, unsigned* const d_out, const size_t numlen, const float min_logLum, const float range, const size_t numBins)
{
	int tid = threadIdx.x;
	int myId = tid + blockDim.x * blockIdx.x;
	if(myId >= numlen)
	{
		return;
	}
	unsigned key = static_cast<unsigned>((d_in[myId] - min_logLum) / range);
	if(key >= numBins)
	{
		key = numBins - 1;
	}
	/*d_out[key] += 1;*/
	atomicAdd(&d_out[key], 1);
	//just for check
	/*if(myId == 0)*/
	/*{*/
	/*}*/
}

void histogram(const float* const d_logluminance, const size_t numlen, unsigned* const d_hist, const float min_logLum, const float range, const size_t numBins)
{
	//BETTER: sum in a block first (all bins in shared memory for each block) and then scan and sum

	//NOTICE: if (x-min)/range is out of bound, bind it to largest bin
	/*printf("here is histogram!\n");*/
	int size = 1024;
	const dim3 threads(size, 1, 1);
	const dim3 blocks((numlen+size-1)/size, 1, 1);
	//NOTICE: set them to 0 first!
	checkCudaErrors(hipMemset(d_hist, 0, sizeof(unsigned) * numBins));
	histogram_kernel<<<blocks, threads>>>(d_logluminance, d_hist, numlen, min_logLum, range, numBins);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

__global__ void
presum_kernel(unsigned* const d_in, unsigned* const d_out, const size_t numBins)
{
	//NOTICE: if we do not know the size when compiling, use extern keyword and pass this parameter when launching the kernel
	extern __shared__ unsigned arr[];
	/*__shared__ unsigned* arr_out;*/
	int tid = threadIdx.x;
	int myId = tid + blockDim.x * blockIdx.x;
	if(myId >= numBins)
	{
		return;
	}
	arr[myId] = d_in[myId];
	__syncthreads();

	for(unsigned span = 1; span < numBins; span <<= 1)
	{
		if(myId < span)
		{
			//thsi can reduce the copy times
			break;
		}
		unsigned val = arr[myId-span];
		//NOTICE: only sync in each block
		__syncthreads();
		arr[myId] += val;
		__syncthreads();
	}
	//set d_out by exclusive presum
	if(myId == 0)
	{
		d_out[0] = 0;
	}
	if(myId < numBins - 1)
	{
		d_out[myId+1] = arr[myId];
	}
}

void presum(unsigned* const d_hist, const size_t numBins, unsigned int* const d_cdf)
{
	/*printf("here is presum!\n");*/
	int size = 1024;
	//NOTICE: there is only 1024 bins, so we can place them in a block! then the problem can be simplified
	//If several blocks, can compute parts in each block then add base!
	const dim3 threads(size, 1, 1);
	const dim3 blocks((numBins+size-1)/size, 1, 1);
	//NOTICE: we want to compute the exclusive scan, rather than the inclusive scan
	/*checkCudaErrors(hipMemset(&d_cdf[0], 0, sizeof(unsigned)));*/
	presum_kernel<<<blocks, threads, sizeof(unsigned) * numBins>>>(d_hist, d_cdf, numBins);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

//NOTICE: size_t is unsigned long
void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

	//NOTICE: numCols is the length of a picture, numRows is the width of a picture
	//blockSize((numCols+15)/16, (numRows+15)/16, 1) is a way to compute right blocks!

	//Be careful if overflow
	size_t numLen = numRows * numCols; 
	float *d_border, h_border;
	checkCudaErrors(hipMalloc((void**)&d_border, sizeof(float)));
	//reduce to find the minium
	reduce(d_logLuminance, numLen, d_border, true); 
	checkCudaErrors(hipMemcpy(&h_border, d_border, sizeof(float), hipMemcpyDeviceToHost));

	min_logLum = h_border;
	//reduce to find the maxium
	reduce(d_logLuminance, numLen, d_border, false);  

	checkCudaErrors(hipMemcpy(&h_border, d_border, sizeof(float), hipMemcpyDeviceToHost));
	max_logLum = h_border;
	//free the device memory
	checkCudaErrors(hipFree(d_border));

	//build the histogram struct
	float range = max_logLum - min_logLum;
	//NOTICE: this is the log luminance, so it can be negative, and the maximum is much smaller than 275
	/*printf("%f %lu\n", range, numBins);*/
	range = range / numBins;
	/*printf("min: %f max: %f range: %f\n", min_logLum, max_logLum, range);*/
	/*fflush(stdout);*/

	unsigned *d_hist;
	checkCudaErrors(hipMalloc((void**)&d_hist, sizeof(unsigned) * numBins));
	histogram(d_logLuminance, numLen, d_hist, min_logLum, range, numBins); 
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	//exclusive scan: not include hist[numBins-1]
	presum(d_hist, numBins, d_cdf);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_hist));
}
