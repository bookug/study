#include "hip/hip_runtime.h"
/*=============================================================================
# Filename: Match.cpp
# Author: Bookug Lobert 
# Mail: 1181955272@qq.com
# Last Modified: 2016-12-15 01:38
# Description: how to use UVA(unified virtual addressing)
https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__UNIFIED.html
https://blog.csdn.net/langb2014/article/details/51348616
=============================================================================*/

#include "Match.cuh"
#include <hip/hip_runtime.h>


#define MAXSTREAM 20
//TODO:
//adjust the maximum
#define MAXLAUNCHEDTHREADS 9999
#define MAXTHREADSPERBLOCK 512
#define MAXLEFTNODE 15
using namespace std;

hipStream_t stream[MAXSTREAM];

int egdeExistJudge(int* id, int* label, int length, int target)
{
	if(length <= 20){
		for(int i = 0; i < length; i ++){
			if(id[i] == target)
				return label[i];
		}
	}else{
		int s = 0, e = length - 1;
	    int mid = (s + e)/2;
	    while (s <= e)
	    {
	            if(id[mid] == target)
	            {
	                    return label[mid];
	            }
	            else if(id[mid] < target)
	            {
	                    s = mid + 1;
	            }
	            else {
	                    e = mid - 1;
	            }
	            mid = (s + e)/2;
	    }
	}
	return -1;
}
__device__ int d_egdeExistJudge(int* id, int* label, int length, int target)
{
	if(length <= 20){
		for(int i = 0; i < length; i ++){
			if(id[i] == target)
				return label[i];
		}
	}else{
		int s = 0, e = length - 1;
	    int mid = (s + e)/2;
	    while (s <= e)
	    {
	            if(id[mid] == target)
	            {
	                    return label[mid];
	            }
	            else if(id[mid] < target)
	            {
	                    s = mid + 1;
	            }
	            else {
	                    e = mid - 1;
	            }
	            mid = (s + e)/2;
	    }
	}
	return -1;
}


Match::Match(Graph* _query, Graph* _data)
{
	printf("enter match here\n");
	this->query = _query;
	this->data = _data;
	this->qsize = _query->vSize();
	this->dsize = _data->vSize();

	this->qcore = new int[qsize];
	this->qin = new int[qsize];
	this->qout = new int[qsize];

	this->dcore = new int[dsize];
	this->din = new int[dsize];
	this->dout = new int[dsize];
	
	this->streamNum = 0;
	
	this->matchOrder = new int[qsize];

	int degrees[qsize];
	memset(degrees,0,qsize*sizeof(int));
	for(int i = 0; i < qsize; i ++)
	{
		degrees[i] = _query->vertices[i].in.size() + _query->vertices[i].out.size();
	}
	//sort node according to the degree
	for(int i = 0; i < qsize; i++)
	{
		int maxDegree = degrees[0];
		int maxPos = 0;
		for(int j = 0; j < qsize; j ++)
		{
			if(degrees[j] > maxDegree)
			{
				maxDegree = degrees[j];
				maxPos = j;
			}
		}
		matchOrder[i] = maxPos;
		degrees[maxPos] = -1;
	}
	//find the min cover node set;
	int covered[qsize] ;
	memset(covered,0,qsize*sizeof(int));
	int coveredNum = 0;
	for(int i = 0; i < qsize; i ++)
	{
		int qid = matchOrder[i];
		if(covered[qid] == 0)
			coveredNum ++;
		covered[qid] = 1;
		for(int j = 0; j < _query->vertices[qid].in.size(); j ++)
		{
			int id = _query->vertices[qid].in[j].vid;
			if(covered[id] == 0)
				coveredNum ++;
			covered[id] = 1;
		}
		for(int j = 0; j < _query->vertices[qid].out.size(); j ++)
		{
			int id = _query->vertices[qid].out[j].vid;
			if(covered[id] == 0)
				coveredNum ++;
			covered[id] = 1;
		}
		if(coveredNum == qsize)
		{
			minCoverPos = i;
			break;
		}
	}
	
	printf("the following is the min cover of query graph");
	for(int i = 0; i <= minCoverPos; i ++)
	{
		printf("%d ",matchOrder[i]);
	}
	printf("\n");
	printf("query graph processed\n\n\n");
	//move csr to GPU
	this->qTotalInNum = _query->totalInNum;
	this->qTotalOutNum = _query->totalOutNum;

	this->dTotalInNum = _data->totalInNum;
	this->dTotalOutNum = _data->totalOutNum;
		
	//cuda init
	int t_0,t_1;
        t_0 = Util::get_cur_time();
	int* warmup = NULL;
	hipMalloc(&warmup, sizeof(int));
	hipFree(warmup);
	cout<<"GPU warmup finished"<<endl;
	size_t size = 0x7fffffff;
	/*size *= 3;   //heap corruption for 3 and 4*/
	size *= 5;
	/*size *= 2;*/
	//NOTICE: the memory alloced by hipMalloc is different from the GPU heap(for new/malloc in kernel functions)
	hipDeviceSetLimit(hipLimitMallocHeapSize, size);
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
	cout<<"check heap limit: "<<size<<endl;	
	t_1 = Util::get_cur_time();
    printf("cuda init done,it uses %d ms\n",t_1 - t_0);

	
	hipMalloc(&qlabels, sizeof(int)*qsize);
	hipMalloc(&qInRowOffset, sizeof(int)*(qsize+1));
	hipMalloc(&qOutRowOffset, sizeof(int)*(1+qsize));
	hipMalloc(&qInValues,sizeof(int)*qTotalInNum);
	hipMalloc(&qInColOffset,sizeof(int)*qTotalInNum);
	hipMalloc(&qOutValues,sizeof(int)*qTotalOutNum);
	hipMalloc(&qOutColOffset,sizeof(int)*qTotalOutNum);
	
	hipMalloc(&dlabels, sizeof(int)*dsize);
	hipMalloc(&dInRowOffset, sizeof(int)*(dsize+1));
	hipMalloc(&dOutRowOffset, sizeof(int)*(1+dsize));
	hipMalloc(&dInValues,sizeof(int)*dTotalInNum);
	hipMalloc(&dInColOffset,sizeof(int)*dTotalInNum);
	hipMalloc(&dOutValues,sizeof(int)*dTotalOutNum);
	hipMalloc(&dOutColOffset,sizeof(int)*dTotalOutNum);
	
	printf("malloc in build match all done\n");
//	printf("the qsize is %d,and the query labels is \n",qsize);
//	for(int i = 0; i < qsize; i++)
//		printf("the qlabel %d is %d\n",i,_query->labels[i]);	
	hipMemcpy((void *)qlabels, (void *)_query->labels, sizeof(int)*qsize, hipMemcpyHostToDevice);

	printf("first cpy done\n");
	hipMemcpy((void *)qInRowOffset, (void *)_query->inRowOffset, sizeof(int)*(qsize+1), hipMemcpyHostToDevice);
	hipMemcpy((void *)qOutRowOffset, (void *)_query->outRowOffset, sizeof(int)*(qsize+1), hipMemcpyHostToDevice);
	hipMemcpy((void *)qInColOffset, (void *)_query->inColOffset, sizeof(int)*qTotalInNum, hipMemcpyHostToDevice);
	hipMemcpy((void *)qOutColOffset, (void *)_query->outColOffset, sizeof(int)*qTotalOutNum, hipMemcpyHostToDevice);
	hipMemcpy((void *)qInValues, (void *)_query->inValues, sizeof(int)*qTotalInNum, hipMemcpyHostToDevice);
	hipMemcpy((void *)qOutValues, (void *)_query->outValues, sizeof(int)*qTotalOutNum, hipMemcpyHostToDevice);

	hipMemcpy((void *)dlabels, (void *)_data->labels, sizeof(int)*dsize, hipMemcpyHostToDevice);
	hipMemcpy((void *)dInRowOffset, (void *)_data->inRowOffset, sizeof(int)*(dsize+1), hipMemcpyHostToDevice);
	hipMemcpy((void *)dOutRowOffset, (void *)_data->outRowOffset, sizeof(int)*(dsize+1), hipMemcpyHostToDevice);
	hipMemcpy((void *)dInColOffset, (void *)_data->inColOffset, sizeof(int)*dTotalInNum, hipMemcpyHostToDevice);
	hipMemcpy((void *)dOutColOffset, (void *)_data->outColOffset, sizeof(int)*dTotalOutNum, hipMemcpyHostToDevice);
	hipMemcpy((void *)dInValues, (void *)_data->inValues, sizeof(int)*dTotalInNum, hipMemcpyHostToDevice);
	hipMemcpy((void *)dOutValues, (void *)_data->outValues, sizeof(int)*dTotalOutNum, hipMemcpyHostToDevice);
	//TODO:
	//hipDeviceSynchronize();
	printf("move csr to GPU done\n");
}

Match::~Match()
{
	delete[] this->qcore;
	delete[] this->qin;
	delete[] this->qout;

	delete[] this->dcore;
	delete[] this->din;
	delete[] this->dout;

	delete[] this->matchOrder;

	hipFree(qlabels);
	hipFree(qInRowOffset);
	hipFree(qInColOffset);
	hipFree(qInValues);
	hipFree(qOutValues);
	hipFree(qOutColOffset);
	hipFree(qOutRowOffset);

	hipFree(dlabels);
	hipFree(dInValues);
	hipFree(dInColOffset);
	hipFree(dInRowOffset);
	hipFree(dOutValues);
	hipFree(dOutColOffset);
	hipFree(dOutRowOffset);		
}

//Two States of core queues: matching [0,size) not-matching -1
//Four States of in/out queues: matching depth   in_queue depth  out_queue depth other -1
//NOTICE:a vertex maybe appear in in_queue and out_queue at the same time
void 
Match::match(IO& io)
{
	if(qsize > dsize)
	{
		return;
	}

	//initialize the structures
	memset(qcore, -1, sizeof(int) * qsize);
	memset(qin, -1, sizeof(int) * qsize);
	memset(qout, -1, sizeof(int) * qsize);
	memset(dcore, -1, sizeof(int) * dsize);
	memset(din, -1, sizeof(int) * dsize);
	memset(dout, -1, sizeof(int) * dsize);

	d_Match * d_m;
	hipMalloc(&d_m,sizeof(d_Match));
	hipMemcpy((void *)d_m, (void *)this,sizeof(Match),hipMemcpyHostToDevice);

	for(int i = 0; i < MAXSTREAM; i ++)
	{
		hipStreamCreate(&stream[i]);
	}
	//call dfs with macthing num(depth)
	printf("start dfs here\n");
	dfs(0, io, minCoverPos,d_m);
	hipDeviceSynchronize();
	printf("end bfs here\n");
	for(int i = 0; i < MAXSTREAM; i ++)
	{
		hipStreamDestroy(stream[i]);
	}
	hipDeviceReset();
}

//NOTICE:here can be different equal functions, maybe simlarity
bool 
Match::equal(LABEL lb1, LABEL lb2)
{
	return lb1 == lb2;
}

bool 
Match::checkCore(vector<Neighbor>& qlist, vector<Neighbor>& dlist)
{
	//data  another array record the mapping label? record the query is better? 2 directions
	int qnum = 0, dnum = 0; 
	int i, j, size1, size2;
	LABEL* temp = new LABEL[qsize];
	//NOTICE+WARN:sizeof(temp) is only 8 bytes
	//memset(temp, -1, sizeof(temp));
	memset(temp, -1, sizeof(LABEL) * qsize);

	size1 = qlist.size();
	for(i = 0; i < size1; ++i)
	{
		j = qlist[i].vid;
		if(qcore[j] >= 0)
		{
			qnum++;
			temp[j] = qlist[i].elb;
		}
	}

	size2 = dlist.size();
	for(i = 0; i < size2; ++i)
	{
		j = dlist[i].vid;
		if(dcore[j] >= 0 && temp[dcore[j]] != -1)
		{
			//check the edge label
			if(dlist[i].elb != temp[dcore[j]])
			{
				delete[] temp;
				return false;
			}
			dnum++;
		}
	}

	delete[] temp;
	if(qnum != dnum)
	{
		return false;
	}
	else
	{
		return true;
	}
}

bool 
Match::checkOther(vector<Neighbor>& qlist, vector<Neighbor>& dlist)
{
	int qin_num = 0, qout_num = 0, qres_num = 0; 
	int din_num = 0, dout_num = 0, dres_num = 0;
	int i, j, size1, size2;

	size1 = qlist.size();
	for(i = 0; i < size1; ++i)
	{
		j = qlist[i].vid;
		if(qin[j] >= 0 && qcore[j] < 0)
		{
			qin_num++;
		}
		if(qout[j] >= 0 && qcore[j] < 0)
		{
			qout_num++;
		}
		if(qin[j] < 0 && qout[j] < 0)
		{
			qres_num++;
		}
	}

	//BETTER?:check if satisfy the pruning limit each time
	size2 = dlist.size();
	for(i = 0; i < size2; ++i)
	{
		j = dlist[i].vid;
		if(din[j] >= 0 && dcore[j] < 0)
		{
			din_num++;
		}
		if(dout[j] >= 0 && dcore[j] < 0)
		{
			dout_num++;
		}
		if(din[j] < 0 && dout[j] < 0)
		{
			dres_num++;
		}
	}

	if(din_num < qin_num || dout_num < qout_num)
	{
		return false;
	}
	else
	{
		return true;
	}
}

bool 
Match::prune()
{
#ifdef DEBUG
	if(qnid < 0 || qnid >= qsize || dnid < 0 || dnid >= dsize)
	{
		cerr<<"ERROR in prune"<<endl;
		return true;
	}
	cerr<<"qnid: "<<qnid<<"\tdnid: "<<dnid<<endl;
	for(int i = 0; i < qsize; ++i)
	{
		cerr<<qcore[i]<<" ";
	}
	cerr<<endl;
	for(int i = 0; i < dsize; ++i)
	{
		cerr<<dcore[i]<<" ";
	}
	cerr<<endl;
#endif

	vector<Neighbor>& qpred = this->query->vertices[qnid].in;
	vector<Neighbor>& qsucc = this->query->vertices[qnid].out;
	vector<Neighbor>& dpred = this->data->vertices[dnid].in;
	vector<Neighbor>& dsucc = this->data->vertices[dnid].out;

	//vertex label: semantic pruning
	if(!equal(this->query->vertices[qnid].label, this->data->vertices[dnid].label))
	{
#ifdef DEBUG
	cerr<<"prune: the vertex label not macthed"<<endl;
#endif
		return true;
	}

	//NOTICE:If not consider labels, core restrictions can be done by computing the num of qnid and dnid matching 
	//with corresponding already-matching sets, just compare the num if equal is ok
	//NOTICE:If considering labels, just build a new array for query which keeps the labels for comparison
	//This array is temporal and small
	if(!checkCore(qpred, dpred))
	{
#ifdef DEBUG
	cerr<<"prune: the pred core not macthed"<<endl;
#endif
		return true;
	}
	if(!checkCore(qsucc, dsucc))
	{
#ifdef DEBUG
	cerr<<"prune: the succ core not macthed"<<endl;
#endif
		return true;
	}

	//compute and compare 6 nums both for pred list
	if(!checkOther(qpred, dpred))
	{
#ifdef DEBUG
	cerr<<"prune: the pred other not macthed"<<endl;
#endif
		return true;
	}
	//compute and compare 6 nums both for succ list
	if(!checkOther(qsucc, dsucc))
	{
#ifdef DEBUG
	cerr<<"prune: the succ other not macthed"<<endl;
#endif
		return true;
	}	
	
	return false;
}

void 
Match::modify(int depth, vector<Neighbor>& list, int* queue)
{
	int size = list.size(), i, j;
	for(i = 0; i < size; ++i)
	{
		j = list[i].vid;
		if(queue[j] < 0)
		{
			queue[j] = depth;
		}
	}
}

void 
Match::update(int depth)
{
	qcore[qnid] = dnid;
	dcore[dnid] = qnid;
	if(qin[qnid] < 0)
		qin[qnid] = depth;
	if(qout[qnid] < 0)
		qout[qnid] = depth;
	if(din[dnid] < 0)
		din[dnid] = depth;
	if(dout[dnid] < 0)
		dout[dnid] = depth;
	
	//add the new in vertices according to qnid
	this->modify(depth, this->query->vertices[qnid].in, qin);
	//add the new out vertices according to qnid
	this->modify(depth, this->query->vertices[qnid].out, qout);
	//add the new in vertices according to dnid
	this->modify(depth, this->data->vertices[dnid].in, din);
	//add the new out vertices according to dnid
	this->modify(depth, this->data->vertices[dnid].out, dout);
}

void 
Match::restore(int depth)
{
	//if(qnid < 0 || qnid >= qsize || dnid < 0 || dnid >= dsize)
	//{
		//cerr<<"ERROR in restore"<<endl;
		//return;
	//}
	qcore[qnid] = -1;
	dcore[dnid] = -1;
	//update the query queues
	for(int i = 0; i < qsize; ++i)
	{
		if(qin[i] == depth)
		{
			qin[i] = -1;
		}
		if(qout[i] == depth)
		{
			qout[i] = -1;
		}
	}
	//update the data queues
	for(int i = 0; i < dsize; ++i)
	{
		if(din[i] == depth)
		{
			din[i] = -1;
		}
		if(dout[i] == depth)
		{
			dout[i] = -1;
		}
	}
}


__global__ void lastJoin1(int * qcore,
                                                int qsize,
                                                int leftNode,
                                                d_Match * dMatch,
                                                int totalThreadNum){
	return ;
}



__global__ void lastJoin(int * qcore, 
						int qsize, 
						int leftNode, 
						d_Match * dMatch,
						int totalThreadNum)
{
//	if(qcore[0] == 4608){
//		if(threadIdx.x == 0)
//			printf("---------this is block %d of target kernel\n",blockIdx.x);
//		printf("target kernek launched!!!q[1] = %d,q[2] = %d\n",qcore[1],qcore[2]);
//	}

	int * cansRowOffset = qcore + qsize;
	int * cansColOffset = cansRowOffset + leftNode + 1;

	int idx = blockIdx.x*MAXTHREADSPERBLOCK + threadIdx.x;
	if(idx >= totalThreadNum)
			return ;
	//TODO: put this array in shared memory or not?
	int *localCans = new int [leftNode];
	int mul = 1;
	for(int i = leftNode-1; i >= 0; i --)
	{
		localCans[i] = cansColOffset[cansRowOffset[i] + (idx/mul)%(cansRowOffset[i+1] - cansRowOffset[i])];
		mul *= (cansRowOffset[i+1] - cansRowOffset[i]);
	}
	__shared__ int leftQidPos[MAXLEFTNODE];
	if(idx%MAXTHREADSPERBLOCK == 0)
	{
		int temp = 0;
		for(int i = 0; i < qsize; i++)
		{
			if(qcore[i] != -1)
				continue;
			leftQidPos[temp] = i;
			temp ++;
		}
	}
	__syncthreads();
//	if(idx == 0){
//		printf("the leftNode is %d\n",leftNode);
		
//	}
       // printf("the idx is %d,the %d pos is %d, the %d pos is %d, the %d pos is %d\n",idx,leftQidPos[0],
	//localCans[0],leftQidPos[1],localCans[1],leftQidPos[2],localCans[2]);

	//use labels to prune
	bool debug = false;
		
	for(int i = 0; i < leftNode; i++)
	{
		if(dMatch->dlabels[localCans[i]] != dMatch->qlabels[leftQidPos[i]])
		{
			delete [] localCans;
			if(debug)
				printf("prune in 1\n");
	//		printf("the idx %d returns\n",idx);
			return;
		}
	}
	for(int i = 0; i < leftNode; i ++)
		for(int j = 0; j < qsize; j++)
		{
			if(qcore[j] == -1)
				continue;
			if(qcore[j] == localCans[i])
			{
				delete [] localCans;
  			if(debug)
          	           printf("prune in 2\n");

//                      printf("the idx %d returns\n",idx);
                	return ;
			}
		}
	for(int i = 0; i < leftNode; i ++)
		for(int j = i+1; j < leftNode; j++)
		{
			if(localCans[i] == localCans[j])
			{
				delete [] localCans;
				if(debug)
           		         printf("prune in 3\n");

//                    printf("the idx %d returns\n",idx);
               			 return ;
			}
		}
				
//	printf("use edge to prune\n");				
//use in edge to prune
	for(int i = 0; i < leftNode; i++)
	{
		//j represent the adj id of i
		for(int j = 0; j < dMatch->qInRowOffset[leftQidPos[i]+1] - dMatch->qInRowOffset[leftQidPos[i]]; j ++)
		{
			int adjQid = dMatch->qInColOffset[dMatch->qInRowOffset[leftQidPos[i]] + j];
			if(qcore[adjQid] == -1)
			{	
				 if(adjQid > leftQidPos[i])
				 	continue;
				 int rightPos = 0;
				 for(int l = 0; l < leftNode; l++)
				 	if(leftQidPos[l] == adjQid)
				 	{
				 		rightPos = l;
				 		break;
				 	}
				 if(d_egdeExistJudge(dMatch->dInColOffset + dMatch->dInRowOffset[localCans[i]], 
								dMatch->dInValues + dMatch->dInRowOffset[localCans[i]], 
								dMatch->dInRowOffset[localCans[i]+1] - dMatch->dInRowOffset[localCans[i]], 
								localCans[rightPos]) 
								!= dMatch->qInValues[dMatch->qInRowOffset[leftQidPos[i]] + j])
				 {
					 if(debug)
                                 		printf("prune in 4\n");

				 	delete [] localCans;
					return ;
				 }

			}
			else if(d_egdeExistJudge(dMatch->dInColOffset + dMatch->dInRowOffset[localCans[i]], 
								dMatch->dInValues + dMatch->dInRowOffset[localCans[i]], 
								dMatch->dInRowOffset[localCans[i]+1] - dMatch->dInRowOffset[localCans[i]], 
								qcore[adjQid]) 
								!= dMatch->qInValues[dMatch->qInRowOffset[leftQidPos[i]] + j])
			{
				
	/*			if(debug)
				{
          		                printf("prune in 4\n");
					printf("the leftNode qid is %d,the adj qid is %d\n",leftQidPos[i],dMatch->qInColOffset[dMatch->qInRowOffset[leftQidPos[i]] + j]);
					printf("the leftNode did is %d,the adj did is %d\n",localCans[i],qcore[adjQid]);
					printf("here is the adj list of leftNode did:\n");
					for(int l = 0; l < dMatch->dInRowOffset[localCans[i]+1] - dMatch->dInRowOffset[localCans[i]]; l ++)
					{
						printf("id is %d, label is %d\n",dMatch->dInColOffset[dMatch->qInRowOffset[localCans[i]]+l],dMatch->dInValues[dMatch->qInRowOffset[localCans[i]]+l]);
					}	
				}*/
				 if(debug)
                                 printf("prune in 5\n");

					delete [] localCans;
	//			printf("the idx %d returns\n",idx);
				return ;
			}
		}
	}
	for(int i = 0; i < leftNode; i++)
	{
		//j represent the adj id of i
		for(int j = 0; j < dMatch->qOutRowOffset[leftQidPos[i]+1] - dMatch->qOutRowOffset[leftQidPos[i]]; j ++)
		{

			int adjQid = dMatch->qOutColOffset[dMatch->qOutRowOffset[leftQidPos[i]] + j];
			if(debug)
				printf("i is %d, j is %d,leftQidPos[i] is %d,localcans[i] is %d, the qcore[%d] is %d\n",i,j,leftQidPos[i],localCans[i],adjQid,qcore[adjQid]);
			if(qcore[adjQid] == -1)
			{	
				 if(adjQid > leftQidPos[i])
				 	continue;
				 int rightPos = 0;
				 for(int l = 0; l < leftNode; l++)
				 	if(leftQidPos[l] == adjQid)
				 	{
				 		rightPos = l;
				 		break;
				 	}
				 if(d_egdeExistJudge(dMatch->dOutColOffset + dMatch->dOutRowOffset[localCans[i]], 
								dMatch->dOutValues + dMatch->dOutRowOffset[localCans[i]], 
								dMatch->dOutRowOffset[localCans[i]+1] - dMatch->dOutRowOffset[localCans[i]], 
								localCans[rightPos]) 
								!= dMatch->qOutValues[dMatch->qOutRowOffset[leftQidPos[i]] + j])
				 {
					 if(debug)
		                                 printf("prune in 6\n");

				 	delete [] localCans;
					return ;
				 }

			}
			else if(d_egdeExistJudge(dMatch->dOutColOffset + dMatch->dOutRowOffset[localCans[i]], 
									dMatch->dOutValues + dMatch->dOutRowOffset[localCans[i]], 
									dMatch->dOutRowOffset[localCans[i]+1] - dMatch->dOutRowOffset[localCans[i]], 
									qcore[adjQid]) 
									!= dMatch->qOutValues[dMatch->qOutRowOffset[leftQidPos[i]] + j])
			{
				delete [] localCans;
				if(debug){
                                	printf("prune in 7,label 1 is %d,label 2 is %d\n",d_egdeExistJudge(dMatch->dOutColOffset + dMatch->dOutRowOffset[localCans[i]],
                                                                        dMatch->dOutValues + dMatch->dOutRowOffset[localCans[i]],
                                                                        dMatch->dOutRowOffset[localCans[i]+1] - dMatch->dOutRowOffset[localCans[i]],
                                                                        qcore[adjQid]), dMatch->qOutValues[dMatch->qOutRowOffset[leftQidPos[i]] + j]);
					for(int l = 0; l < dMatch->dOutRowOffset[localCans[i]+1] - dMatch->dOutRowOffset[localCans[i]]; l ++)
					{
						printf("%d\n",dMatch->dOutColOffset[dMatch->dOutRowOffset[localCans[i]] + l]);
					}
					
				}
	//			printf("the idx %d returns\n",idx);
				return ;
			}
		}
	}
	if(debug)
		printf("ready to output for target\n");

//	for(int i = 0; i < leftNode; i ++)
//	{
//		qcore[leftQidPos[i]] = localCans[i];
//	}

	//TODO : join all the varities before output
	int t = 0;
//	printf("the idx is %d\n",idx);
	//TODO: better it

	char *outPutString = new char[qsize*15];
	int outPos = 0;
	for(int i = 0; i < qsize; i ++)
	{
		outPutString[outPos++] = '(';
	
		int div = 1;
		int tempNum = i;
		while(tempNum >= 10)
		{
			tempNum /= 10;
			div *= 10;
		}
		tempNum = i;
		while(div > 0)
		{
			outPutString[outPos++] = '0' + tempNum/div;
			tempNum = tempNum%div;
			div = div/10;
		}

	//	sprintf(outPutString + outPos,"%d",i);
	//	outPos = strlen(outPutString);
		outPutString[outPos++] = ',';
		outPutString[outPos++] = ' ';
		int addNum = 0;
		if(qcore[i] == -1)
		{
			addNum = localCans[t++];
	//		sprintf(outPutString + outPos,"%d",localCans[t++]);
	//		outPos += strlen(outPutString);
		}
		else
		{
			addNum = qcore[i];
	//		sprintf(outPutString + outPos,"%d",qcore[i]);
	//		outPos += strlen(outPutString);
		}

		div = 1;
		tempNum = addNum;
		while(tempNum >= 10)
		{
			tempNum /= 10;
			div *= 10;
		}
		tempNum = addNum;
		while(div > 0)
		{
			outPutString[outPos++] = '0' + tempNum/div;
			tempNum = tempNum%div;
			div = div/10;
		}

		outPutString[outPos++] = ')';
		outPutString[outPos++] = ' ';
	//	sprintf(outPutString + outPos,"%s",") ");
        //outPos += 2;
	}
	outPutString[outPos] = '\0';
//	strcpy(outPutString,'\n');
	printf("%s\n",outPutString);
//	printf("%s from block %d,idx %d\n",outPutString,blockIdx.x,threadIdx.x);
	delete [] outPutString;

//	delete [] outPutString;
	return ;
}


void
Match::dfs(int num, IO& io, int queryIdRange, d_Match * dMatch)
{
	//TODO:
	//modify the output 
	if(num == this->qsize)
	{
#ifdef DEBUG
		cerr<<"find a mapping here"<<endl;
#endif
		io.output(this->qcore, this->qsize);
		return;
	}
//	printf("the num is %d\n",num);
	//NOTICE:to avoid duplicates, only consider the smallest qid in candidates
	//int depth = num;

	//prepare the candidates from all cases
	int i, j;
	this->qnid = -1;
	vector<int> cans;

	//consider the out queue
	for(i = 0; i <= queryIdRange; ++i)
	{
		if(qout[matchOrder[i]] < 0 || qcore[matchOrder[i]] >= 0)
		{
			continue;
		}
		this->qnid = matchOrder[i];
		for(int j = 0; j < qsize; j ++)
		{
			if(qcore[j] == -1)
				continue;
			if(egdeExistJudge(query->outColOffset + query->outRowOffset[j], 
							  query->outValues + query->outRowOffset[j], 
							  query->outRowOffset[j+1] - query->outRowOffset[j],
							  matchOrder[i]) != -1)
			{
				int tempData = qcore[j];
				for(int k = 0; k < data->outRowOffset[tempData+1]-data->outRowOffset[tempData]; k++)
				{
					int adjData = data->outColOffset[data->outRowOffset[tempData]+k];
					if(dout[adjData] < 0 || dcore[adjData] >= 0)
					{
						continue;
					}
					cans.push_back(adjData);
				}
				break;
			}
		}/*
		int allCansCount = 0;
		for(int j = 0; j < dsize; ++j)
                {
                        if(dout[j] < 0 || dcore[j] >= 0)
                        {
                                continue;
                        }
                        allCansCount ++;
                }
		printf("allCans is %d,cans infact is %d,%d less\n",allCansCount,cans.size(),allCansCount - cans.size());*/
		break;
		/*
		for(j = 0; j < dsize; ++j)
		{
			if(dout[j] < 0 || dcore[j] >= 0)
			{
				continue;
			}
			cans.push_back(j);
		}*/
	}
	//consider the in queue
	if(qnid == -1)
	{
		for(i = 0; i <= queryIdRange; ++i)
		{
			if(qin[matchOrder[i]] < 0 || qcore[matchOrder[i]] >= 0)
			{
				continue;
			}
			this->qnid = matchOrder[i];
			for(int j = 0; j < qsize; j ++)
			{
				if(qcore[j] == -1)
					continue;
				if(egdeExistJudge(query->inColOffset + query->inRowOffset[j], 
								  query->inValues + query->inRowOffset[j], 
								  query->inRowOffset[j+1] - query->inRowOffset[j],
								  matchOrder[i]) != -1)
				{
					int tempData = qcore[j];
					for(int k = 0; k < data->inRowOffset[tempData+1]-data->inRowOffset[tempData]; k++)
					{
						int adjData = data->inColOffset[data->inRowOffset[tempData]+k];
						if(din[adjData] < 0 || dcore[adjData] >= 0)
						{
							continue;
						}
						cans.push_back(adjData);
					}
					break;
				}
			}/*
			int allCansCount = 0;
	                for(int j = 0; j < dsize; ++j)
        	        {
                	        if(din[j] < 0 || dcore[j] >= 0)
                	        {
                        	        continue;
                        	}
                        	allCansCount ++;
                	}
                	printf("allCans is %d,cans infact is %d,%d less\n",allCansCount,cans.size(),allCansCount - cans.size());
			*/
			break;
		}
	}

	if(cans.size() == 0 && qnid != -1)
		return ;

	//consider the other queue
	if(qnid == -1)
	{
		//TODO:
		// add restrictions to the range of qid
		for(i = 0; i <= queryIdRange; ++i)
		{
			if(qin[matchOrder[i]] >= 0 || qout[matchOrder[i]] >= 0)
			{
				continue;
			}
			this->qnid = matchOrder[i];
			for(j = 0; j < dsize; ++j)
			{
				if(din[j] >= 0 || dout[j] >= 0)
				{
					continue;
				}
				cans.push_back(j);
			}
			break;
		}
	}

	int size = cans.size(), qnid2, dnid2;
	for(i = 0; i < size; ++i)
	{
		this->dnid = cans[i];
	//	 printf("the num is %d, the qnid is %d, the dnid is %d\n",num,this->qnid,this->dnid);
		if(prune())
		{
#ifdef DEBUG
			cerr<<"prune dnid: "<<dnid<<endl;
#endif
			continue;
		}
		else
		{
#ifdef DEBUG
			cerr<<"not prune dnid: "<<dnid<<endl;
#endif
		}
		update(num);
		//BETTER?:place qnid and dnid in function parameters?
		qnid2 = this->qnid; dnid2 = this->dnid;
		//TODO: add constraintions on when to launch kernels
		if(num >= this->minCoverPos)
		{
			int leftQueryNode = qsize - num - 1;
			int *cansStartPoint[leftQueryNode];
			int cansLenth[leftQueryNode];
			memset(cansLenth,0,sizeof(int)*leftQueryNode);
			//a reverseTable from qid to pos in cans
			int reverseTable[qsize];
			memset(reverseTable,-1,sizeof(int)*qsize);
			int fillNum = 0;
			for(int i = 0; i < qsize; i ++)
			{
				if(qcore[i] == -1)
				{
					reverseTable[i] = fillNum ++;
				}
			}
			//find the cans of all node left
			for(int k = 0; k <= minCoverPos; k ++)
			{
				//j represent the qid whose cans is to be find out
				for(int j = 0; j < qsize; j ++)
				{
					if(qcore[j] != -1 || cansLenth[reverseTable[j]] != 0)
						continue;
					if(egdeExistJudge(query->inColOffset + query->inRowOffset[matchOrder[k]],
									  query->inValues + query->inRowOffset[matchOrder[k]],
									  query->inRowOffset[matchOrder[k]+1] - query->inRowOffset[matchOrder[k]],
									  j) != -1)
					{
						cansStartPoint[reverseTable[j]] = data->inColOffset + data->inRowOffset[qcore[matchOrder[k]]];
						cansLenth[reverseTable[j]] = data->inRowOffset[qcore[matchOrder[k]]+1] - data->inRowOffset[qcore[matchOrder[k]]];
					}else if(egdeExistJudge(query->outColOffset + query->outRowOffset[matchOrder[k]],
									  query->outValues + query->outRowOffset[matchOrder[k]],
									  query->outRowOffset[matchOrder[k]+1] - query->outRowOffset[matchOrder[k]],
									  j) != -1)
					{
						cansStartPoint[reverseTable[j]] = data->outColOffset + data->outRowOffset[qcore[matchOrder[k]]];
						cansLenth[reverseTable[j]] = data->outRowOffset[qcore[matchOrder[k]]+1] - data->outRowOffset[qcore[matchOrder[k]]];
					}
				}
			}
			bool debug = false;
		/*	if(qcore[0] == 4608){
				debug = true;
				for(int l = 0; l < leftQueryNode; l ++)
				{
					printf("the len of cans[%d] is %d\n",l,cansLenth[l]);
					for(int p = 0; p < cansLenth[l]; p ++)
						printf("%d\n",cansStartPoint[l][p]);
				}
			}*/
			//calculate the total nums of threads to be launched
			int totalThreadNum = 1;
			int totalCansNum = 0;
			for(int k = 0; k < leftQueryNode; k ++)
			{
				totalThreadNum *= cansLenth[k];
				totalCansNum += cansLenth[k];
			}
			if(totalThreadNum == 0)
			{
				this->qnid = qnid2; this->dnid = dnid2;
				restore(num);	
				continue;
			}
		//	if(debug)
		//		printf("the totalThreadNum of target kernel is %d\n",totalThreadNum);

			//NOTICE:UVA technology
			if(totalThreadNum < MAXLAUNCHEDTHREADS)
			{
				streamNum = (streamNum+1)%MAXSTREAM;
				//copy the needed data to device
				int * h_toDevive;
				//TODO: put this in stream???
				unsigned int flags = hipHostMallocMapped;
				hipHostAlloc((void **)&h_toDevive,sizeof(int)*(qsize+totalCansNum+leftQueryNode+1),flags);

			//	hipHostMalloc((void **)&h_toDevive,sizeof(int)*(qsize+totalCansNum+leftQueryNode+1));
				memcpy(h_toDevive,qcore,qsize*sizeof(int));
				int * tempPtr = h_toDevive + qsize;
				tempPtr[0] = 0;
				for(int k = 1; k <= leftQueryNode; k ++)
				{
					tempPtr[k] = tempPtr[k-1] + cansLenth[k-1];
				}
				int * tempPtr2 = tempPtr + (leftQueryNode + 1);
				for(int k = 0; k < leftQueryNode; k ++)
				{
					memcpy(tempPtr2 + tempPtr[k],cansStartPoint[k],cansLenth[k]*sizeof(int));
				}

				int * d_toDevice;
				//NOTICE: the critical point of UVA technology
				hipHostGetDevicePointer((void **)&d_toDevice, (void *)h_toDevive, 0);
			//	hipMalloc(&d_toDevice, sizeof(int)*(qsize+totalCansNum+leftQueryNode+1));

			//	hipMemcpyAsync(d_toDevice,h_toDevive,sizeof(int)*(qsize+totalCansNum+leftQueryNode+1),hipMemcpyHostToDevice,stream[streamNum%MAXSTREAM]);
				
		//		printf("prepare work for launch the kernel is done\n");
		//		printf("the totalLaunched threads is %d\n",totalThreadNum);
				//TODO: launch kernel
				//TODO: only launch one block???
				if(debug)
					printf("the kernel is to be launched, the total lauchedThread is %d,streamNum is %d\n",totalThreadNum,streamNum);
				if(totalThreadNum <= MAXTHREADSPERBLOCK){
					//if(totalThreadNum != 0)
						lastJoin<<<1,totalThreadNum,0,stream[streamNum%MAXSTREAM]>>>(d_toDevice,qsize,leftQueryNode,dMatch,totalThreadNum);
				}else{
					
			//		printf("muti-blocks are launched,and the grid size is %d\n",(totalThreadNum + MAXTHREADSPERBLOCK-1)/MAXTHREADSPERBLOCK);	
					lastJoin<<<(totalThreadNum+MAXTHREADSPERBLOCK-1)/MAXTHREADSPERBLOCK,MAXTHREADSPERBLOCK,0,stream[streamNum%MAXSTREAM]>>>(d_toDevice,qsize,leftQueryNode,dMatch,totalThreadNum);
					if(debug){
						hipError_t error = hipGetLastError();
						printf("CUDA error: %s\n", hipGetErrorString(error));
					}
				}//TODO: use stream to free the memory ????
			//	printf("the total cans is %d\n",cansRowOffset[leftQueryNode]);
		//		if(streamNum == MAXSTREAM -1)
			//	hipDeviceSynchronize();
	//				hipHostFree(h_toDevive);
		//		hipFree(d_toDevice);
			}else{
				//TODO:
				//use cpu to handle the node left
				printf("too large totalcans: %d to launch the kernel,so continue on CPU\n",totalThreadNum);

				dfs(num+1, io, qsize-1, dMatch);
			}

		}else{
			dfs(num+1, io, queryIdRange, dMatch);
		}
		this->qnid = qnid2; this->dnid = dnid2;
		restore(num);
	}
}

