#include "hip/hip_runtime.h"
/*=============================================================================
# Filename: test.cu
# Author: Bookug Lobert 
# Mail: zengli-bookug@pku.edu.cn
# Last Modified: 2018-07-29 20:28
# Description: 
https://blog.csdn.net/dreampursue/article/details/6256426#
=============================================================================*/

/*
 * Copyright 1993-2011 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
/*
 * This sample demonstrates a combination of Peer-to-Peer (P2P) and Unified
 * Virtual Address Space (UVA) features new to SDK 4.0
 */
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
const char *sSDKsample = "simpleP2P";
__global__ void SimpleKernel(float *src, float *dst)
{
    // Just a dummy kernel, doing enough for us to verify that everything
    // worked
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = src[idx] * 2.0f;
}
int main(int argc, char **argv)
{
    printf("[%s] starting.../n", sSDKsample);
    // Number of GPUs
    printf("Checking for multiple GPUs.../n");
    int gpu_n;
    cutilSafeCall(hipGetDeviceCount(&gpu_n));
    printf("CUDA-capable device count: %i/n", gpu_n);
    if (gpu_n < 2)
    {
        printf("Two or more Tesla(s) with (SM 2.0) class GPUs are required for %s./n", sSDKsample);
        printf("Waiving test./n");
        printf("PASSED/n");
        exit(EXIT_SUCCESS);
    }
    // Query device properties
    hipDeviceProp_t prop_0, prop_1;
    cutilSafeCall(hipGetDeviceProperties(∝_0, 0));
    cutilSafeCall(hipGetDeviceProperties(∝_1, 1));
    // Check for TCC
#ifdef _WIN32
    if (prop_0.tccDriver == 0 || prop_1.tccDriver == 0)
    {
        printf("Need to have both GPUs running under TCC driver to use P2P / UVA functionality./n");
        printf("PASSED/n");
        exit(EXIT_SUCCESS);
    }
#endif // WIN32
    // Check possibility for peer access
    printf("Checking for peer access.../n");
    int can_access_peer_0_1, can_access_peer_1_0;
    cutilSafeCall(hipDeviceCanAccessPeer(&can_access_peer_0_1, 0, 1));
    cutilSafeCall(hipDeviceCanAccessPeer(&can_access_peer_1_0, 1, 0));
    if (can_access_peer_0_1 == 0 || can_access_peer_1_0 == 0)
    {
        printf("Two or more Tesla(s) with (SM 2.0) class GPUs are required for %s./n", sSDKsample);
        printf("Peer access is not available between GPU0 <-> GPU1, waiving test./n");
        printf("PASSED/n");
        exit(EXIT_SUCCESS);
    }
    // Enable peer access
    printf("Enabling peer access.../n");
    cutilSafeCall(hipSetDevice(0));
    cutilSafeCall(hipDeviceEnablePeerAccess(1, 0));
    cutilSafeCall(hipSetDevice(1));
    cutilSafeCall(hipDeviceEnablePeerAccess(0, 0));
    // Check that we got UVA on both devices
    printf("Checking for UVA.../n");
    const bool has_uva = prop_0.unifiedAddressing && prop_1.unifiedAddressing;
    if (has_uva == false)
    {
        printf("At least one of the two GPUs has no UVA support/n");
    }
    // Allocate buffers
    const size_t buf_size = 1024 * 1024 * 16 * sizeof(float);
    printf("Allocating buffers (%iMB on GPU0, GPU1 and Host).../n", int(buf_size / 1024 / 1024));
    cutilSafeCall(hipSetDevice(0));
    float* g0;
    cutilSafeCall(hipMalloc(&g0, buf_size));
    cutilSafeCall(hipSetDevice(1));
    float* g1;
    cutilSafeCall(hipMalloc(&g1, buf_size));
    float* h0;
    if (has_uva)
        cutilSafeCall(hipHostMalloc(&h0, buf_size)); // Automatically portable with UVA
    else
        cutilSafeCall(hipHostAlloc(&h0, buf_size, hipHostMallocPortable));
    float *g0_peer, *g1_peer;
    if (has_uva == false)
    {
        // Need explicit mapping without UVA
        cutilSafeCall(hipSetDevice(0));
        cutilSafeCall(cudaPeerRegister(g1, 1, cudaPeerRegisterMapped));
        cutilSafeCall(cudaPeerGetDevicePointer((void **) &g1_peer, g1, 1, 0));
        cutilSafeCall(hipSetDevice(1));
        cutilSafeCall(cudaPeerRegister(g0, 0, cudaPeerRegisterMapped));
        cutilSafeCall(cudaPeerGetDevicePointer((void **) &g0_peer, g0, 0, 0));
    }
    // Create CUDA event handles
    printf("Creating event handles.../n");
    hipEvent_t start_event, stop_event;
    float time_memcpy;
    int eventflags = hipEventBlockingSync;
    cutilSafeCall(hipEventCreateWithFlags(&start_event, eventflags));
    cutilSafeCall(hipEventCreateWithFlags(&stop_event, eventflags));
    // P2P memcopy() benchmark
    cutilSafeCall(hipEventRecord(start_event, 0));
    for (int i=0; i<100; i++)
    {
        // With UVA we don't need to specify source and target devices, the
        // runtime figures this out by itself from the pointers
        if (has_uva)
        {
            // Ping-pong copy between GPUs
            if (i % 2 == 0)
                cutilSafeCall(hipMemcpy(g1, g0, buf_size, hipMemcpyDefault));
            else
                cutilSafeCall(hipMemcpy(g0, g1, buf_size, hipMemcpyDefault));
        }
        else
        {
            // Ping-pong copy between GPUs
            if (i % 2 == 0)
                cutilSafeCall(hipMemcpyPeer(g1, 1, g0, 0, buf_size));
            else
                cutilSafeCall(hipMemcpyPeer(g0, 0, g1, 1, buf_size));
        }
    }
    cutilSafeCall(hipEventRecord(stop_event, 0));
    cutilSafeCall(hipEventSynchronize(stop_event));
    cutilSafeCall(hipEventElapsedTime(&time_memcpy, start_event, stop_event));
    printf("hipMemcpyPeer / hipMemcpy between GPU0 and GPU1: %.2fGB/s/n",
        (1.0f / (time_memcpy / 1000.0f)) * ((100.0f * buf_size)) / 1024.0f / 1024.0f / 1024.0f);
 
    // Prepare host buffer and copy to GPU 0
    printf("Preparing host buffer and memcpy to GPU0.../n");
    for (int i=0; i<buf_size / sizeof(float); i++)
    {
        h0[i] = float(i % 4096);
    }
    cutilSafeCall(hipSetDevice(0));
    if (has_uva)
        cutilSafeCall(hipMemcpy(g0, h0, buf_size, hipMemcpyDefault));
    else
        cutilSafeCall(hipMemcpy(g0, h0, buf_size, hipMemcpyHostToDevice));
    // Kernel launch configuration
    const dim3 threads(512, 1);
    const dim3 blocks((buf_size / sizeof(float)) / threads.x, 1);
 
    // Run kernel on GPU 1, reading input from the GPU 0 buffer, writing
    // output to the GPU 1 buffer
    printf("Run kernel on GPU1, taking source data from GPU0 and writing to GPU1.../n");
    cutilSafeCall(hipSetDevice(1));
    if (has_uva)
        SimpleKernel<<<blocks, threads>>> (g0, g1);
    else
        SimpleKernel<<<blocks, threads>>> (g0_peer, g1);
    // Run kernel on GPU 0, reading input from the GPU 1 buffer, writing
    // output to the GPU 0 buffer
    printf("Run kernel on GPU0, taking source data from GPU1 and writing to GPU0.../n");
    cutilSafeCall(hipSetDevice(0));
    if (has_uva)
        SimpleKernel<<<blocks, threads>>> (g1, g0);
    else
        SimpleKernel<<<blocks, threads>>> (g1_peer, g0);
 
    // Copy data back to host and verify
    printf("Copy data back to host from GPU0 and verify.../n");
    if (has_uva)
        cutilSafeCall(hipMemcpy(h0, g0, buf_size, hipMemcpyDefault));
    else
        cutilSafeCall(hipMemcpy(h0, g0, buf_size, hipMemcpyHostToDevice));
    int error_count = 0;
    for (int i=0; i<buf_size / sizeof(float); i++)
    {
        // Re-generate input data and apply 2x '* 2.0f' computation of both
        // kernel runs
        if (h0[i] != float(i % 4096) * 2.0f * 2.0f)
        {
            printf("Verification error, element %i/n", i);
            if (error_count++ > 10)
                break;
        }
    }
    printf((error_count == 0) ? "PASSED/n" : "FAILED/n");
    // Disable peer access (also unregisters memory for non-UVA cases)
    printf("Enabling peer access.../n");
    cutilSafeCall(hipSetDevice(0));
    cutilSafeCall(hipDeviceDisablePeerAccess(1));
    cutilSafeCall(hipSetDevice(1));
    cutilSafeCall(hipDeviceDisablePeerAccess(0));
    // Cleanup and shutdown
    printf("Shutting down.../n");
    cutilSafeCall(hipEventDestroy(start_event));
    cutilSafeCall(hipEventDestroy(stop_event));
    cutilSafeCall(hipSetDevice(0));
    cutilSafeCall(hipFree(g0));
    cutilSafeCall(hipSetDevice(1));
    cutilSafeCall(hipFree(g1));
    cutilSafeCall(hipHostFree(h0));
    hipDeviceReset();
    cutilExit(argc, argv);
}

