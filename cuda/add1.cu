#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"			//CUDA运行时API
#include ""	
#include <stdio.h>
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    // Add vectors in parallel.
    hipError_t cudaStatus;
	int num = 0;
	hipDeviceProp_t prop;
	cudaStatus = hipGetDeviceCount(&num);
	for(int i = 0;i<num;i++)
	{
		hipGetDeviceProperties(&prop,i);
	}
	cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",c[0],c[1],c[2],c[3],c[4]);
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}
// 重点理解这个函数
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size)
{
    int *dev_a = 0;	//GPU设备端数据指针
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;		//状态指示

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);	//选择运行平台
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    // 分配GPU设备端内存
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // 拷贝数据到GPU
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // 运行核函数
	addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();	//同步线程
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);		//拷贝结果回主机
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
Error:
    hipFree(dev_c);	//释放GPU设备端内存
    hipFree(dev_a);
    hipFree(dev_b);    
    return cudaStatus;
}
