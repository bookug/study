//nvGRAPH Traversal example

#include <nvgraph.h>
#include <stdio.h>

void check_status(nvgraphStatus_t status){
    if ((int)status != 0)    {
        printf("ERROR : %d\n",status);
        exit(0);
    }
}
int main(int argc, char **argv){
    //Example of graph (CSR format)
    const size_t  n = 7, nnz = 12, vertex_numsets = 2, edge_numset = 0;
    int source_offsets_h[] = {0, 1, 3, 4, 6, 8, 10, 12};
    int destination_indices_h[] = {5, 0, 2, 0, 4, 5, 2, 3, 3, 4, 1, 5};
    //where to store results (distances from source) and where to store results (predecessors in search tree) 
    int bfs_distances_h[n], bfs_predecessors_h[n];
    // nvgraph variables
    nvgraphStatus_t status;
    nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSRTopology32I_t CSR_input;
    hipDataType* vertex_dimT;
    size_t distances_index = 0;
    size_t predecessors_index = 1;
    vertex_dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
    vertex_dimT[distances_index] = HIP_R_32I;
    vertex_dimT[predecessors_index] = HIP_R_32I;
    //Creating nvgraph objects
    check_status(nvgraphCreate (&handle));
    check_status(nvgraphCreateGraphDescr (handle, &graph));
    // Set graph connectivity and properties (tranfers)
    CSR_input = (nvgraphCSRTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    CSR_input->nvertices = n;
    CSR_input->nedges = nnz;
    CSR_input->source_offsets = source_offsets_h;
    CSR_input->destination_indices = destination_indices_h;
    check_status(nvgraphSetGraphStructure(handle, graph, (void*)CSR_input, NVGRAPH_CSR_32));
    check_status(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    int source_vert = 1;
    //Setting the traversal parameters  
    nvgraphTraversalParameter_t traversal_param;
    nvgraphTraversalParameterInit(&traversal_param);
    nvgraphTraversalSetDistancesIndex(&traversal_param, distances_index);
    nvgraphTraversalSetPredecessorsIndex(&traversal_param, predecessors_index);
    nvgraphTraversalSetUndirectedFlag(&traversal_param, false);
    //Computing traversal using BFS algorithm
    check_status(nvgraphTraversal(handle, graph, NVGRAPH_TRAVERSAL_BFS, &source_vert, traversal_param));
    // Get result
    check_status(nvgraphGetVertexData(handle, graph, (void*)bfs_distances_h, distances_index));
    check_status(nvgraphGetVertexData(handle, graph, (void*)bfs_predecessors_h, predecessors_index));
    // expect bfs distances_h = (1 0 1 3 3 2 2147483647)
    for (int i = 0; i<n; i++)  printf("Distance to vertex %d: %i\n",i, bfs_distances_h[i]); printf("\n");
    // expect bfs predecessors = (1 -1 1 5 5 0 -1)
    for (int i = 0; i<n; i++)  printf("Predecessor of vertex %d: %i\n",i, bfs_predecessors_h[i]); printf("\n");
    free(vertex_dimT);
    free(CSR_input);
    check_status(nvgraphDestroyGraphDescr (handle, graph));
    check_status(nvgraphDestroy (handle));
    return 0;
}

