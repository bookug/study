//nvGRAPH convert topology example

#include <nvgraph.h>
#include <stdio.h>

void check(nvgraphStatus_t status) {
    if (status != NVGRAPH_STATUS_SUCCESS) {
        printf("ERROR : %d\n",status);
        exit(0);
    }
}
int main(int argc, char **argv) {
    size_t  n = 6, nnz = 10;
    // nvgraph variables
    nvgraphHandle_t handle;
    nvgraphCSCTopology32I_t CSC_input;
    nvgraphCSRTopology32I_t CSR_output;
    float *src_weights_d, *dst_weights_d;
    hipDataType edge_dimT = HIP_R_32F;
    // Allocate source data
    CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    CSC_input->nvertices = n; CSC_input->nedges = nnz;
    hipMalloc( (void**)&(CSC_input->destination_offsets), (n+1)*sizeof(int));
    hipMalloc( (void**)&(CSC_input->source_indices), nnz*sizeof(int));
    hipMalloc( (void**)&src_weights_d, nnz*sizeof(float));
    // Copy source data
    float src_weights_h[] = {0.333333f, 0.5f, 0.333333f, 0.5f, 0.5f, 1.0f, 0.333333f, 0.5f, 0.5f, 0.5f};
    int destination_offsets_h[] = {0, 1, 3, 4, 6, 8, 10};
    int source_indices_h[] = {2, 0, 2, 0, 4, 5, 2, 3, 3, 4};
    hipMemcpy(CSC_input->destination_offsets, destination_offsets_h, (n+1)*sizeof(int), hipMemcpyDefault);
    hipMemcpy(CSC_input->source_indices, source_indices_h, nnz*sizeof(int), hipMemcpyDefault);
    hipMemcpy(src_weights_d, src_weights_h, nnz*sizeof(float), hipMemcpyDefault);
    // Allocate destination data
    CSR_output = (nvgraphCSRTopology32I_t) malloc(sizeof(struct nvgraphCSRTopology32I_st));
    hipMalloc( (void**)&(CSR_output->source_offsets), (n+1)*sizeof(int));
    hipMalloc( (void**)&(CSR_output->destination_indices), nnz*sizeof(int));
    hipMalloc( (void**)&dst_weights_d, nnz*sizeof(float));
    // Starting nvgraph and convert
    check(nvgraphCreate (&handle));
    check(nvgraphConvertTopology(handle, NVGRAPH_CSC_32, CSC_input, src_weights_d,
        &edge_dimT, NVGRAPH_CSR_32, CSR_output, dst_weights_d));
    // Free memory
    check(nvgraphDestroy(handle));
    hipFree(CSC_input->destination_offsets);
    hipFree(CSC_input->source_indices);
    hipFree(CSR_output->source_offsets);
    hipFree(CSR_output->destination_indices);
    hipFree(src_weights_d);
    hipFree(dst_weights_d);
    free(CSC_input);
    free(CSR_output);
    return 0;
}

