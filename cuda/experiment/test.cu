#include <stdio.h>
#include <hip/hip_runtime.h> 
 
#include <hip/hip_runtime_api.h> 
#include <cassert>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    printf("CUDA error at: %s:%d\n", file, line);
    printf("%s %s\n", hipGetErrorString(err), func);
    exit(1);
  }
}

//Dynamic memory allocation in the kernel function of GPU
//https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#dynamic-global-memory-allocation-and-operations
//very slow, dozens of times slower than pre-assigned memory

//NOTICE: the limit of block number is given by compute/arch capablity
//nvcc -arch=sm_35 will allow block num >= 65536
//https://en.wikipedia.org/wiki/CUDA
//WARN: this setting will cause the kernel function unable to work
//As a result, we should set block num < 65536
#define NUM_BLOCKS 65536
/*#define NUM_BLOCKS 400*/
#define BLOCK_WIDTH 1
/*#define BLOCK_WIDTH 1024*/

__global__ void hello()
{
	/*__shared__ int array[128];*/
	printf("Hello, world! I am a thread in block %d\n", blockIdx.x);
	/*__syncthreads();*/
}

int main(int argc, const char* argv[])
{
	//NOTICE: this API is out-ofo-date
	/*cudaPrintfInit(100*NUM_BLOCKS*BLOCK_WIDTH);*/
	size_t io_buffer_size = 0;
	hipDeviceGetLimit(&io_buffer_size, hipLimitPrintfFifoSize);
	printf("io buffer size: %u\n", io_buffer_size);   //1M by default
	//NOTICE: no need to assign space for all threads' output, because there are at most 3840 threads running in parallel really
	//The former analysis is wrong!  IO buffer is flushed at the end of kernel execution, so overwriting will occur!
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 400*1024*200);
	hipDeviceGetLimit(&io_buffer_size, hipLimitPrintfFifoSize);
	printf("io buffer size: %u\n", io_buffer_size);   //1M by default

	hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
	//Below checks if the kernel launches successfully
	checkCudaErrors(hipGetLastError());

	//force the printf()s to flush
	hipDeviceSynchronize();
	//Below checks if the kernel runs and ends successfully
	checkCudaErrors(hipGetLastError());

	printf("That's all!\n");

	return 0;
}

