#include "hip/hip_runtime.h"
/*=============================================================================
# Filename: test.cu
# Author: bookug 
# Mail: bookug@qq.com
# Last Modified: 2018-10-24 19:56
# Description: 
This program tests I/O and thread capacity on GPU(using Titan X Pascal)
=============================================================================*/

#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime_api.h> 
#include <cassert>

#include "Util.h"
using namespace std; 

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    printf("CUDA error at: %s:%d\n", file, line);
    printf("%s %s\n", hipGetErrorString(err), func);
    exit(1);
  }
}

//NOTICE: the limit of block number is given by compute/arch capablity
//nvcc -arch=sm_35 will allow block num >= 65536
//https://en.wikipedia.org/wiki/CUDA
//WARN: this setting will cause the kernel function unable to work
//As a result, we should set block num < 65536
/*#define NUM_BLOCKS 65536*/

//this is ok to start
#define NUM_BLOCKS 1000000000

/*#define NUM_BLOCKS 400*/
#define BLOCK_WIDTH 1
/*#define BLOCK_WIDTH 1024*/

__global__ void hello(unsigned* d_data)
{
	/*__shared__ int array[128];*/

	/*printf("Hello, world! I am a thread in block %d\n", blockIdx.x);*/

	/*__syncthreads();*/

    //HACK: we can hack the global load/store transaction number/size here
    //Or we can explore the mechanism of register allocation.
    unsigned ele = d_data[threadIdx.x];
    ele = 2 * ele;
    d_data[threadIdx.x] = ele;
}

int main(int argc, const char* argv[])
{
	//NOTICE: this API is out-of-date
	/*cudaPrintfInit(100*NUM_BLOCKS*BLOCK_WIDTH);*/
	size_t io_buffer_size = 0;
	hipDeviceGetLimit(&io_buffer_size, hipLimitPrintfFifoSize);
	printf("io buffer size: %u\n", io_buffer_size);   //1M by default
	//NOTICE: no need to assign space for all threads' output, because there are at most 3840 threads running in parallel really
	//The former analysis is wrong!  IO buffer is flushed at the end of kernel execution, so overwriting will occur!
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 400*1024*200);
	hipDeviceGetLimit(&io_buffer_size, hipLimitPrintfFifoSize);
	printf("io buffer size: %u\n", io_buffer_size);   //1M by default

    unsigned* d_data = NULL;
    hipMalloc(&d_data, sizeof(unsigned)*32);
    /*hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();*/
    /*hello<<<1000000000L, 1024>>>();*/
    hello<<<1, 32>>>(d_data);
	//Below checks if the kernel launches successfully
	checkCudaErrors(hipGetLastError());
	//force the printf()s to flush
	hipDeviceSynchronize();
	//Below checks if the kernel runs and ends successfully
	checkCudaErrors(hipGetLastError());
    hipFree(d_data);

    //test the latency of small transfer between CPU and GPU
    /*unsigned *h_data[3];*/
    /*hipMalloc( (void **) &d_data, 3 * sizeof(unsigned));*/
    /*long t1, t2;*/
    /*int limit = 1000, tt=0;*/
    /*for(int i = 0; i < limit; ++i)*/
    /*{*/
        /*t1 = Util::get_cur_time();*/
        /*hipMemcpy(d_data, h_data, 3 * sizeof(unsigned), hipMemcpyHostToDevice);*/
        /*t2 = Util::get_cur_time();*/
        /*tt += t2-t1;*/
    /*}*/
    /*printf("transfer 12 bytes 1000 times used: %ld ms\n", tt);*/

	printf("That's all!\n");

	return 0;
}

