#include "hip/hip_runtime.h"
/*=============================================================================
# Filename: test.cu
# Author: bookug 
# Mail: bookug@qq.com
# Last Modified: 2018-10-24 19:56
# Description: 
This program tests I/O and thread capacity on GPU(using Titan X Pascal)
=============================================================================*/

#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime_api.h> 
#include <cassert>

#include "Util.h"
using namespace std; 

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    printf("CUDA error at: %s:%d\n", file, line);
    printf("%s %s\n", hipGetErrorString(err), func);
    exit(1);
  }
}

//NOTICE: the limit of block number is given by compute/arch capablity
//nvcc -arch=sm_35 will allow block num >= 65536
//https://en.wikipedia.org/wiki/CUDA
//WARN: this setting will cause the kernel function unable to work
//As a result, we should set block num < 65536
/*#define NUM_BLOCKS 65536*/

//this is ok to start
#define NUM_BLOCKS 1000000000

/*#define NUM_BLOCKS 400*/
#define BLOCK_WIDTH 1
/*#define BLOCK_WIDTH 1024*/

//Initially, this function uses 328B constant memory
//If we add more constant variables, this size will grow
__global__ void hello(unsigned* d_data)
{
    __shared__ unsigned duck;
	/*__shared__ int array[128];*/

	/*printf("Hello, world! I am a thread in block %d\n", blockIdx.x);*/

	/*__syncthreads();*/

    //HACK: we can hack the global load/store transaction number/size here
    //Or we can explore the mechanism of register allocation.
    /*unsigned ele = d_data[threadIdx.x];*/
    /*ele = 2 * ele;*/
    /*d_data[threadIdx.x] = ele;*/

    //shared variable does not occupy registers and it does not occupy gld/gst transactions
    duck = 0;

    //To hack the gst write transactions(whether write cache is needed)
    //when only use this instruction in the context, only 6 registers occupied, the same as none instructions
    /*d_data[threadIdx.x] = 0;*/
    //below will add 2 registers for usage, and the gst transactions number+4 (32 bytes once)
    /*d_data[threadIdx.x] = 1;*/

    //TODO: test multiple warps write continuously
    int idx = threadIdx.x % 32;
    int group = threadIdx.x / 32;
    //below uses 32 gst transactions
    //EXPLAIN: though the addresses written by different warps are continuous, these warps may be not run in the same time(although with sync function here)
    __syncthreads();
    if(idx == 0)
    {
        //WARN: we should not use sync function here(in judgement) because it will cause deadlock
        d_data[group] = 0;
    }
    //below uses 4 gst transactions
    /*if(group == 0)*/
    /*{*/
        /*d_data[idx] = 0;*/
    /*}*/

    //test if adoptingg 128B mechanism: -Xptxas -dlcm=ca  (close, -dlcm=cg)
    //If with no specification, below needs 8 gld transactions
    /*unsigned ele = d_data[threadIdx.x];   //this single instruction  adds 4 regsiters usage. In real running, registers usage may be more or less*/
}

int main(int argc, const char* argv[])
{
	//NOTICE: this API is out-of-date
	/*cudaPrintfInit(100*NUM_BLOCKS*BLOCK_WIDTH);*/
	size_t io_buffer_size = 0;
	hipDeviceGetLimit(&io_buffer_size, hipLimitPrintfFifoSize);
	printf("io buffer size: %u\n", io_buffer_size);   //1M by default
	//NOTICE: no need to assign space for all threads' output, because there are at most 3840 threads running in parallel really
	//The former analysis is wrong!  IO buffer is flushed at the end of kernel execution, so overwriting will occur!
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 400*1024*200);
	hipDeviceGetLimit(&io_buffer_size, hipLimitPrintfFifoSize);
	printf("io buffer size: %u\n", io_buffer_size);   //1M by default

    unsigned* d_data = NULL;
    hipMalloc(&d_data, sizeof(unsigned)*32);
    /*hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();*/
    /*hello<<<1000000000L, 1024>>>();*/
    /*hello<<<1, 32>>>(d_data);*/
    hello<<<1, 1024>>>(d_data);
	//Below checks if the kernel launches successfully
	checkCudaErrors(hipGetLastError());
	//force the printf()s to flush
	hipDeviceSynchronize();
	//Below checks if the kernel runs and ends successfully
	checkCudaErrors(hipGetLastError());
    hipFree(d_data);

    //test the latency of small transfer between CPU and GPU
    /*unsigned *h_data[3];*/
    /*hipMalloc( (void **) &d_data, 3 * sizeof(unsigned));*/
    /*long t1, t2;*/
    /*int limit = 1000, tt=0;*/
    /*for(int i = 0; i < limit; ++i)*/
    /*{*/
        /*t1 = Util::get_cur_time();*/
        /*hipMemcpy(d_data, h_data, 3 * sizeof(unsigned), hipMemcpyHostToDevice);*/
        /*t2 = Util::get_cur_time();*/
        /*tt += t2-t1;*/
    /*}*/
    /*printf("transfer 12 bytes 1000 times used: %ld ms\n", tt);*/

	printf("That's all!\n");

	return 0;
}

