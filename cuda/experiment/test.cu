#include "hip/hip_runtime.h"
/*=============================================================================
# Filename: test.cu
# Author: bookug 
# Mail: bookug@qq.com
# Last Modified: 2018-10-24 19:56
# Description: 
This program tests I/O and thread capacity on GPU(using Titan X Pascal)
=============================================================================*/

//WARN: no shared memory configs for Maxwell/Pasacal, no split of L1/shared mem
//https://stackoverflow.com/questions/52305331/how-to-call-cudadevicesetsharedmemconfig-and-cudadevicesetcacheconfig

#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime_api.h> 
#include <cassert>

#include "Util.h"
using namespace std; 

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    printf("CUDA error at: %s:%d\n", file, line);
    printf("%s %s\n", hipGetErrorString(err), func);
    exit(1);
  }
}

//NOTICE: the limit of block number is given by compute/arch capablity
//nvcc -arch=sm_35 will allow block num >= 65536
//https://en.wikipedia.org/wiki/CUDA
//WARN: this setting will cause the kernel function unable to work
//As a result, we should set block num < 65536
/*#define NUM_BLOCKS 65536*/

//this is ok to start
#define NUM_BLOCKS 1000000000

/*#define NUM_BLOCKS 400*/
#define BLOCK_WIDTH 1
/*#define BLOCK_WIDTH 1024*/

//Initially, this function uses 328B constant memory
//(this may include all constant numbers, which will also occupy registers when running)
//If we add more constant variables, this size will grow
__global__ void hello(unsigned* d_data)
{
    __shared__ unsigned duck;
	/*__shared__ int array[128];*/

	/*printf("Hello, world! I am a thread in block %d\n", blockIdx.x);*/

    if(threadIdx.x > 512)
    {
        return;
    }
    //NOTICE: for a block, if some threads has returned, we can still use __syncthreads()
    //However, these threads can not be used to finish works again
    __syncthreads();
    if(threadIdx.x == 0)
    {
        printf("after sync!\n");
    }
    if(threadIdx.x > 512)
    {
        printf("the thread still alive\n");
    }

    //HACK: we can hack the global load/store transaction number/size here
    //Or we can explore the mechanism of register allocation.
    /*unsigned ele = d_data[threadIdx.x];*/
    /*ele = 2 * ele;*/
    /*d_data[threadIdx.x] = ele;*/
    //NOTICE: below requires 4 gst transactions
    /*d_data[threadIdx.x] = 0;*/

    //shared variable does not occupy registers and it does not occupy gld/gst transactions
    duck = 0;

    //To hack the gst write transactions(whether write cache is needed)
    //when only use this instruction in the context, only 6 registers occupied, the same as none instructions
    /*d_data[threadIdx.x] = 0;*/
    //below will add 2 registers for usage, and the gst transactions number+4 (32 bytes once)
    /*d_data[threadIdx.x] = 1;*/

    //test multiple warps write continuously
    int idx = threadIdx.x % 32;
    int group = threadIdx.x / 32;
    //below uses 32 gst transactions (this should be tested using block size 1024)
    //EXPLAIN: though the addresses written by different warps are continuous, these warps may be not run in the same time(although with sync function here)
    /*__syncthreads();*/
    /*if(idx == 0)*/
    /*{*/
        /*//WARN: we should not use sync function here(in judgement) because it will cause deadlock*/
        /*d_data[group] = 0;*/
    /*}*/
    //below uses 4 gst transactions, but they are faster than 4 separated gst transactions
    /*if(group == 0)*/
    /*{*/
        /*d_data[idx] = 0;*/
    /*}*/

    //test if adoptingg 128B mechanism: -Xptxas -dlcm=ca  (close, -dlcm=cg)
    //If with no specification, below needs 8 gld transactions
    /*unsigned ele = d_data[threadIdx.x];   //this single instruction  adds 4 regsiters usage. In real running, registers usage may be more or less*/

    //below test the speed of 128B transactions and 32B transactions
    //nvcc -arch=sm_35 -lcudadevrt -rdc=true -G --ptxas-options=-v -lcurand -Xptxas -dlcm=ca test.cu -o test.exe
    //On titan xp, though L1 cache is used, still 8 gld transactions for a warp
    //nvprof -m gld_transactions -m gst_transactions ./test.exe ans.txt 3>& prof.log
    //On titan xp, though L1 cache is forbidden, still 8 gld transactions for a warp
    unsigned ele;
    //below requires 8 gld transactions
    /*ele = d_data[idx];*/
    //below requires 4 gld transactions
    /*if(idx < 2)*/
    /*{*/
        /*ele = d_data[idx*8];*/
    /*}*/
    //below requires 1 gld transactions
    /*if(idx < 2)*/
    /*{*/
        /*ele = d_data[idx];*/
    /*}*/
    //below requires 2 gld transactions
    /*if(idx < 8)*/
    /*{*/
        /*ele = d_data[idx];*/
    /*}*/
    //below requires 2 gld transactions
    //NOTICE: this claims the fact that each 4 unsigned numbers(16B) requires a gld transaction
    //The 16B size is optimized for scatter read, which is the advantage of read-only cache(texture cache)
    //The constant cache is optimized for broadcasting.
    //
    /*if(idx < 5)*/
    /*{*/
        /*ele = d_data[idx];*/
    /*}*/

    //test the read cahe
    //below uses 16 gld transactions, no read cache for time
    /*ele = d_data[idx];*/
    /*ele = d_data[idx];*/
    //test the write cache
    //below uses 8 gld transactions, no write cache for time
    /*d_data[idx] = 0;*/
    /*d_data[idx] = 0;*/

    //test broadcasting
    //below also uses 4 gld transactions?
    ele = d_data[0];

    //test overlapping
    //below requires 4 transactions, which seems reads of each 4 threads are combined into a small group and small groups may be not executed at the same time
    /*ele = d_data[idx%4];*/

    //test writing to the same address
    //below uses 1 gst transaction
    d_data[0] = 0;

    //TODO:test relations among gld transactions, throughput and efficiency
    //if one 128B read is better then 4 separated 32B reads?
    //TODO: test the transactions when using constant memory instead of registers
    //the speed of 32 threads writing to the same address in shared mem, compared with only one
    //TODO: compare gld and dram transactions
    //TODO: test the efficiency of one thread do a single transaction, save vs not-saved
    //TODO:test the transactions of read/write by memcpy with a single thread
    //TODO:test the speed of CudaMemcpy, memcpy in kernel and multiple threads copying

    //TODO: test the efficiency of load balance among warps(combine the tasks of different warps)
    //TODO: tets the tiem cost of adding unnecessary __syncthreads calls
}

__global__ void
spill_kernel(unsigned* ptr0, unsigned* ptr1)
{
    //https://stackoverflow.com/questions/12167926/forcing-cuda-to-use-register-for-a-variable?answertab=active#tab-top
    int i = threadIdx.x;
    int idx = threadIdx.x % 32;
    int group = threadIdx.x / 32;
    unsigned x = i;
    unsigned y = x * x;
    unsigned z = y * y;
    unsigned p = x + y + z;
    unsigned q = p * p;
    unsigned r = q * q;
    //NOTICE: the compiler will place the below content in local memory
    /*unsigned xxx[30];*/
    unsigned xxx[9];
    xxx[7] = 18;
    unsigned myid = 9;  //this inst not change register and constant memory usage
    unsigned offset = idx * idx;  //this inst adds one register
    unsigned p0 = i;   //this inst not adds register
    //insts below not add registers
    unsigned p1 = i * i;
    unsigned p2 = i * i * i;
    unsigned p3 = group * group;

    unsigned f0 = 100; //no adds
    unsigned f1 = f0 * f0;  //no adds
    //QUERY: if 10 is a boottleneck? later all will be palced in local memory?
}

int main(int argc, const char* argv[])
{
	//NOTICE: this API is out-of-date
	/*cudaPrintfInit(100*NUM_BLOCKS*BLOCK_WIDTH);*/
	size_t io_buffer_size = 0;
	hipDeviceGetLimit(&io_buffer_size, hipLimitPrintfFifoSize);
	printf("io buffer size: %u\n", io_buffer_size);   //1M by default
	//NOTICE: no need to assign space for all threads' output, because there are at most 3840 threads running in parallel really
	//The former analysis is wrong!  IO buffer is flushed at the end of kernel execution, so overwriting will occur!
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 400*1024*200);
	hipDeviceGetLimit(&io_buffer_size, hipLimitPrintfFifoSize);
	printf("io buffer size: %u\n", io_buffer_size);   //1M by default

    unsigned* d_data = NULL;
    hipMalloc(&d_data, sizeof(unsigned)*32);
    /*hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();*/
    /*hello<<<1000000000L, 1024>>>();*/
    /*hello<<<1, 32>>>(d_data);*/
    hello<<<1, 1024>>>(d_data);
	//Below checks if the kernel launches successfully
	checkCudaErrors(hipGetLastError());
	//force the printf()s to flush
	hipDeviceSynchronize();
	//Below checks if the kernel runs and ends successfully
	checkCudaErrors(hipGetLastError());
    hipFree(d_data);

    //test register spill
    unsigned ptr[10];
    spill_kernel<<<1,32>>>(ptr+0, ptr+1);

    //test the latency of small transfer between CPU and GPU
    /*unsigned *h_data[3];*/
    /*hipMalloc( (void **) &d_data, 3 * sizeof(unsigned));*/
    /*long t1, t2;*/
    /*int limit = 1000, tt=0;*/
    /*for(int i = 0; i < limit; ++i)*/
    /*{*/
        /*t1 = Util::get_cur_time();*/
        /*hipMemcpy(d_data, h_data, 3 * sizeof(unsigned), hipMemcpyHostToDevice);*/
        /*t2 = Util::get_cur_time();*/
        /*tt += t2-t1;*/
    /*}*/
    /*printf("transfer 12 bytes 1000 times used: %ld ms\n", tt);*/

	printf("That's all!\n");

	return 0;
}

