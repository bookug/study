#include "hip/hip_runtime.h"
/*=============================================================================
# Filename: memory.cu
# Author: bookug 
# Mail: bookug@qq.com
# Last Modified: 2018-10-24 01:07
# Description: 
warp operations are very fast and important because threads in a warp are naturally synchronized
束表决函数：简单的理解就是在一个warp内进行表决
__all(int predicate)：指的是predicate与0进行比较，如果当前线程所在的Wrap所有线程对应predicate不为0，则返回1。
__any(int predicate)：指的是predicate与0进行比较，如果当前线程所在的Wrap有一个线程对应的predicate值不为0，则返回1。
__ballot(int predicate)：指的是当前线程所在的Wrap中第N个线程对应的predicate值不为0，则将整数0的第N位进行置位。

//__shfl __shfl_up  ...
//https://blog.csdn.net/bruce_0712/article/details/64926471
//NOTICE: in __shfl_up or other functions, the thread not in range will receive itself!!!

置位可以用或操作符“|”实现：y = x | (1 << n)  对x的第n位进行置位
清楚可以用与操作符”&“实现：y = x & (~(1 << n))
取反可以用异或操作符”^“实现： y = x ^ (1 << n)
Bit提取操作： bit = (x & (1 << n)) >> n

如何判断一个数是否为2的整数次幂   x & (x-1) == 0
如何提取一个数的最低位的1     x & (-x)
已知一个数是2的幂，如何提取幂次    可以使用log，有更好的方法么？
C语言的log函数耗时钟周期多，且取的是自然对数，要取以2为底的对数还得转化。(or use log2 function)
一种比较好的方式是右移位运算，做二分查找，最多只需五次
=============================================================================*/

#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime_api.h> 
#include <cassert>
#include <hiprand.h> 

#include "Util.h" 

using namespace std;

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)
#define ERROR_EXIT -1
#define NUM_BLOCKS 1
#define BLOCK_WIDTH 32

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    printf("CUDA error at: %s:%d\n", file, line);
    printf("%s %s\n", hipGetErrorString(err), func);
    exit(1);
  }
}
/* Check the return value of CUDA Runtime API */
#define CHECK_CUDA(err) do{\
    if((err) != hipSuccess){\
            fprintf(stderr, "CUDA Runtime API error %d at file %s line %d: %s.\n",\
                                                   (int)(err), __FILE__, __LINE__, hipGetErrorString((err)));\
            exit(ERROR_EXIT);\
        }}while(0)

/* Check the return value of CURAND api. */
#define CHECK_CURAND(err) do{\
    if( (err) != HIPRAND_STATUS_SUCCESS  ){\
            fprintf(stderr, "CURAND error %d at file %s line %d.\n", (int)(err), __FILE__, __LINE__);\
        exit(ERROR_EXIT);\
        }}while(0)

void initGPU(int dev)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    hipSetDevice(dev);
	//NOTE: 48KB shared memory per block, 1024 threads per block, 30 SMs and 128 cores per SM
    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %luB; compute v%d.%d; clock: %d kHz; shared mem: %dB; block threads: %d; SM count: %d\n",
               devProps.name, devProps.totalGlobalMem, 
               (int)devProps.major, (int)devProps.minor, 
               (int)devProps.clockRate,
			   devProps.sharedMemPerBlock, devProps.maxThreadsPerBlock, devProps.multiProcessorCount);
    }
	cout<<"GPU selected"<<endl;
	//GPU initialization needs several seconds, so we do it first and only once
	//https://devtalk.nvidia.com/default/topic/392429/first-cudamalloc-takes-long-time-/
	int* warmup = NULL;
	/*unsigned long bigg = 0x7fffffff;*/
	/*hipMalloc(&warmup, bigg);*/
	/*cout<<"warmup malloc"<<endl;*/
	hipMalloc(&warmup, sizeof(int));
	hipFree(warmup);
	cout<<"GPU warmup finished"<<endl;
	unsigned long size = 0x7fffffff;
	/*size *= 3;   //heap corruption for 3 and 4*/
	size *= 2;
    //NOTICE: it is ok to expand the heap memory capacity to 8G if using long type
    //However, it is not suggested because the remaining memory is too small
    //In fact, the program not ends if using 8G heap memory
    //Once you set the memory limit, then you will see the limit occupation if using nvidia-smi command to see, but it not really occupy so much at once
    /*size *= 2;*/
	//NOTICE: the memory alloced by hipMalloc is different from the GPU heap(for new/malloc in kernel functions)
	hipDeviceSetLimit(hipLimitMallocHeapSize, size);
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
	cout<<"check heap limit: "<<size<<endl;

	// Runtime API
	// hipFuncCachePreferShared: shared memory is 48 KB
	// hipFuncCachePreferEqual: shared memory is 32 KB
	// hipFuncCachePreferL1: shared memory is 16 KB
	// hipFuncCachePreferNone: no preference
	/*hipFuncSetCacheConfig(reinterpret_cast<const void*>(MyKernel), hipFuncCachePreferShared)*/
	//The initial configuration is 48 KB of shared memory and 16 KB of L1 cache
	//The maximum L2 cache size is 3 MB.
	//also 48 KB read-only cache: if accessed via texture/surface memory, also called texture cache;
	//or use _ldg() or const __restrict__
	//64KB constant memory, ? KB texture memory. cache size?
	//CPU的L1 cache是根据时间和空间局部性做出的优化，但是GPU的L1仅仅被设计成针对空间局部性而不包括时间局部性。频繁的获取L1不会导致某些数据驻留在cache中，只要下次用不到，直接删。
	//L1 cache line 128B, L2 cache line 32B, notice that load is cached while store not
	//mmeory read/write is in unit of a cache line
	//the word size of GPU is 32 bits
}

__global__ void warp_kernel()
{
    __shared__ int vote;
    //this is also called a lane
    int idx = threadIdx.x & 0x1f;
    //USAGE: the function of vote control can be easily implemented
    vote = idx;
    __syncthreads();
    //NOTICE: bit operations are suggested instead of / and %, which are really costly
    int warp = threadIdx.x >> 5;
    if(idx == 0)
    {
        printf("thread %d control the warp %d\n", vote, warp);
    }

    //NOTICE: the return value of __ballot, __any, __all
    int flag = 0;
    if(idx < 15)
    {
        flag = 3;
    }
    int check = __shfl_up(flag, 4);
    //NOTICE: it is not safe for thread 0 because it will receive 3 as result, not the former 4-th one
    //because 0-4 is out of bound, so it will use the value of itself instead
    if(idx == 0)
    {
        printf("check: %d\n", check);
    }
    int t1 = __any(flag);
    int t2 = __all(flag);
    unsigned t3 = __ballot(flag);
    //WARN: thsi may cause problem, the return value of __ballot may be 1<<31, i.e. 2147483648 which exceeds the maximum integer value
    /*int t3 = __ballot(flag);*/

    if(idx == 0)
    {
        printf("check: %d %d %u\n",t1, t2, t3);
        //logarithmic function may be not accurate and can cause problem
        int x = log2((double)16777216);  //this is wrong, it should be 24, but it outputs 23
        /*int x = log2((double)1);  // this is right, 0*/
        printf("x: %d\n", x);
    }

    //use shfl to do warp prefix-scan
    int val = 0;
    if(idx == 0 || idx == 7 || (idx>1 &&idx<6))
    {
        val = 1;
    }
    unsigned size = 8;
    //no need to use 32 here, we can stop when we count what we need
    for(unsigned stride = 1; stride <= size; stride<<=1)
    {
        //WARN: this logic is wrong because the 0-th thread will not send its value to the 1-th thread
        /*if(idx >= stride)*/
        /*{*/
            /*int tmp = __shfl_up(val, stride, 32);*/
            /*val += tmp;*/
        /*}*/
        int tmp = __shfl_up(val, stride);
        //NOTICE: for prefix we must do this judgement, but no need for reduce-sum
        if(idx >= stride)
        {
            val += tmp;
        }
    }
}

int main(int argc, const char* argv[])
{
	int dev = 0;
	if(argc == 2)
	{
		dev = atoi(argv[1]);
	}
    initGPU(dev);

    //this should be -2,  x的补码是-x，等于反码加1，所以负数在计算机中可以直接用补码表示，因此得到统一
    cout<<~1<<endl;
    unsigned tmp = false;
    cout<<tmp<<endl;
    tmp = true;
    cout<<tmp<<endl;

    long t1 = Util::get_cur_time();

    warp_kernel<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
	//force the printf()s to flush
	hipDeviceSynchronize();
	//Below checks if the kernel runs and ends successfully
	checkCudaErrors(hipGetLastError());

    long t2 = Util::get_cur_time();
    printf("warp_kernel used %lu ms\n", t2 - t1);

	printf("That's all!\n");

	return 0;
}

