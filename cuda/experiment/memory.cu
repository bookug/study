#include "hip/hip_runtime.h"
/*=============================================================================
# Filename: memory.cu
# Author: bookug 
# Mail: bookug@qq.com
# Last Modified: 2018-10-24 01:07
# Description: 
verify the efficiency of dynamic memory allocation scheduling
=============================================================================*/

#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime_api.h> 
#include <cassert>
#include <hiprand.h> 

#include "Util.h" 

using namespace std;

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)
#define ERROR_EXIT -1
/*#define NUM_BLOCKS 1*/
#define NUM_BLOCKS 1000
#define BLOCK_WIDTH 1024
#define BASE 1000
#define LIMIT 900

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    printf("CUDA error at: %s:%d\n", file, line);
    printf("%s %s\n", hipGetErrorString(err), func);
    exit(1);
  }
}
/* Check the return value of CUDA Runtime API */
#define CHECK_CUDA(err) do{\
    if((err) != hipSuccess){\
            fprintf(stderr, "CUDA Runtime API error %d at file %s line %d: %s.\n",\
                                                   (int)(err), __FILE__, __LINE__, hipGetErrorString((err)));\
            exit(ERROR_EXIT);\
        }}while(0)

/* Check the return value of CURAND api. */
#define CHECK_CURAND(err) do{\
    if( (err) != HIPRAND_STATUS_SUCCESS  ){\
            fprintf(stderr, "CURAND error %d at file %s line %d.\n", (int)(err), __FILE__, __LINE__);\
        exit(ERROR_EXIT);\
        }}while(0)

extern "C"
void randomGenerator(float *dataHost, int number, unsigned long long seed)
{   
    //SEE: https://blog.csdn.net/warren912/article/details/19962823
    float *dataDev;
    CHECK_CUDA( hipMalloc( (void **) &dataDev, number * sizeof(float)  )  );
 
    hiprandGenerator_t gen;
    CHECK_CURAND( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT)  );
    CHECK_CURAND( hiprandSetPseudoRandomGeneratorSeed(gen, seed)  );
    CHECK_CURAND( hiprandGenerateUniform(gen, dataDev, number)  );
    CHECK_CURAND( hiprandDestroyGenerator(gen)  );
 
    CHECK_CUDA( hipMemcpy(dataHost, dataDev, number * sizeof(float), hipMemcpyDeviceToHost)  );
    CHECK_CUDA( hipFree(dataDev)  );
 
    return;
}

void initGPU(int dev)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    hipSetDevice(dev);
	//NOTE: 48KB shared memory per block, 1024 threads per block, 30 SMs and 128 cores per SM
    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %luB; compute v%d.%d; clock: %d kHz; shared mem: %dB; block threads: %d; SM count: %d\n",
               devProps.name, devProps.totalGlobalMem, 
               (int)devProps.major, (int)devProps.minor, 
               (int)devProps.clockRate,
			   devProps.sharedMemPerBlock, devProps.maxThreadsPerBlock, devProps.multiProcessorCount);
    }
	cout<<"GPU selected"<<endl;
	//GPU initialization needs several seconds, so we do it first and only once
	//https://devtalk.nvidia.com/default/topic/392429/first-cudamalloc-takes-long-time-/
	int* warmup = NULL;
	/*unsigned long bigg = 0x7fffffff;*/
	/*hipMalloc(&warmup, bigg);*/
	/*cout<<"warmup malloc"<<endl;*/
	hipMalloc(&warmup, sizeof(int));
	hipFree(warmup);
	cout<<"GPU warmup finished"<<endl;
	unsigned long size = 0x7fffffff;
	/*size *= 3;   //heap corruption for 3 and 4*/
	size *= 2;
    //NOTICE: it is ok to expand the heap memory capacity to 8G if using long type
    //However, it is not suggested because the remaining memory is too small
    //In fact, the program not ends if using 8G heap memory
    //Once you set the memory limit, then you will see the limit occupation if using nvidia-smi command to see, but it not really occupy so much at once
    /*size *= 2;*/
	//NOTICE: the memory alloced by hipMalloc is different from the GPU heap(for new/malloc in kernel functions)
	hipDeviceSetLimit(hipLimitMallocHeapSize, size);
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
	cout<<"check heap limit: "<<size<<endl;

	// Runtime API
	// hipFuncCachePreferShared: shared memory is 48 KB
	// hipFuncCachePreferEqual: shared memory is 32 KB
	// hipFuncCachePreferL1: shared memory is 16 KB
	// hipFuncCachePreferNone: no preference
	/*hipFuncSetCacheConfig(reinterpret_cast<const void*>(MyKernel), hipFuncCachePreferShared)*/
	//The initial configuration is 48 KB of shared memory and 16 KB of L1 cache
	//The maximum L2 cache size is 3 MB.
	//also 48 KB read-only cache: if accessed via texture/surface memory, also called texture cache;
	//or use _ldg() or const __restrict__
	//64KB constant memory, ? KB texture memory. cache size?
	//CPU的L1 cache是根据时间和空间局部性做出的优化，但是GPU的L1仅仅被设计成针对空间局部性而不包括时间局部性。频繁的获取L1不会导致某些数据驻留在cache中，只要下次用不到，直接删。
	//L1 cache line 128B, L2 cache line 32B, notice that load is cached while store not
	//mmeory read/write is in unit of a cache line
	//the word size of GPU is 32 bits
}


//Dynamic memory allocation in the kernel function of GPU
//https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#dynamic-global-memory-allocation-and-operations
//very slow, dozens of times slower than pre-assigned memory

__global__ void apply1_kernel(float* d_data)
{
    unsigned size = BASE * d_data[threadIdx.x];
    unsigned* ptr = (unsigned*)malloc(size * sizeof(unsigned));
    /*printf("check: thread %d apply %u\n", threadIdx.x, size);*/
}

__global__ void apply2_kernel(float* d_data)
{
    //BETTER: the block size and shared memory usage can be adjusted according to the architecture of GPU

    /*__shared__ int vote;*/
    /*vote = threadIdx.x;*/
    /*__syncthreads();*/
    /*printf("thread %d control the warp %d", vote, threadIdx.x >> 5);*/

    //NOTICE: here we use warp reduce and apply for heap memory as a warp/block
    //if warp_reduce() is extracted to be a device function, then volatile is needed
    //https://blog.csdn.net/q583956932/article/details/81608798
    __shared__ unsigned size[1024];
    size[threadIdx.x] = BASE * d_data[threadIdx.x];
    //here each warp is synchronized
    /*int warp = threadIdx.x >> 5;*/
    int idx = threadIdx.x & 0x1f;
    for(int i = 1; i < 32; i <<= 1)
    {
        int k = 32 - i;
        if(idx < k)
        {
            size[threadIdx.x] += size[threadIdx.x+i];
        }
    }
    if(idx == 0)
    {
        unsigned* ptr = (unsigned*)malloc(size[threadIdx.x] * sizeof(unsigned));
        /*printf("check: warp %d apply %u\n", warp, size[threadIdx.x]);*/
    }
    //TODO: if pointers are needed, then we need to do a prefix-scan on warp, and the pointer of each thread is ptr+offset
    //or using block?  combine apply2 and apply3 because it will be more easy to load balance
}

__global__ void apply3_kernel(float* d_data)
{
    unsigned size = BASE * d_data[threadIdx.x];
    if(size > LIMIT)
    {
        //BETTER: utilizing the original array space, and use the final 3*4 bytes to place flag and pointer of linked list
        unsigned* ptr = (unsigned*)malloc(size * sizeof(unsigned));
    }
}

void apply3(float* d_data)
{
    unsigned* d_result = NULL;
    CHECK_CUDA( hipMalloc( (void **) &d_result, LIMIT * BLOCK_WIDTH * NUM_BLOCKS * sizeof(unsigned)  )  );
    apply3_kernel<<<NUM_BLOCKS, BLOCK_WIDTH>>>(d_data);
}

int main(int argc, const char* argv[])
{
	int dev = 0;
	if(argc == 2)
	{
		dev = atoi(argv[1]);
	}
    initGPU(dev);

    //BETTER: using srand()
    //ULL is a suffix to indicate the type(in case of overflow)
    unsigned long long seed = 1234ULL;
    float* h_data = (float*)malloc(BLOCK_WIDTH * sizeof(float));
    randomGenerator(h_data, BLOCK_WIDTH, seed);
    float* d_data = NULL;
    CHECK_CUDA( hipMalloc( (void **) &d_data, BLOCK_WIDTH * sizeof(float)  )  );
    CHECK_CUDA( hipMemcpy(d_data, h_data, BLOCK_WIDTH * sizeof(float), hipMemcpyHostToDevice)  );
    xfree(h_data);

    long t1 = Util::get_cur_time();

    //RESULT: apply1 uses 45s, apply2 uses 30ms, apply3 uses 1s (512), 16ms(1000), 88ms(900)
    /*apply1_kernel<<<NUM_BLOCKS, BLOCK_WIDTH>>>(d_data);*/
    /*apply2_kernel<<<NUM_BLOCKS, BLOCK_WIDTH>>>(d_data);*/
    apply3(d_data);
	//Below checks if the kernel launches successfully
	checkCudaErrors(hipGetLastError());

	//force the printf()s to flush
	hipDeviceSynchronize();
	//Below checks if the kernel runs and ends successfully
	checkCudaErrors(hipGetLastError());

    long t2 = Util::get_cur_time();
    printf("apply used %lu ms\n", t2 - t1);

    //NOTICE: if we do not release the memory dynamically allocated in kernel functions, they will be resident on GPU mmeory and can be used by later kernel functions(hipDeviceSynchronize does not collect the memory)
    //however, if the whole program exits, these mmeory will also be recycled by GPU
    getchar();
    //stop here to see memory cost using nvidia-smi

    CHECK_CUDA(hipFree(d_data));
    d_data = NULL;

	printf("That's all!\n");

	return 0;
}

